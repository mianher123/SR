#include "hip/hip_runtime.h"

texture<int, 1, hipReadModeElementType> TR;
texture<int, 1, hipReadModeElementType> TG;
texture<int, 1, hipReadModeElementType> TB;
texture<float, 1, hipReadModeElementType> Td0;
texture<float, 1, hipReadModeElementType> Td1;

__device__ void setup_col(int *row0, int *row1, int ww, int h, int index, int *ans){
	for(int i=0; i<h; ++i){
		if(i==0){
			row0[i]=(int)(tex1Dfetch(Td0, 1)*ans[0*ww +index]
						 +tex1Dfetch(Td0, 2)*ans[1*ww +index]);
			row1[i]=(int)(tex1Dfetch(Td1, 1)*ans[0*ww +index]
						 +tex1Dfetch(Td1, 2)*ans[1*ww +index]);
		}
		else if(i==h-1){
			row0[i]=(int)(tex1Dfetch(Td0, 0)*ans[(i-1)*ww +index]
						 +tex1Dfetch(Td0, 1)*ans[i*ww +index]);
			row1[i]=(int)(tex1Dfetch(Td1, 0)*ans[(i-1)*ww +index]
						 +tex1Dfetch(Td1, 1)*ans[i*ww +index]);
		}
		else{
			row0[i]=(int)(tex1Dfetch(Td0, 0)*ans[(i-1)*ww +index]
						 +tex1Dfetch(Td0, 1)*ans[i*ww +index]
						 +tex1Dfetch(Td0, 2)*ans[(i+1)*ww +index]);
			row1[i]=(int)(tex1Dfetch(Td1, 0)*ans[(i-1)*ww +index]
						 +tex1Dfetch(Td1, 1)*ans[i*ww +index]
						 +tex1Dfetch(Td1, 2)*ans[(i+1)*ww +index]);
		}
	}
}

__device__ void setup_row(int *row0, int *row1, int w, int index, texture<int, 1, hipReadModeElementType> rgb){
	for(int i=0; i<w; ++i){
		if(i==0){
			row0[i]=(int)(tex1Dfetch(Td0, 1)*tex1Dfetch(rgb, index*w +0)
						 +tex1Dfetch(Td0, 2)*tex1Dfetch(rgb, index*w +1));
			row1[i]=(int)(tex1Dfetch(Td1, 1)*tex1Dfetch(rgb, index*w +0)
						 +tex1Dfetch(Td1, 2)*tex1Dfetch(rgb, index*w +1));
		}
		else if(i==w-1){
			row0[i]=(int)(tex1Dfetch(Td0, 0)*tex1Dfetch(rgb, index*w +i-1)
						 +tex1Dfetch(Td0, 1)*tex1Dfetch(rgb, index*w +i));
			row1[i]=(int)(tex1Dfetch(Td1, 0)*tex1Dfetch(rgb, index*w +i-1)
						 +tex1Dfetch(Td1, 1)*tex1Dfetch(rgb, index*w +i));
		}
		else{
			row0[i]=(int)(tex1Dfetch(Td0, 0)*tex1Dfetch(rgb, index*w +i-1)
						 +tex1Dfetch(Td0, 1)*tex1Dfetch(rgb, index*w +i)
						 +tex1Dfetch(Td0, 2)*tex1Dfetch(rgb, index*w +i+1));
			row1[i]=(int)(tex1Dfetch(Td1, 0)*tex1Dfetch(rgb, index*w +i-1)
						 +tex1Dfetch(Td1, 1)*tex1Dfetch(rgb, index*w +i)
						 +tex1Dfetch(Td1, 2)*tex1Dfetch(rgb, index*w +i+1));
		}
	}
}

__global__ void run_col(int round, int *ans_R, int *ans_G, int *ans_B, int w, int h, int ww, int hh){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<ww){
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		for(int i=0; i<h; ++i){ // compute weight
			R_ori+=ans_R[i*ww +round+tid];
			G_ori+=ans_G[i*ww +round+tid];
			B_ori+=ans_B[i*ww +round+tid];
		}

		int row0[270];
		int row1[270];
		//red
		setup_col(row0, row1, ww, h, round+tid, ans_R);
		e_aft=0;
		for(int i=0; i<hh; ++i){
			if(i%2==0) ans_R[i*ww +round+tid]=row0[3*i/2];
			else ans_R[i*ww +round+tid]=row1[3*(i-1)/2 +2];
			e_aft+=ans_R[i*ww +round+tid];
		}
		R_rate=(float)e_aft/((float)R_ori*2.0/3.0);
		for(int i=0; i<hh; ++i){
			ans_R[i*ww +round+tid]=(int)((float)ans_R[i*ww +round+tid]/R_rate);
			if(ans_R[i*ww +round+tid]>255) ans_R[i*ww +round+tid]=255;
			else if(ans_R[i*ww +round+tid]<0) ans_R[i*ww +round+tid]=0;
		}
		// green
		setup_col(row0, row1, ww, h, round+tid, ans_G);
		e_aft=0;
		for(int i=0; i<hh; ++i){
			if(i%2==0) ans_G[i*ww +round+tid]=row0[3*i/2];
			else ans_G[i*ww +round+tid]=row1[3*(i-1)/2 +2];
			e_aft+=ans_G[i*ww +round+tid];
		}
		G_rate=(float)e_aft/((float)G_ori*2.0/3.0);
		for(int i=0; i<hh; ++i){
			ans_G[i*ww +round+tid]=(int)((float)ans_G[i*ww +round+tid]/G_rate);
			if(ans_G[i*ww +round+tid]>255) ans_G[i*ww +round+tid]=255;
			else if(ans_G[i*ww +round+tid]<0) ans_G[i*ww +round+tid]=0;
		}
		// blue
		setup_col(row0, row1, ww, h, round+tid, ans_B);
		e_aft=0;
		for(int i=0; i<hh; ++i){
			if(i%2==0) ans_B[i*ww +round+tid]=row0[3*i/2];
			else ans_B[i*ww +round+tid]=row1[3*(i-1)/2 +2];
			e_aft+=ans_B[i*ww +round+tid];
		}
		B_rate=(float)e_aft/((float)B_ori*2.0/3.0);
		for(int i=0; i<hh; ++i){
			ans_B[i*ww +round+tid]=(int)((float)ans_B[i*ww +round+tid]/B_rate);
			if(ans_B[i*ww +round+tid]>255) ans_B[i*ww +round+tid]=255;
			else if(ans_B[i*ww +round+tid]<0) ans_B[i*ww +round+tid]=0;
		}
	}
}

__global__ void run_row(int round, int *ans_R, int *ans_G, int *ans_B, int w, int h, int ww, int hh){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<h){
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		for(int i=0; i<w; ++i){ // compute weight
			R_ori+=tex1Dfetch(TR, (round+tid)*w +i);
			G_ori+=tex1Dfetch(TG, (round+tid)*w +i);
			B_ori+=tex1Dfetch(TB, (round+tid)*w +i);
		}

		int row0[360];
		int row1[360];
		// red
		setup_row(row0, row1, w, round+tid, TR);
		e_aft=0;
		for(int i=0; i<w*2/3; ++i){
			if(i%2==0) ans_R[(round+tid)*w*2/3 +i]=row0[3*i/2];
			else ans_R[(round+tid)*w*2/3 +i]=row1[3*(i-1)/2 +2];
			e_aft+=ans_R[(round+tid)*w*2/3 +i];
		}
		R_rate=(float)e_aft/((float)R_ori*2.0/3.0);
		for(int i=0; i<w*2/3; ++i){
			ans_R[(round+tid)*w*2/3 +i]=(int)((float)ans_R[(round+tid)*w*2/3 +i]/R_rate);
			if(ans_R[(round+tid)*w*2/3 +i]>255) ans_R[(round+tid)*w*2/3 +i]=255;
			else if(ans_R[(round+tid)*w*2/3 +i]<0) ans_R[(round+tid)*w*2/3 +i]=0;
		}
		// green
		setup_row(row0, row1, w, round+tid, TG);
		e_aft=0;
		for(int i=0; i<w*2/3; ++i){
			if(i%2==0) ans_G[(round+tid)*w*2/3 +i]=row0[3*i/2];
			else ans_G[(round+tid)*w*2/3 +i]=row1[3*(i-1)/2 +2];
			e_aft+=ans_G[(round+tid)*w*2/3 +i];
		}
		G_rate=(float)e_aft/((float)G_ori*2.0/3.0);
		for(int i=0; i<w*2/3; ++i){
			ans_G[(round+tid)*w*2/3 +i]=(int)((float)ans_G[(round+tid)*w*2/3 +i]/G_rate);
			if(ans_G[(round+tid)*w*2/3 +i]>255) ans_G[(round+tid)*w*2/3 +i]=255;
			else if(ans_G[(round+tid)*w*2/3 +i]<0) ans_G[(round+tid)*w*2/3 +i]=0;
		}
		// blue
		setup_row(row0, row1, w, round+tid, TB);
		e_aft=0;
		for(int i=0; i<w*2/3; ++i){
			if(i%2==0) ans_B[(round+tid)*w*2/3 +i]=row0[3*i/2];
			else ans_B[(round+tid)*w*2/3 +i]=row1[3*(i-1)/2 +2];
			e_aft+=ans_B[(round+tid)*w*2/3 +i];
		}
		B_rate=(float)e_aft/((float)B_ori*2.0/3.0);
		for(int i=0; i<w*2/3; ++i){
			ans_B[(round+tid)*w*2/3 +i]=(int)((float)ans_B[(round+tid)*w*2/3 +i]/B_rate);
			if(ans_B[(round+tid)*w*2/3 +i]>255) ans_B[(round+tid)*w*2/3 +i]=255;
			else if(ans_B[(round+tid)*w*2/3 +i]<0) ans_B[(round+tid)*w*2/3 +i]=0;
		}
	}
}

void SR_kernel_down(int *ori_R, int *ori_G, int *ori_B, int *aft_R, int *aft_G, int *aft_B, int w, int h){
	float d0[3]={-0.022, 0.974, 0.227};
	float d1[3]={0.227, 0.974, -0.022};

	int *R, *G, *B;
	int *ans_R, *ans_G, *ans_B;
	int *d_d0, *d_d1;
	int ww=w*2/3;
	int hh=h*2/3;

	hipMalloc((void**)&R, w*h*sizeof(int));
	hipMalloc((void**)&G, w*h*sizeof(int));
	hipMalloc((void**)&B, w*h*sizeof(int));
	hipMalloc((void**)&ans_R, w*h*sizeof(int)*2/3);
	hipMalloc((void**)&ans_G, w*h*sizeof(int)*2/3);
	hipMalloc((void**)&ans_B, w*h*sizeof(int)*2/3);
	hipMalloc((void**)&d_d0, 3*sizeof(float));
	hipMalloc((void**)&d_d1, 3*sizeof(float));
	
	hipMemcpy(R, ori_R, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(G, ori_G, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B, ori_B, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_d0, d0, 3*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_d1, d1, 3*sizeof(float), hipMemcpyHostToDevice);

	hipBindTexture(0, TR, R);
	hipBindTexture(0, TG, G);
	hipBindTexture(0, TB, B);
	hipBindTexture(0, Td0, d_d0);
	hipBindTexture(0, Td1, d_d1);

	int threads=64;
	int blocks=64;
	for(int i=0; i<(h-1)/(threads*blocks) +1; ++i)
		run_row<<<threads, blocks>>>(i*threads*blocks, ans_R, ans_G, ans_B, w, h, ww, hh);

	for(int i=0; i<(ww-1)/(threads*blocks) +1; ++i)
		run_col<<<threads, blocks>>>(i*threads*blocks, ans_R, ans_G, ans_B, w, h, ww, hh);
	

	hipMemcpy(aft_R, ans_R, w*h*sizeof(int)*4/9, hipMemcpyDeviceToHost);
	hipMemcpy(aft_G, ans_G, w*h*sizeof(int)*4/9, hipMemcpyDeviceToHost);
	hipMemcpy(aft_B, ans_B, w*h*sizeof(int)*4/9, hipMemcpyDeviceToHost);

	hipUnbindTexture(TR);
	hipUnbindTexture(TG);
	hipUnbindTexture(TB);
	hipFree(R);
	hipFree(G);
	hipFree(B);
	hipFree(ans_R);
	hipFree(ans_G);
	hipFree(ans_B);
	hipFree(d_d0);
	hipFree(d_d1);
}
