#include "hip/hip_runtime.h"

texture<int, 1, hipReadModeElementType> TR;
texture<int, 1, hipReadModeElementType> TG;
texture<int, 1, hipReadModeElementType> TB;
/*
texture<float, 1, hipReadModeElementType> Td0;
texture<float, 1, hipReadModeElementType> Td1;
*/
__constant__ float d_d0[3];
__constant__ float d_d1[3];

//extern __shared__ int row[];

extern "C" void set_filter(float *d0, float *d1){
	hipMemcpyToSymbol(HIP_SYMBOL(d_d0), d0, 3 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_d1), d1, 3 * sizeof(float));
}

__device__ void setup_col(int *row0, int *row1, int ww, int h, int index, int *ans){
	row0[0]=(int)(d_d0[1]*ans[index]+d_d0[2]*ans[ww +index]);
	row1[0]=(int)(d_d1[1]*ans[index]+d_d1[2]*ans[ww +index]);
	row0[h-1]=(int)(d_d0[0]*ans[(h-2)*ww +index]+d_d0[1]*ans[(h-1)*ww +index]);
	row1[h-1]=(int)(d_d1[0]*ans[(h-2)*ww +index]+d_d1[1]*ans[(h-1)*ww +index]);

	#pragma unroll
	for(int i=1; i<h-1; ++i){
		row0[i]=(int)(d_d0[0]*ans[(i-1)*ww +index]
					 +d_d0[1]*ans[i*ww +index]
					 +d_d0[2]*ans[(i+1)*ww +index]);
		row1[i]=(int)(d_d1[0]*ans[(i-1)*ww +index]
					 +d_d1[1]*ans[i*ww +index]
					 +d_d1[2]*ans[(i+1)*ww +index]);		
	}
}

__device__ void setup_row(int *row0, int *row1, int w, int index, texture<int, 1, hipReadModeElementType> rgb){
	/*
	if(tid==0){
		row[0]=(int)(d_d0[1]*tex1Dfetch(rgb, index*w +0)
					+d_d0[2]*tex1Dfetch(rgb, index*w +1));
		row[w]=(int)(d_d1[1]*tex1Dfetch(rgb, index*w +0)
					+d_d1[2]*tex1Dfetch(rgb, index*w +1));
	}
	else if(tid==w-1){
		row[w-1]=(int)(d_d0[0]*tex1Dfetch(rgb, index*w +w-2)
					+d_d0[1]*tex1Dfetch(rgb, index*w +w-1));
		row[w+w-1]=(int)(d_d1[0]*tex1Dfetch(rgb, index*w +w-2)
					+d_d1[1]*tex1Dfetch(rgb, index*w +w-1));
	}
	else{
		row[tid]=(int)(d_d0[0]*tex1Dfetch(rgb, index*w +tid-1)
						 +d_d0[1]*tex1Dfetch(rgb, index*w +tid)
						 +d_d0[2]*tex1Dfetch(rgb, index*w +tid+1));
		row[w+tid]=(int)(d_d1[0]*tex1Dfetch(rgb, index*w +tid-1)
						 +d_d1[1]*tex1Dfetch(rgb, index*w +tid)
						 +d_d1[2]*tex1Dfetch(rgb, index*w +tid+1));
	}*/
	
	row0[0]=(int)(d_d0[1]*tex1Dfetch(rgb, index*w +0)
					+d_d0[2]*tex1Dfetch(rgb, index*w +1));
	row1[0]=(int)(d_d1[1]*tex1Dfetch(rgb, index*w +0)
					+d_d1[2]*tex1Dfetch(rgb, index*w +1));
	row0[w-1]=(int)(d_d0[0]*tex1Dfetch(rgb, index*w +w-2)
					+d_d0[1]*tex1Dfetch(rgb, index*w +w-1));
	row1[w-1]=(int)(d_d1[0]*tex1Dfetch(rgb, index*w +w-2)
					+d_d1[1]*tex1Dfetch(rgb, index*w +w-1));

	#pragma unroll
	for(int i=1; i<w-1; ++i){
		row0[i]=(int)(d_d0[0]*tex1Dfetch(rgb, index*w +i-1)
						 +d_d0[1]*tex1Dfetch(rgb, index*w +i)
						 +d_d0[2]*tex1Dfetch(rgb, index*w +i+1));
		row1[i]=(int)(d_d1[0]*tex1Dfetch(rgb, index*w +i-1)
						 +d_d1[1]*tex1Dfetch(rgb, index*w +i)
						 +d_d1[2]*tex1Dfetch(rgb, index*w +i+1));		
	}
	
}

__global__ void run_col(int round, int *ans_R, int *ans_G, int *ans_B, int w, int h, int ww, int hh){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<ww){
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		int index=round+tid;
		for(int i=0; i<h; ++i){ // compute weight
			R_ori+=ans_R[i*ww +index];
			G_ori+=ans_G[i*ww +index];
			B_ori+=ans_B[i*ww +index];
		}

		int row0[720];
		int row1[720];
		// red
		setup_col(row0, row1, ww, h, index, ans_R);
		e_aft=0;
		for(int i=0; i<hh; ++i){
			if(i%2==0) ans_R[i*ww +index]=row0[3*i/2];
			else ans_R[i*ww +index]=row1[3*(i-1)/2 +2];
			e_aft+=ans_R[i*ww +index];
		}
		R_rate=(float)e_aft/((float)R_ori*2.0/3.0);
		for(int i=0; i<hh; ++i){
			ans_R[i*ww +index]=(int)((float)ans_R[i*ww +index]/R_rate);
			//if(ans_R[i*ww +index]>255) ans_R[i*ww +index]=255;
			//else if(ans_R[i*ww +index]<0) ans_R[i*ww +index]=0;
		}
		// green
		setup_col(row0, row1, ww, h, index, ans_G);
		e_aft=0;
		for(int i=0; i<hh; ++i){
			if(i%2==0) ans_G[i*ww +index]=row0[3*i/2];
			else ans_G[i*ww +index]=row1[3*(i-1)/2 +2];
			e_aft+=ans_G[i*ww +index];
		}
		G_rate=(float)e_aft/((float)G_ori*2.0/3.0);
		for(int i=0; i<hh; ++i){
			ans_G[i*ww +index]=(int)((float)ans_G[i*ww +index]/G_rate);
			//if(ans_G[i*ww +index]>255) ans_G[i*ww +index]=255;
			//else if(ans_G[i*ww +index]<0) ans_G[i*ww +index]=0;
		}
		// blue
		setup_col(row0, row1, ww, h, index, ans_B);
		e_aft=0;
		for(int i=0; i<hh; ++i){
			if(i%2==0) ans_B[i*ww +index]=row0[3*i/2];
			else ans_B[i*ww +index]=row1[3*(i-1)/2 +2];
			e_aft+=ans_B[i*ww +index];
		}
		B_rate=(float)e_aft/((float)B_ori*2.0/3.0);
		for(int i=0; i<hh; ++i){
			ans_B[i*ww +index]=(int)((float)ans_B[i*ww +index]/B_rate);
			//if(ans_B[i*ww +index]>255) ans_B[i*ww +index]=255;
			//else if(ans_B[i*ww +index]<0) ans_B[i*ww +index]=0;
		}
	}
}

__global__ void run_row(int round, int *ans_R, int *ans_G, int *ans_B, int w, int h, int ww, int hh){
	//int bid = blockIdx.x;
	//int tid = threadIdx.x;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<h){
		//__shared__ int weight[4]; // R_ori, G_ori, B_ori, e_aft
		//__shared__ float rate[3]; // R_rate, G_rate, B_rate
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		int index=(round+tid)*w;
		
		for(int i=0; i<w; ++i){ // compute weight
			R_ori+=tex1Dfetch(TR, index +i);
			G_ori+=tex1Dfetch(TG, index +i);
			B_ori+=tex1Dfetch(TB, index +i);
		}

		int row0[1280];
		int row1[1280];
		// red
		setup_row(row0, row1, w, round+tid, TR);
		e_aft=0;
		index=(round+tid)*w*2/3;
		for(int i=0; i<w*2/3; ++i){
			if(i%2==0) ans_R[index +i]=row0[3*i/2];
			else ans_R[index +i]=row1[3*(i-1)/2 +2];
			e_aft+=ans_R[index +i];
		}
		R_rate=(float)e_aft/((float)R_ori*2.0/3.0);
		for(int i=0; i<w*2/3; ++i){
			ans_R[index +i]=(int)((float)ans_R[index +i]/R_rate);
			//if(ans_B[index +i]>255) ans_B[index +i]=255;
			//else if(ans_B[index +i]<0) ans_B[index +i]=0;
		}
		// green
		/*
		setup_row(row0, row1, w, round+bid, TG, tid);
		__syncthreads();
		index=(round+bid)*w*2/3;
		if(tid<w*2/3){
			if(tid%2==0) ans_G[index +tid]=row[3*tid/2];
			else ans_G[index +tid]=row[w+ 3*(tid-1)/2 +2];
			__syncthreads();
			if(tid==0){
				weight[3]=0;
				for(int i=0; i<w*2/3; ++i)
					weight[3]+=ans_G[index+i];
				rate[1]=(float)weight[3]/((float)weight[1]*2.0/3.0);
			}
			__syncthreads();
			ans_G[index +tid]=(int)((float)ans_G[index +tid]/rate[1]);
		}
		__syncthreads();*/
		
		setup_row(row0, row1, w, round+tid, TG);
		e_aft=0;
		for(int i=0; i<w*2/3; ++i){
			if(i%2==0) ans_G[index +i]=row0[3*i/2];
			else ans_G[index +i]=row1[3*(i-1)/2 +2];
			e_aft+=ans_G[index +i];
		}
		G_rate=(float)e_aft/((float)G_ori*2.0/3.0);
		for(int i=0; i<w*2/3; ++i){
			ans_G[index +i]=(int)((float)ans_G[index +i]/G_rate);
		}
		
		// blue
		/*
		setup_row(row0, row1, w, round+bid, TB, tid);
		__syncthreads();
		index=(round+bid)*w*2/3;
		if(tid<w*2/3){
			if(tid%2==0) ans_B[index +tid]=row[3*tid/2];
			else ans_B[index +tid]=row[w+ 3*(tid-1)/2 +2];
			__syncthreads();
			if(tid==0){
				weight[3]=0;
				for(int i=0; i<w*2/3; ++i)
					weight[3]+=ans_B[index+i];
				rate[2]=(float)weight[3]/((float)weight[2]*2.0/3.0);
			}
			__syncthreads();
			ans_B[index +tid]=(int)((float)ans_B[index +tid]/rate[2]);
		}
		__syncthreads();*/
		
		setup_row(row0, row1, w, round+tid, TB);
		e_aft=0;
		for(int i=0; i<w*2/3; ++i){
			if(i%2==0) ans_B[index +i]=row0[3*i/2];
			else ans_B[index +i]=row1[3*(i-1)/2 +2];
			e_aft+=ans_B[index +i];
		}
		B_rate=(float)e_aft/((float)B_ori*2.0/3.0);
		for(int i=0; i<w*2/3; ++i){
			ans_B[index +i]=(int)((float)ans_B[index +i]/B_rate);
			//if(ans_B[index +i]>255) ans_B[index +i]=255;
			//else if(ans_B[index +i]<0) ans_B[index +i]=0;
		}
		
	}
}

void SR_kernel_down(int *ori_R, int *ori_G, int *ori_B, int *aft_R, int *aft_G, int *aft_B, int w, int h){
	float d0[3]={-0.022, 0.974, 0.227};
	float d1[3]={0.227, 0.974, -0.022};

	int *R, *G, *B;
	int *ans_R, *ans_G, *ans_B;
	int ww=w*2/3;
	int hh=h*2/3;

	hipMalloc((void**)&R, w*h*sizeof(int));
	hipMalloc((void**)&G, w*h*sizeof(int));
	hipMalloc((void**)&B, w*h*sizeof(int));
	hipMalloc((void**)&ans_R, w*h*sizeof(int)*2/3);
	hipMalloc((void**)&ans_G, w*h*sizeof(int)*2/3);
	hipMalloc((void**)&ans_B, w*h*sizeof(int)*2/3);
	
	hipMemcpy(R, ori_R, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(G, ori_G, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B, ori_B, w*h*sizeof(int), hipMemcpyHostToDevice);

	hipBindTexture(0, TR, R);
	hipBindTexture(0, TG, G);
	hipBindTexture(0, TB, B);
	set_filter(d0, d1);

	int threads=64;
	int blocks=64;
	/* i want each block do a row, and each thread in a block handle a pixel */
	for(int i=0; i<(h-1)/(threads*blocks) +1; ++i)
		run_row<<<blocks, threads>>>(i*threads*blocks, ans_R, ans_G, ans_B, w, h, ww, hh);
	
	for(int i=0; i<(ww-1)/(threads*blocks) +1; ++i)
		run_col<<<blocks, threads>>>(i*threads*blocks, ans_R, ans_G, ans_B, w, h, ww, hh);
	

	hipMemcpy(aft_R, ans_R, w*h*sizeof(int)*4/9, hipMemcpyDeviceToHost);
	hipMemcpy(aft_G, ans_G, w*h*sizeof(int)*4/9, hipMemcpyDeviceToHost);
	hipMemcpy(aft_B, ans_B, w*h*sizeof(int)*4/9, hipMemcpyDeviceToHost);

	hipUnbindTexture(TR);
	hipUnbindTexture(TG);
	hipUnbindTexture(TB);
	hipFree(R);
	hipFree(G);
	hipFree(B);
	hipFree(ans_R);
	hipFree(ans_G);
	hipFree(ans_B);
}
