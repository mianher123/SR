#include "hip/hip_runtime.h"

texture<unsigned char, 1, hipReadModeElementType> TR;
texture<unsigned char, 1, hipReadModeElementType> TG;
texture<unsigned char, 1, hipReadModeElementType> TB;
texture<unsigned char ,1, hipReadModeElementType> TansR;
texture<unsigned char ,1, hipReadModeElementType> TansG;
texture<unsigned char ,1, hipReadModeElementType> TansB;
/*
texture<float, 1, hipReadModeElementType> Td0;
texture<float, 1, hipReadModeElementType> Td1;
*/
__constant__ float d_d0[3];
__constant__ float d_d1[3];

//extern __shared__ int row[];

extern "C" void set_filter(float *d0, float *d1){
	hipMemcpyToSymbol(HIP_SYMBOL(d_d0), d0, 3*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_d1), d1, 3*sizeof(float));
}

__device__ void setup_col(unsigned char *row0, unsigned char *row1, int ww, int h, int index, unsigned char *ans){
	int temp;

	temp=(int)(d_d0[1]*((int)ans[index])+d_d0[2]*((int)ans[ww +index]));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row0[0]=(unsigned char)temp;

	temp=(int)(d_d1[1]*((int)ans[index])+d_d1[2]*((int)ans[ww +index]));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row1[0]=(unsigned char)temp;

	temp=(int)(d_d0[0]*((int)ans[(h-2)*ww +index])+d_d0[1]*((int)ans[(h-1)*ww +index]));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row0[h-1]=(unsigned char)temp;

	temp=(int)(d_d1[0]*((int)ans[(h-2)*ww +index])+d_d1[1]*((int)ans[(h-1)*ww +index]));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row1[h-1]=(unsigned char)temp;

	#pragma unroll
	for(int i=1; i<h-1; ++i){
		temp=(int)(d_d0[0]*((int)ans[(i-1)*ww +index])+d_d0[1]*((int)ans[i*ww +index])+d_d0[2]*((int)ans[(i+1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[i]=(unsigned char)temp;

		temp=(int)(d_d1[0]*((int)ans[(i-1)*ww +index])+d_d1[1]*((int)ans[i*ww +index])+d_d1[2]*((int)ans[(i+1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[i]=(unsigned char)temp;
	}
}

__device__ void setup_row(unsigned char *row0, unsigned char *row1, int w, int index, texture<unsigned char, 1, hipReadModeElementType> rgb){
	int temp;
	temp=(int)(d_d0[1]*(int)tex1Dfetch(rgb, index*w +0)+d_d0[2]*(int)tex1Dfetch(rgb, index*w +1));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row0[0]=(unsigned char)temp;

	temp=(int)(d_d1[1]*(int)tex1Dfetch(rgb, index*w +0)+d_d1[2]*(int)tex1Dfetch(rgb, index*w +1));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row1[0]=(unsigned char)temp;

	temp=(int)(d_d0[0]*(int)tex1Dfetch(rgb, index*w +w-2)+d_d0[1]*(int)tex1Dfetch(rgb, index*w +w-1));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row0[w-1]=(unsigned char)temp;

	temp=(int)(d_d1[0]*(int)tex1Dfetch(rgb, index*w +w-2)+d_d1[1]*(int)tex1Dfetch(rgb, index*w +w-1));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row1[w-1]=(unsigned char)temp;

	#pragma unroll
	for(int i=1; i<w-1; ++i){
		temp=(int)(d_d0[0]*(int)tex1Dfetch(rgb, index*w +i-1)+d_d0[1]*(int)tex1Dfetch(rgb, index*w +i)+d_d0[2]*(int)tex1Dfetch(rgb, index*w +i+1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[i]=(unsigned char)temp;

		temp=(int)(d_d1[0]*(int)tex1Dfetch(rgb, index*w +i-1)+d_d1[1]*(int)tex1Dfetch(rgb, index*w +i)+d_d1[2]*(int)tex1Dfetch(rgb, index*w +i+1));	
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[i]=(unsigned char)temp;
	}
}

__device__ unsigned char clamp(int value){
	if(value > 255) return (unsigned char)255;
	else if(value < 0) return (unsigned char)0;
	else return value;
}

__global__ void run_col(int round, unsigned char *ans_R, unsigned char *ans_G, unsigned char *ans_B, int w, int h, int ww, int hh){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<ww){
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		int index=round+tid;
		#pragma unroll
		for(int i=0; i<h; ++i){ // compute weight
			R_ori+=(int)ans_R[i*ww +index];
			G_ori+=(int)ans_G[i*ww +index];
			B_ori+=(int)ans_B[i*ww +index];
		}

		//unsigned char row0[720];
		//unsigned char row1[720];
		/////////////////////////////// red ////////////////////////////////////
		//setup_col(row0, row1, ww, h, index, ans_R);
		int temp;
		int bi;
		/*
		temp=(int)(d_d0[1]*((int)tex1Dfetch(TansR, index))+d_d0[2]*((int)tex1Dfetch(TansR, index +ww)));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[0]=(unsigned char)temp;

		temp=(int)(d_d1[1]*((int)tex1Dfetch(TansR, index))+d_d1[2]*((int)tex1Dfetch(TansR, index +ww)));	
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[0]=(unsigned char)temp;

		temp=(int)(d_d0[0]*((int)tex1Dfetch(TansR, index +(h-2)*ww))+d_d0[1]*((int)tex1Dfetch(TansR, index +(h-1)*ww)));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[h-1]=(unsigned char)temp;

		temp=(int)(d_d1[0]*((int)tex1Dfetch(TansR, index +(h-2))*ww)+d_d1[1]*((int)tex1Dfetch(TansR, index +(h-1)*ww)));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[h-1]=(unsigned char)temp;

		#pragma unroll
		for(int i=1; i<h-1; ++i){
			temp=(int)(d_d0[0]*((int)tex1Dfetch(TansR, index +(i-1)*ww))+d_d0[1]*((int)tex1Dfetch(TansR, index +i*ww))+d_d0[2]*((int)tex1Dfetch(TansR, index +(i+1)*ww)));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row0[i]=(unsigned char)temp;
	
			temp=(int)(d_d1[0]*((int)tex1Dfetch(TansR, index +(i-1)*ww))+d_d1[1]*((int)tex1Dfetch(TansR, index +i*ww))+d_d1[2]*((int)tex1Dfetch(TansR, index +(i+1)*ww)));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row1[i]=(unsigned char)temp;
		}*/
		/*
		temp=(int)(d_d0[1]*((int)ans_R[index])+d_d0[2]*((int)ans_R[ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[0]=(unsigned char)temp;

		temp=(int)(d_d1[1]*((int)ans_R[index])+d_d1[2]*((int)ans_R[ww +index]));	
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[0]=(unsigned char)temp;

		temp=(int)(d_d0[0]*((int)ans_R[(h-2)*ww +index])+d_d0[1]*((int)ans_R[(h-1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[h-1]=(unsigned char)temp;

		temp=(int)(d_d1[0]*((int)ans_R[(h-2)*ww +index])+d_d1[1]*((int)ans_R[(h-1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[h-1]=(unsigned char)temp;

		#pragma unroll
		for(int i=1; i<h-1; ++i){
			temp=(int)(d_d0[0]*((int)ans_R[(i-1)*ww +index])+d_d0[1]*((int)ans_R[i*ww +index])+d_d0[2]*((int)ans_R[(i+1)*ww +index]));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row0[i]=(unsigned char)temp;
	
			temp=(int)(d_d1[0]*((int)ans_R[(i-1)*ww +index])+d_d1[1]*((int)ans_R[i*ww +index])+d_d1[2]*((int)ans_R[(i+1)*ww +index]));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row1[i]=(unsigned char)temp;
		}*/
		// setup_col() finish
		
		e_aft=0;
		// i==0
		temp=(int)(d_d0[1]*(int)tex1Dfetch(TansR, index)+d_d0[2]*(int)tex1Dfetch(TansR, ww+ index));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		ans_R[index]=(unsigned char)temp;
		e_aft+=temp;
		
		#pragma unroll
		for(int i=2; i<=hh-2; i+=2){
			bi=3*i/2;
			temp=(int)(d_d0[0]*(int)tex1Dfetch(TansR, (bi-1)*ww +index)+d_d0[1]*(int)tex1Dfetch(TansR, bi*ww +index)+d_d0[2]*(int)tex1Dfetch(TansR, (bi+1)*ww +index));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			ans_R[i*ww +index]=(unsigned char)temp;
			e_aft+=temp;
		}

		// i==hh-1
		temp=(int)(d_d1[0]*(int)tex1Dfetch(TansR, (h-2)*ww +index)+d_d1[1]*(int)tex1Dfetch(TansR, (h-1)*ww +index));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		ans_R[(hh-1)*ww +index]=(unsigned char)temp;
		e_aft+=temp;

		#pragma unroll
		for(int i=1; i<=hh-3; i+=2){
			bi=3*(i-1)/2 +2;
			temp=(int)(d_d1[0]*(int)tex1Dfetch(TansR, (bi-1)*ww +index)+d_d1[1]*(int)tex1Dfetch(TansR, bi*ww +index)+d_d1[2]*(int)tex1Dfetch(TansR, (bi+1)*ww +index));	
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			ans_R[i*ww +index]=(unsigned char)temp;
			e_aft+=temp;
		}
		/*
		#pragma unroll
		for(int i=0; i<hh; ++i){
			if(i%2==0) ans_R[i*ww +index]=row0[3*i/2];
			else ans_R[i*ww +index]=row1[3*(i-1)/2 +2];
			e_aft+=ans_R[i*ww +index];
		}*/
		R_rate=(float)e_aft/((float)R_ori*2.0/3.0);
		if(R_rate<1.0){
			#pragma unroll
			for(int i=0; i<hh; ++i){
				temp=(int)ans_R[i*ww +index];
				temp=(int)((float)temp/R_rate);
				if(temp>255) temp=255;
				else if(temp<0) temp=0;
				ans_R[i*ww +index]=(unsigned char)temp;
			}
		}
		else{
			#pragma unroll
			for(int i=0; i<hh; ++i){
				temp=(int)ans_R[i*ww +index];
				temp=(int)((float)temp/R_rate);
				ans_R[i*ww +index]=(unsigned char)temp;
			}
		}
		////////////////////////// green ///////////////////////////
		//setup_col(row0, row1, ww, h, index, ans_G);
		/*
		temp=(int)(d_d0[1]*((int)ans_G[index])+d_d0[2]*((int)ans_G[ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[0]=(unsigned char)temp;

		temp=(int)(d_d1[1]*((int)ans_G[index])+d_d1[2]*((int)ans_G[ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[0]=(unsigned char)temp;

		temp=(int)(d_d0[0]*((int)ans_G[(h-2)*ww +index])+d_d0[1]*((int)ans_G[(h-1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[h-1]=(unsigned char)temp;
	
		temp=(int)(d_d1[0]*((int)ans_G[(h-2)*ww +index])+d_d1[1]*((int)ans_G[(h-1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[h-1]=(unsigned char)temp;

		#pragma unroll
		for(int i=1; i<h-1; ++i){
			temp=(int)(d_d0[0]*((int)ans_G[(i-1)*ww +index])+d_d0[1]*((int)ans_G[i*ww +index])+d_d0[2]*((int)ans_G[(i+1)*ww +index]));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row0[i]=(unsigned char)temp;

			temp=(int)(d_d1[0]*((int)ans_G[(i-1)*ww +index])+d_d1[1]*((int)ans_G[i*ww +index])+d_d1[2]*((int)ans_G[(i+1)*ww +index]));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row1[i]=(unsigned char)temp;
		}*/
		// setup_col() finish
		e_aft=0;
		e_aft=0;
		// i==0
		temp=(int)(d_d0[1]*(int)tex1Dfetch(TansG, index)+d_d0[2]*(int)tex1Dfetch(TansG, ww+ index));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		ans_G[index]=(unsigned char)temp;
		e_aft+=temp;
		
		#pragma unroll
		for(int i=2; i<=hh-2; i+=2){
			bi=3*i/2;
			temp=(int)(d_d0[0]*(int)tex1Dfetch(TansG, (bi-1)*ww +index)+d_d0[1]*(int)tex1Dfetch(TansG, bi*ww +index)+d_d0[2]*(int)tex1Dfetch(TansG, (bi+1)*ww +index));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			ans_G[i*ww +index]=(unsigned char)temp;
			e_aft+=temp;
		}

		// i==hh-1
		temp=(int)(d_d1[0]*(int)tex1Dfetch(TansG, (h-2)*ww +index)+d_d1[1]*(int)tex1Dfetch(TansG, (h-1)*ww +index));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		ans_G[(hh-1)*ww +index]=(unsigned char)temp;
		e_aft+=temp;

		#pragma unroll
		for(int i=1; i<=hh-3; i+=2){
			bi=3*(i-1)/2 +2;
			temp=(int)(d_d1[0]*(int)tex1Dfetch(TansG, (bi-1)*ww +index)+d_d1[1]*(int)tex1Dfetch(TansG, bi*ww +index)+d_d1[2]*(int)tex1Dfetch(TansG, (bi+1)*ww +index));	
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			ans_G[i*ww +index]=(unsigned char)temp;
			e_aft+=temp;
		}/*
		#pragma unroll
		for(int i=0; i<hh; ++i){
			if(i%2==0) ans_G[i*ww +index]=row0[3*i/2];
			else ans_G[i*ww +index]=row1[3*(i-1)/2 +2];
			e_aft+=ans_G[i*ww +index];
		}*/
		G_rate=(float)e_aft/((float)G_ori*2.0/3.0);
		if(G_rate<1.0){
			#pragma unroll
			for(int i=0; i<hh; ++i){
				temp=(int)ans_G[i*ww +index];
				temp=(int)((float)temp/G_rate);
				if(temp>255) temp=255;
				else if(temp<0) temp=0;
				ans_G[i*ww +index]=(unsigned char)temp;
			}
		}
		else{
			#pragma unroll
			for(int i=0; i<hh; ++i){
				temp=(int)ans_G[i*ww +index];
				temp=(int)((float)temp/G_rate);
				ans_G[i*ww +index]=(unsigned char)temp;
			}
		}
		///////////////////////////////// blue /////////////////////////////////
		//setup_col(row0, row1, ww, h, index, ans_B);
		/*
		temp=(int)(d_d0[1]*((int)ans_B[index])+d_d0[2]*((int)ans_B[ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[0]=(unsigned char)temp;

		temp=(int)(d_d1[1]*((int)ans_B[index])+d_d1[2]*((int)ans_B[ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[0]=(unsigned char)temp;

		temp=(int)(d_d0[0]*((int)ans_B[(h-2)*ww +index])+d_d0[1]*((int)ans_B[(h-1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[h-1]=(unsigned char)temp;

		temp=(int)(d_d1[0]*((int)ans_B[(h-2)*ww +index])+d_d1[1]*((int)ans_B[(h-1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[h-1]=(unsigned char)temp;

		#pragma unroll
		for(int i=1; i<h-1; ++i){
			temp=(int)(d_d0[0]*((int)ans_B[(i-1)*ww +index])+d_d0[1]*((int)ans_B[i*ww +index])+d_d0[2]*((int)ans_B[(i+1)*ww +index]));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row0[i]=(unsigned char)temp;
			
			temp=(int)(d_d1[0]*((int)ans_B[(i-1)*ww +index])+d_d1[1]*((int)ans_B[i*ww +index])+d_d1[2]*((int)ans_B[(i+1)*ww +index]));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row1[i]=(unsigned char)temp;
		}*/
		// setup_col() finish

		e_aft=0;
		e_aft=0;
		// i==0
		temp=(int)(d_d0[1]*(int)tex1Dfetch(TansB, index)+d_d0[2]*(int)tex1Dfetch(TansB, ww+ index));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		ans_B[index]=(unsigned char)temp;
		e_aft+=temp;
		
		#pragma unroll
		for(int i=2; i<=hh-2; i+=2){
			bi=3*i/2;
			temp=(int)(d_d0[0]*(int)tex1Dfetch(TansB, (bi-1)*ww +index)+d_d0[1]*(int)tex1Dfetch(TansB, bi*ww +index)+d_d0[2]*(int)tex1Dfetch(TansB, (bi+1)*ww +index));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			ans_B[i*ww +index]=(unsigned char)temp;
			e_aft+=temp;
		}

		// i==hh-1
		temp=(int)(d_d1[0]*(int)tex1Dfetch(TansB, (h-2)*ww +index)+d_d1[1]*(int)tex1Dfetch(TansB, (h-1)*ww +index));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		ans_B[(hh-1)*ww +index]=(unsigned char)temp;
		e_aft+=temp;

		#pragma unroll
		for(int i=1; i<=hh-3; i+=2){
			bi=3*(i-1)/2 +2;
			temp=(int)(d_d1[0]*(int)tex1Dfetch(TansB, (bi-1)*ww +index)+d_d1[1]*(int)tex1Dfetch(TansB, bi*ww +index)+d_d1[2]*(int)tex1Dfetch(TansB, (bi+1)*ww +index));	
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			ans_B[i*ww +index]=(unsigned char)temp;
			e_aft+=temp;
		}/*
		#pragma unroll
		for(int i=0; i<hh; ++i){
			if(i%2==0) ans_B[i*ww +index]=row0[3*i/2];
			else ans_B[i*ww +index]=row1[3*(i-1)/2 +2];
			e_aft+=ans_B[i*ww +index];
		}*/
		B_rate=(float)e_aft/((float)B_ori*2.0/3.0);
		if(B_rate<1.0){
			#pragma unroll
			for(int i=0; i<hh; ++i){
				temp=(int)ans_B[i*ww +index];
				temp=(int)((float)temp/B_rate);
				if(temp>255) temp=255;
				else if(temp<0) temp=0;
				ans_B[i*ww +index]=(unsigned char)temp;
			}
		}
		else{
			#pragma unroll
			for(int i=0; i<hh; ++i){
				temp=(int)ans_B[i*ww +index];
				temp=(int)((float)temp/B_rate);
				ans_B[i*ww +index]=(unsigned char)temp;
			}
		}
	}
}

__global__ void run_row(
	int row_base,
	unsigned char *ans_R, unsigned char *ans_G, unsigned char *ans_B,
	int w, int h, int ww, int hh,
	unsigned char *temp_R, unsigned char *temp_G, unsigned char *temp_B){

	// Calculate processing row number.
	int row_num = blockDim.x * blockIdx.x + threadIdx.x + row_base;

	if(row_num < h){ 

		// Sum of pixels on this row in the original image.
		//int R_ori_sum=0, G_ori_sum=0, B_ori_sum=0; 
		int ori_sum;

		// Sum of pixels on this row after filtering.
		int aft_sum;

		//float R_ratio, G_ratil, B_ratio;
		float norm_factor;

		// Index of base for this row.
		int row_ori_index_base = row_num*w;
		int row_aft_index_base = row_num*ww;

		// Storage for current column.
		int ori_index_offset;
		int aft_index_offset;

		// Register storage for currently processing piece of original image.
		unsigned char ori_piece[3];

		// Temporary storage for pixel value.
		unsigned char temp0, temp1;

		// Initialize sum of row.
		ori_sum = 0;
		aft_sum = 0;
		ori_index_offset = 0;
		aft_index_offset = 0;

		#pragma unroll
		for(; aft_index_offset <= ww-2; ori_index_offset += 3, aft_index_offset += 2){

			ori_piece[0] = tex1Dfetch(TR, row_ori_index_base + ori_index_offset);
			ori_piece[1] = tex1Dfetch(TR, row_ori_index_base + ori_index_offset + 1);
			ori_piece[2] = tex1Dfetch(TR, row_ori_index_base + ori_index_offset + 2);
			ori_sum += (int)ori_piece[0] + (int)ori_piece[1] + (int)ori_piece[2];

			temp0 = clamp (
				d_d0[0]*(int)ori_piece[0] +
				d_d0[1]*(int)ori_piece[1] +
				d_d0[2]*(int)ori_piece[2] );

			temp1 = clamp(
				d_d1[0]*(int)ori_piece[0] +
				d_d1[1]*(int)ori_piece[1] +
				d_d1[2]*(int)ori_piece[2]);

			ans_R[row_aft_index_base + aft_index_offset] = (unsigned char)temp0;
			ans_R[row_aft_index_base + aft_index_offset + 1] = (unsigned char)temp1;
			aft_sum += (int)temp0 + (int)temp1;
		}

		norm_factor =  ((float)ori_sum*2.0/3.0) / (float)aft_sum;
		for(int i=0; i<ww; ++i){
			temp0 = clamp ( (int) ((float)ans_R[row_aft_index_base +i] * norm_factor) );
			temp_R[row_aft_index_base + i]=ans_R[row_aft_index_base + i] = (unsigned char)temp0;
		}

		ori_sum = 0;
		aft_sum = 0;
		ori_index_offset = 0;
		aft_index_offset = 0;

		#pragma unroll
		for(; aft_index_offset <= ww-2; ori_index_offset += 3, aft_index_offset += 2){

			ori_piece[0] = tex1Dfetch(TG, row_ori_index_base + ori_index_offset);
			ori_piece[1] = tex1Dfetch(TG, row_ori_index_base + ori_index_offset + 1);
			ori_piece[2] = tex1Dfetch(TG, row_ori_index_base + ori_index_offset + 2);
			ori_sum += (int)ori_piece[0] + (int)ori_piece[1] + (int)ori_piece[2];

			temp0 = clamp (
				d_d0[0]*(int)ori_piece[0] +
				d_d0[1]*(int)ori_piece[1] +
				d_d0[2]*(int)ori_piece[2] );

			temp1 = clamp(
				d_d1[0]*(int)ori_piece[0] +
				d_d1[1]*(int)ori_piece[1] +
				d_d1[2]*(int)ori_piece[2]);

			ans_G[row_aft_index_base + aft_index_offset] = (unsigned char)temp0;
			ans_G[row_aft_index_base + aft_index_offset + 1] = (unsigned char)temp1;
			aft_sum += (int)temp0 + (int)temp1;
		}

		norm_factor =  ((float)ori_sum*2.0/3.0) / (float)aft_sum;
		for(int i=0; i<ww; ++i){
			temp0 = clamp ( (int) ((float)ans_G[row_aft_index_base +i] * norm_factor) );
			temp_G[row_aft_index_base + i] = ans_G[row_aft_index_base + i] = (unsigned char)temp0;
		}
		
		ori_sum = 0;
		aft_sum = 0;
		ori_index_offset = 0;
		aft_index_offset = 0;

		#pragma unroll
		for(; aft_index_offset <= ww-2; ori_index_offset += 3, aft_index_offset += 2){

			ori_piece[0] = tex1Dfetch(TB, row_ori_index_base + ori_index_offset);
			ori_piece[1] = tex1Dfetch(TB, row_ori_index_base + ori_index_offset + 1);
			ori_piece[2] = tex1Dfetch(TB, row_ori_index_base + ori_index_offset + 2);
			ori_sum += (int)ori_piece[0] + (int)ori_piece[1] + (int)ori_piece[2];

			temp0 = clamp (
				d_d0[0]*(int)ori_piece[0] +
				d_d0[1]*(int)ori_piece[1] +
				d_d0[2]*(int)ori_piece[2] );

			temp1 = clamp(
				d_d1[0]*(int)ori_piece[0] +
				d_d1[1]*(int)ori_piece[1] +
				d_d1[2]*(int)ori_piece[2]);

			ans_B[row_aft_index_base + aft_index_offset] = (unsigned char)temp0;
			ans_B[row_aft_index_base + aft_index_offset + 1] = (unsigned char)temp1;
			aft_sum += (int)temp0 + (int)temp1;
		}

		norm_factor =  ((float)ori_sum*2.0/3.0) / (float)aft_sum;
		for(int i=0; i<ww; ++i){
			temp0 = clamp ( (int) ((float)ans_B[row_aft_index_base +i] * norm_factor) );
			temp_B[row_aft_index_base + i] = ans_B[row_aft_index_base + i] = (unsigned char)temp0;
		}
	}
}

void SR_kernel_down(
	unsigned char *ori_R, unsigned char *ori_G, unsigned char *ori_B,
	unsigned char *aft_R, unsigned char *aft_G, unsigned char *aft_B,
	int w, int h){

	float d0[3]={0.227, 0.974, -0.022};
	float d1[3]={-0.022, 0.974, 0.227};

	unsigned char *R, *G, *B;
	unsigned char *ans_R, *ans_G, *ans_B;
	unsigned char *temp_R, *temp_G, *temp_B;
	int ww=w*2/3;
	int hh=h*2/3;

	hipMalloc((void**)&R, w*h*sizeof(unsigned char));
	hipMalloc((void**)&G, w*h*sizeof(unsigned char));
	hipMalloc((void**)&B, w*h*sizeof(unsigned char));
	hipMalloc((void**)&temp_R, ww*h*sizeof(unsigned char));
	hipMalloc((void**)&temp_G, ww*h*sizeof(unsigned char));
	hipMalloc((void**)&temp_B, ww*h*sizeof(unsigned char));
	hipMalloc((void**)&ans_R, w*h*sizeof(unsigned char)*2/3);
	hipMalloc((void**)&ans_G, w*h*sizeof(unsigned char)*2/3);
	hipMalloc((void**)&ans_B, w*h*sizeof(unsigned char)*2/3);
	
	hipMemcpy(R, ori_R, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(G, ori_G, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(B, ori_B, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);

	hipBindTexture(0, TR, R);
	hipBindTexture(0, TG, G);
	hipBindTexture(0, TB, B);
	set_filter(d0, d1);

	int threads=64;
	int blocks=64;
	/* i want each block do a row, and each thread in a block handle a pixel */
	for(int i=0; i<(h-1)/(threads*blocks) +1; ++i)
		run_row<<<blocks, threads>>>(i*threads*blocks, ans_R, ans_G, ans_B, w, h, ww, hh, temp_R, temp_G, temp_B);
	
	hipBindTexture(0, TansR, temp_R);
	hipBindTexture(0, TansG, temp_G);
	hipBindTexture(0, TansB, temp_B);

	for(int i=0; i<(ww-1)/(threads*blocks) +1; ++i)
		run_col<<<blocks, threads>>>(i*threads*blocks, ans_R, ans_G, ans_B, w, h, ww, hh);
	

	hipMemcpy(aft_R, ans_R, w*h*sizeof(unsigned char)*4/9, hipMemcpyDeviceToHost);
	hipMemcpy(aft_G, ans_G, w*h*sizeof(unsigned char)*4/9, hipMemcpyDeviceToHost);
	hipMemcpy(aft_B, ans_B, w*h*sizeof(unsigned char)*4/9, hipMemcpyDeviceToHost);

	hipUnbindTexture(TR);
	hipUnbindTexture(TG);
	hipUnbindTexture(TB);
	hipUnbindTexture(TansR);
	hipUnbindTexture(TansG);
	hipUnbindTexture(TansB);
	hipFree(R);
	hipFree(G);
	hipFree(B);
	hipFree(ans_R);
	hipFree(ans_G);
	hipFree(ans_B);
	hipFree(temp_R);
	hipFree(temp_G);
	hipFree(temp_B);
}
