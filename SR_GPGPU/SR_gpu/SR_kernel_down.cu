#include "hip/hip_runtime.h"

texture<unsigned char, 1, hipReadModeElementType> TR;
texture<unsigned char, 1, hipReadModeElementType> TG;
texture<unsigned char, 1, hipReadModeElementType> TB;
texture<unsigned char ,1, hipReadModeElementType> TansR;
texture<unsigned char ,1, hipReadModeElementType> TansG;
texture<unsigned char ,1, hipReadModeElementType> TansB;
/*
texture<float, 1, hipReadModeElementType> Td0;
texture<float, 1, hipReadModeElementType> Td1;
*/
__constant__ float d_d0[3];
__constant__ float d_d1[3];

//extern __shared__ int row[];

extern "C" void set_filter(float *d0, float *d1){
	hipMemcpyToSymbol(HIP_SYMBOL(d_d0), d0, 3*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_d1), d1, 3*sizeof(float));
}

__device__ unsigned char clamp(int value){
	if(value > 255) return (unsigned char)255;
	else if(value < 0) return (unsigned char)0;
	else return value;
}

__global__ void run_col(
	int col_base,
	unsigned char *ans_R, unsigned char *ans_G, unsigned char *ans_B,
	int w, int h, int ww, int hh){


	// Calculate processing column number.
	int col_num = blockDim.x * blockIdx.x + threadIdx.x + col_base;

	if(col_num < ww){ 

		// Sum of pixels on this column in the original and the output image.
		int ori_sum, aft_sum;

		//float R_ratio, G_ratil, B_ratio;
		float norm_factor;

		// Index of base for this column.
		int col_ori_index_base = col_num;
		int col_aft_index_base = col_num;

		// Storage for current row offset in the original and the output image.
		int ori_index_offset, aft_index_offset;

		// Register storage for currently processing piece of original image.
		unsigned char ori_piece[4];

		// Temporary storage for pixel value.
		unsigned char temp0, temp1;

		// RED
		// Initialize offset and sum of column.
		ori_sum = 0; aft_sum = 0; ori_index_offset = 0; aft_index_offset = 0;

		// Apply filter for each group of two pixels.
		ori_piece[3] = tex1Dfetch(TansR, col_ori_index_base);
		#pragma unroll
		for(; aft_index_offset <= (hh-2)*ww; ori_index_offset += 3*ww, aft_index_offset += 2*ww){

			ori_piece[0] = ori_piece[3];
			ori_piece[1] = tex1Dfetch(TansR, col_ori_index_base + ori_index_offset);
			ori_piece[2] = tex1Dfetch(TansR, col_ori_index_base + ori_index_offset + ww);
			ori_piece[3] = tex1Dfetch(TansR, col_ori_index_base + ori_index_offset + 2*ww);
			ori_sum += (int)ori_piece[1] + (int)ori_piece[2] + (int)ori_piece[3];

			temp0 = clamp (
				d_d0[0]*(int)ori_piece[0] +
				d_d0[1]*(int)ori_piece[1] +
				d_d0[2]*(int)ori_piece[2] );

			temp1 = clamp(
				d_d1[0]*(int)ori_piece[1] +
				d_d1[1]*(int)ori_piece[2] +
				d_d1[2]*(int)ori_piece[3]);

			ans_R[col_aft_index_base + aft_index_offset] = (unsigned char)temp0;
			ans_R[col_aft_index_base + aft_index_offset + ww] = (unsigned char)temp1;
			aft_sum += (int)temp0 + (int)temp1;
		}

		// Normalization.
		norm_factor =  ((float)ori_sum*2.0/3.0) / (float)aft_sum;
		for(int i=0; i<hh; ++i){
			temp0 = clamp ( (int) ((float)ans_R[col_aft_index_base + i*ww] * norm_factor) );
			ans_R[col_aft_index_base + i*ww] = (unsigned char)temp0;
		}

		// GREEN
		// Initialize offset and sum of column.
		ori_sum = 0; aft_sum = 0; ori_index_offset = 0; aft_index_offset = 0;

		// Apply filter for each group of two pixels.
		ori_piece[3] = tex1Dfetch(TansG, col_ori_index_base);
		#pragma unroll
		for(; aft_index_offset <= (hh-2)*ww; ori_index_offset += 3*ww, aft_index_offset += 2*ww){

			ori_piece[0] = ori_piece[3];
			ori_piece[1] = tex1Dfetch(TansG, col_ori_index_base + ori_index_offset);
			ori_piece[2] = tex1Dfetch(TansG, col_ori_index_base + ori_index_offset + ww);
			ori_piece[3] = tex1Dfetch(TansG, col_ori_index_base + ori_index_offset + 2*ww);
			ori_sum += (int)ori_piece[1] + (int)ori_piece[2] + (int)ori_piece[3];

			temp0 = clamp (
				d_d0[0]*(int)ori_piece[0] +
				d_d0[1]*(int)ori_piece[1] +
				d_d0[2]*(int)ori_piece[2] );

			temp1 = clamp(
				d_d1[0]*(int)ori_piece[1] +
				d_d1[1]*(int)ori_piece[2] +
				d_d1[2]*(int)ori_piece[3]);

			ans_G[col_aft_index_base + aft_index_offset] = (unsigned char)temp0;
			ans_G[col_aft_index_base + aft_index_offset + ww] = (unsigned char)temp1;
			aft_sum += (int)temp0 + (int)temp1;
		}

		// Normalization.
		norm_factor =  ((float)ori_sum*2.0/3.0) / (float)aft_sum;
		for(int i=0; i<hh; ++i){
			temp0 = clamp ( (int) ((float)ans_G[col_aft_index_base + i*ww] * norm_factor) );
			ans_G[col_aft_index_base + i*ww] = (unsigned char)temp0;
		}
		
		// BLUE
		// Initialize offset and sum of column.
		ori_sum = 0; aft_sum = 0; ori_index_offset = 0; aft_index_offset = 0;

		// Apply filter for each group of two pixels.
		ori_piece[3] = tex1Dfetch(TansB, col_ori_index_base);
		#pragma unroll
		for(; aft_index_offset <= (hh-2)*ww; ori_index_offset += 3*ww, aft_index_offset += 2*ww){

			ori_piece[0] = ori_piece[3];
			ori_piece[1] = tex1Dfetch(TansB, col_ori_index_base + ori_index_offset);
			ori_piece[2] = tex1Dfetch(TansB, col_ori_index_base + ori_index_offset + ww);
			ori_piece[3] = tex1Dfetch(TansB, col_ori_index_base + ori_index_offset + 2*ww);
			ori_sum += (int)ori_piece[1] + (int)ori_piece[2] + (int)ori_piece[3];

			temp0 = clamp (
				d_d0[0]*(int)ori_piece[0] +
				d_d0[1]*(int)ori_piece[1] +
				d_d0[2]*(int)ori_piece[2] );

			temp1 = clamp(
				d_d1[0]*(int)ori_piece[1] +
				d_d1[1]*(int)ori_piece[2] +
				d_d1[2]*(int)ori_piece[3]);

			ans_B[col_aft_index_base + aft_index_offset] = (unsigned char)temp0;
			ans_B[col_aft_index_base + aft_index_offset + ww] = (unsigned char)temp1;
			aft_sum += (int)temp0 + (int)temp1;
		}

		// Normalization.
		norm_factor =  ((float)ori_sum*2.0/3.0) / (float)aft_sum;
		for(int i=0; i<hh; ++i){
			temp0 = clamp ( (int) ((float)ans_B[col_aft_index_base + i*ww] * norm_factor) );
			ans_B[col_aft_index_base + i*ww] = (unsigned char)temp0;
		}
	}
}


__global__ void run_row(
	int row_base,
	unsigned char *ans_R, unsigned char *ans_G, unsigned char *ans_B,
	int w, int h, int ww, int hh,
	unsigned char *temp_R, unsigned char *temp_G, unsigned char *temp_B){

	// Calculate processing row number.
	int row_num = blockDim.x * blockIdx.x + threadIdx.x + row_base;

	if(row_num < h){ 

		// Sum of pixels on this row in the original and the output image.
		int ori_sum, aft_sum;

		//float R_ratio, G_ratil, B_ratio;
		float norm_factor;

		// Index of base for this row.
		int row_ori_index_base = row_num*w;
		int row_aft_index_base = row_num*ww;

		// Storage for current column offset in the original and the output image.
		int ori_index_offset, aft_index_offset;

		// Register storage for currently processing piece of original image.
		unsigned char ori_piece[4];

		// Temporary storage for pixel value.
		unsigned char temp0, temp1;

		// RED
		// Initialize offset and sum of row.
		ori_sum = 0; aft_sum = 0; ori_index_offset = 0; aft_index_offset = 0;

		// Apply filter for each group of two pixels.
		ori_piece[3] = tex1Dfetch(TR, row_ori_index_base);
		#pragma unroll
		for(; aft_index_offset <= ww-2; ori_index_offset += 3, aft_index_offset += 2){

			ori_piece[0] = ori_piece[3];
			ori_piece[1] = tex1Dfetch(TR, row_ori_index_base + ori_index_offset);
			ori_piece[2] = tex1Dfetch(TR, row_ori_index_base + ori_index_offset +1);
			ori_piece[3] = tex1Dfetch(TR, row_ori_index_base + ori_index_offset +2);
			ori_sum += (int)ori_piece[1] + (int)ori_piece[2] + (int)ori_piece[3];

			temp0 = clamp (
				d_d0[0]*(int)ori_piece[0] +
				d_d0[1]*(int)ori_piece[1] +
				d_d0[2]*(int)ori_piece[2] );

			temp1 = clamp(
				d_d1[0]*(int)ori_piece[1] +
				d_d1[1]*(int)ori_piece[2] +
				d_d1[2]*(int)ori_piece[3] );

			ans_R[row_aft_index_base + aft_index_offset] = (unsigned char)temp0;
			ans_R[row_aft_index_base + aft_index_offset + 1] = (unsigned char)temp1;
			aft_sum += (int)temp0 + (int)temp1;
		}

		// Normalization.
		norm_factor =  ((float)ori_sum*2.0/3.0) / (float)aft_sum;
		for(int i=0; i<ww; ++i){
			temp0 = clamp ( (int) ((float)ans_R[row_aft_index_base +i] * norm_factor) );
			temp_R[row_aft_index_base + i]=ans_R[row_aft_index_base + i] = (unsigned char)temp0;
		}

		// GREEN
		// Initialize offset and sum of row.
		ori_sum = 0;
		aft_sum = 0;
		ori_index_offset = 0;
		aft_index_offset = 0;

		// Apply filter for each group of two pixels.
		ori_piece[3] = tex1Dfetch(TG, row_ori_index_base);
		#pragma unroll
		for(; aft_index_offset <= ww-2; ori_index_offset += 3, aft_index_offset += 2){

			ori_piece[0] = ori_piece[3];
			ori_piece[1] = tex1Dfetch(TG, row_ori_index_base + ori_index_offset);
			ori_piece[2] = tex1Dfetch(TG, row_ori_index_base + ori_index_offset +1);
			ori_piece[3] = tex1Dfetch(TG, row_ori_index_base + ori_index_offset +2);
			ori_sum += (int)ori_piece[1] + (int)ori_piece[2] + (int)ori_piece[3];

			temp0 = clamp (
				d_d0[0]*(int)ori_piece[0] +
				d_d0[1]*(int)ori_piece[1] +
				d_d0[2]*(int)ori_piece[2] );

			temp1 = clamp(
				d_d1[0]*(int)ori_piece[1] +
				d_d1[1]*(int)ori_piece[2] +
				d_d1[2]*(int)ori_piece[3] );

			ans_G[row_aft_index_base + aft_index_offset] = (unsigned char)temp0;
			ans_G[row_aft_index_base + aft_index_offset + 1] = (unsigned char)temp1;
			aft_sum += (int)temp0 + (int)temp1;
		}

		// Normalization.
		norm_factor =  ((float)ori_sum*2.0/3.0) / (float)aft_sum;
		for(int i=0; i<ww; ++i){
			temp0 = clamp ( (int) ((float)ans_G[row_aft_index_base +i] * norm_factor) );
			temp_G[row_aft_index_base + i] = ans_G[row_aft_index_base + i] = (unsigned char)temp0;
		}
		
		// BLUE
		// Initialize offset and sum of row.
		ori_sum = 0;
		aft_sum = 0;
		ori_index_offset = 0;
		aft_index_offset = 0;

		// Apply filter for each group of two pixels.
		ori_piece[3] = tex1Dfetch(TB, row_ori_index_base);
		#pragma unroll
		for(; aft_index_offset <= ww-2; ori_index_offset += 3, aft_index_offset += 2){

			ori_piece[0] = ori_piece[3];
			ori_piece[1] = tex1Dfetch(TB, row_ori_index_base + ori_index_offset);
			ori_piece[2] = tex1Dfetch(TB, row_ori_index_base + ori_index_offset +1);
			ori_piece[3] = tex1Dfetch(TB, row_ori_index_base + ori_index_offset +2);
			ori_sum += (int)ori_piece[1] + (int)ori_piece[2] + (int)ori_piece[3];

			temp0 = clamp (
				d_d0[0]*(int)ori_piece[0] +
				d_d0[1]*(int)ori_piece[1] +
				d_d0[2]*(int)ori_piece[2] );

			temp1 = clamp(
				d_d1[0]*(int)ori_piece[1] +
				d_d1[1]*(int)ori_piece[2] +
				d_d1[2]*(int)ori_piece[3] );

			ans_B[row_aft_index_base + aft_index_offset] = (unsigned char)temp0;
			ans_B[row_aft_index_base + aft_index_offset + 1] = (unsigned char)temp1;
			aft_sum += (int)temp0 + (int)temp1;
		}

		// Normalization.
		norm_factor =  ((float)ori_sum*2.0/3.0) / (float)aft_sum;
		for(int i=0; i<ww; ++i){
			temp0 = clamp ( (int) ((float)ans_B[row_aft_index_base +i] * norm_factor) );
			temp_B[row_aft_index_base + i] = ans_B[row_aft_index_base + i] = (unsigned char)temp0;
		}
	}
}

void SR_kernel_down(
	unsigned char *ori_R, unsigned char *ori_G, unsigned char *ori_B,
	unsigned char *aft_R, unsigned char *aft_G, unsigned char *aft_B,
	int w, int h){

	float d0[3]={0.227, 0.974, -0.022};
	float d1[3]={-0.022, 0.974, 0.227};

	unsigned char *R, *G, *B;
	unsigned char *ans_R, *ans_G, *ans_B;
	unsigned char *temp_R, *temp_G, *temp_B;
	int ww=w*2/3;
	int hh=h*2/3;

	hipMalloc((void**)&R, w*h*sizeof(unsigned char));
	hipMalloc((void**)&G, w*h*sizeof(unsigned char));
	hipMalloc((void**)&B, w*h*sizeof(unsigned char));
	hipMalloc((void**)&temp_R, ww*h*sizeof(unsigned char));
	hipMalloc((void**)&temp_G, ww*h*sizeof(unsigned char));
	hipMalloc((void**)&temp_B, ww*h*sizeof(unsigned char));
	hipMalloc((void**)&ans_R, w*h*sizeof(unsigned char)*2/3);
	hipMalloc((void**)&ans_G, w*h*sizeof(unsigned char)*2/3);
	hipMalloc((void**)&ans_B, w*h*sizeof(unsigned char)*2/3);
	
	hipMemcpy(R, ori_R, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(G, ori_G, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(B, ori_B, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);

	hipBindTexture(0, TR, R);
	hipBindTexture(0, TG, G);
	hipBindTexture(0, TB, B);
	set_filter(d0, d1);

	int threads=64;
	int blocks=64;
	/* i want each block do a row, and each thread in a block handle a pixel */
	for(int i=0; i<(h-1)/(threads*blocks) +1; ++i)
		run_row<<<blocks, threads>>>(i*threads*blocks, ans_R, ans_G, ans_B, w, h, ww, hh, temp_R, temp_G, temp_B);
	
	hipBindTexture(0, TansR, temp_R);
	hipBindTexture(0, TansG, temp_G);
	hipBindTexture(0, TansB, temp_B);

	for(int i=0; i<(ww-1)/(threads*blocks) +1; ++i)
		run_col<<<blocks, threads>>>(i*threads*blocks, ans_R, ans_G, ans_B, w, h, ww, hh);
	

	hipMemcpy(aft_R, ans_R, w*h*sizeof(unsigned char)*4/9, hipMemcpyDeviceToHost);
	hipMemcpy(aft_G, ans_G, w*h*sizeof(unsigned char)*4/9, hipMemcpyDeviceToHost);
	hipMemcpy(aft_B, ans_B, w*h*sizeof(unsigned char)*4/9, hipMemcpyDeviceToHost);

	hipUnbindTexture(TR);
	hipUnbindTexture(TG);
	hipUnbindTexture(TB);
	hipUnbindTexture(TansR);
	hipUnbindTexture(TansG);
	hipUnbindTexture(TansB);
	hipFree(R);
	hipFree(G);
	hipFree(B);
	hipFree(ans_R);
	hipFree(ans_G);
	hipFree(ans_B);
	hipFree(temp_R);
	hipFree(temp_G);
	hipFree(temp_B);
}
