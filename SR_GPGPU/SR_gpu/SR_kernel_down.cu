#include "hip/hip_runtime.h"

texture<unsigned char, 1, hipReadModeElementType> TR;
texture<unsigned char, 1, hipReadModeElementType> TG;
texture<unsigned char, 1, hipReadModeElementType> TB;
/*
texture<float, 1, hipReadModeElementType> Td0;
texture<float, 1, hipReadModeElementType> Td1;
*/
__constant__ float d_d0[3];
__constant__ float d_d1[3];

//extern __shared__ int row[];

extern "C" void set_filter(float *d0, float *d1){
	hipMemcpyToSymbol(HIP_SYMBOL(d_d0), d0, 3*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_d1), d1, 3*sizeof(float));
}

__device__ void setup_col(unsigned char *row0, unsigned char *row1, int ww, int h, int index, unsigned char *ans){
	int temp;

	temp=(int)(d_d0[1]*((int)ans[index])+d_d0[2]*((int)ans[ww +index]));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row0[0]=(unsigned char)temp;

	temp=(int)(d_d1[1]*((int)ans[index])+d_d1[2]*((int)ans[ww +index]));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row1[0]=(unsigned char)temp;

	temp=(int)(d_d0[0]*((int)ans[(h-2)*ww +index])+d_d0[1]*((int)ans[(h-1)*ww +index]));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row0[h-1]=(unsigned char)temp;

	temp=(int)(d_d1[0]*((int)ans[(h-2)*ww +index])+d_d1[1]*((int)ans[(h-1)*ww +index]));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row1[h-1]=(unsigned char)temp;

	#pragma unroll
	for(int i=1; i<h-1; ++i){
		temp=(int)(d_d0[0]*((int)ans[(i-1)*ww +index])+d_d0[1]*((int)ans[i*ww +index])+d_d0[2]*((int)ans[(i+1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[i]=(unsigned char)temp;

		temp=(int)(d_d1[0]*((int)ans[(i-1)*ww +index])+d_d1[1]*((int)ans[i*ww +index])+d_d1[2]*((int)ans[(i+1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[i]=(unsigned char)temp;
	}
}

__device__ void setup_row(unsigned char *row0, unsigned char *row1, int w, int index, texture<unsigned char, 1, hipReadModeElementType> rgb){
	int temp;
	temp=(int)(d_d0[1]*(int)tex1Dfetch(rgb, index*w +0)+d_d0[2]*(int)tex1Dfetch(rgb, index*w +1));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row0[0]=(unsigned char)temp;

	temp=(int)(d_d1[1]*(int)tex1Dfetch(rgb, index*w +0)+d_d1[2]*(int)tex1Dfetch(rgb, index*w +1));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row1[0]=(unsigned char)temp;

	temp=(int)(d_d0[0]*(int)tex1Dfetch(rgb, index*w +w-2)+d_d0[1]*(int)tex1Dfetch(rgb, index*w +w-1));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row0[w-1]=(unsigned char)temp;

	temp=(int)(d_d1[0]*(int)tex1Dfetch(rgb, index*w +w-2)+d_d1[1]*(int)tex1Dfetch(rgb, index*w +w-1));
	if(temp>255) temp=255;
	else if(temp<0) temp=0;
	row1[w-1]=(unsigned char)temp;

	#pragma unroll
	for(int i=1; i<w-1; ++i){
		temp=(int)(d_d0[0]*(int)tex1Dfetch(rgb, index*w +i-1)+d_d0[1]*(int)tex1Dfetch(rgb, index*w +i)+d_d0[2]*(int)tex1Dfetch(rgb, index*w +i+1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[i]=(unsigned char)temp;

		temp=(int)(d_d1[0]*(int)tex1Dfetch(rgb, index*w +i-1)+d_d1[1]*(int)tex1Dfetch(rgb, index*w +i)+d_d1[2]*(int)tex1Dfetch(rgb, index*w +i+1));	
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[i]=(unsigned char)temp;
	}
}

__global__ void run_col(int round, unsigned char *ans_R, unsigned char *ans_G, unsigned char *ans_B, int w, int h, int ww, int hh){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<ww){
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		int index=round+tid;
		for(int i=0; i<h; ++i){ // compute weight
			R_ori+=(int)ans_R[i*ww +index];
			G_ori+=(int)ans_G[i*ww +index];
			B_ori+=(int)ans_B[i*ww +index];
		}

		unsigned char row0[720];
		unsigned char row1[720];
		/////////////////////////////// red ////////////////////////////////////
		//setup_col(row0, row1, ww, h, index, ans_R);
		int temp;
		temp=(int)(d_d0[1]*((int)ans_R[index])+d_d0[2]*((int)ans_R[ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[0]=(unsigned char)temp;

		temp=(int)(d_d1[1]*((int)ans_R[index])+d_d1[2]*((int)ans_R[ww +index]));	
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[0]=(unsigned char)temp;

		temp=(int)(d_d0[0]*((int)ans_R[(h-2)*ww +index])+d_d0[1]*((int)ans_R[(h-1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[h-1]=(unsigned char)temp;

		temp=(int)(d_d1[0]*((int)ans_R[(h-2)*ww +index])+d_d1[1]*((int)ans_R[(h-1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[h-1]=(unsigned char)temp;

		#pragma unroll
		for(int i=1; i<h-1; ++i){
			temp=(int)(d_d0[0]*((int)ans_R[(i-1)*ww +index])+d_d0[1]*((int)ans_R[i*ww +index])+d_d0[2]*((int)ans_R[(i+1)*ww +index]));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row0[i]=(unsigned char)temp;
	
			temp=(int)(d_d1[0]*((int)ans_R[(i-1)*ww +index])+d_d1[1]*((int)ans_R[i*ww +index])+d_d1[2]*((int)ans_R[(i+1)*ww +index]));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row1[i]=(unsigned char)temp;
		}
		// setup_col() finish

		e_aft=0;
		for(int i=0; i<hh; ++i){
			if(i%2==0) ans_R[i*ww +index]=row0[3*i/2];
			else ans_R[i*ww +index]=row1[3*(i-1)/2 +2];
			e_aft+=ans_R[i*ww +index];
		}
		R_rate=(float)e_aft/((float)R_ori*2.0/3.0);
		if(R_rate<1.0){
			for(int i=0; i<hh; ++i){
				temp=(int)ans_R[i*ww +index];
				temp=(int)((float)temp/R_rate);
				if(temp>255) temp=255;
				else if(temp<0) temp=0;
				ans_R[i*ww +index]=(unsigned char)temp;
			}
		}
		else{
			for(int i=0; i<hh; ++i){
				temp=(int)ans_R[i*ww +index];
				temp=(int)((float)temp/R_rate);
				ans_R[i*ww +index]=(unsigned char)temp;
			}
		}
		////////////////////////// green ///////////////////////////
		//setup_col(row0, row1, ww, h, index, ans_G);
		temp=(int)(d_d0[1]*((int)ans_G[index])+d_d0[2]*((int)ans_G[ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[0]=(unsigned char)temp;

		temp=(int)(d_d1[1]*((int)ans_G[index])+d_d1[2]*((int)ans_G[ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[0]=(unsigned char)temp;

		temp=(int)(d_d0[0]*((int)ans_G[(h-2)*ww +index])+d_d0[1]*((int)ans_G[(h-1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[h-1]=(unsigned char)temp;
	
		temp=(int)(d_d1[0]*((int)ans_G[(h-2)*ww +index])+d_d1[1]*((int)ans_G[(h-1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[h-1]=(unsigned char)temp;

		#pragma unroll
		for(int i=1; i<h-1; ++i){
			temp=(int)(d_d0[0]*((int)ans_G[(i-1)*ww +index])+d_d0[1]*((int)ans_G[i*ww +index])+d_d0[2]*((int)ans_G[(i+1)*ww +index]));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row0[i]=(unsigned char)temp;

			temp=(int)(d_d1[0]*((int)ans_G[(i-1)*ww +index])+d_d1[1]*((int)ans_G[i*ww +index])+d_d1[2]*((int)ans_G[(i+1)*ww +index]));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row1[i]=(unsigned char)temp;
		}
		// setup_col() finish
		e_aft=0;
		for(int i=0; i<hh; ++i){
			if(i%2==0) ans_G[i*ww +index]=row0[3*i/2];
			else ans_G[i*ww +index]=row1[3*(i-1)/2 +2];
			e_aft+=ans_G[i*ww +index];
		}
		G_rate=(float)e_aft/((float)G_ori*2.0/3.0);
		if(G_rate<1.0){
			for(int i=0; i<hh; ++i){
				temp=(int)ans_G[i*ww +index];
				temp=(int)((float)temp/G_rate);
				if(temp>255) temp=255;
				else if(temp<0) temp=0;
				ans_G[i*ww +index]=(unsigned char)temp;
			}
		}
		else{
			for(int i=0; i<hh; ++i){
				temp=(int)ans_G[i*ww +index];
				temp=(int)((float)temp/G_rate);
				ans_G[i*ww +index]=(unsigned char)temp;
			}
		}
		///////////////////////////////// blue /////////////////////////////////
		//setup_col(row0, row1, ww, h, index, ans_B);
		temp=(int)(d_d0[1]*((int)ans_B[index])+d_d0[2]*((int)ans_B[ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[0]=(unsigned char)temp;

		temp=(int)(d_d1[1]*((int)ans_B[index])+d_d1[2]*((int)ans_B[ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[0]=(unsigned char)temp;

		temp=(int)(d_d0[0]*((int)ans_B[(h-2)*ww +index])+d_d0[1]*((int)ans_B[(h-1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[h-1]=(unsigned char)temp;

		temp=(int)(d_d1[0]*((int)ans_B[(h-2)*ww +index])+d_d1[1]*((int)ans_B[(h-1)*ww +index]));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[h-1]=(unsigned char)temp;

		#pragma unroll
		for(int i=1; i<h-1; ++i){
			temp=(int)(d_d0[0]*((int)ans_B[(i-1)*ww +index])+d_d0[1]*((int)ans_B[i*ww +index])+d_d0[2]*((int)ans_B[(i+1)*ww +index]));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row0[i]=(unsigned char)temp;
			
			temp=(int)(d_d1[0]*((int)ans_B[(i-1)*ww +index])+d_d1[1]*((int)ans_B[i*ww +index])+d_d1[2]*((int)ans_B[(i+1)*ww +index]));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row1[i]=(unsigned char)temp;
		}
		// setup_col() finish

		e_aft=0;
		for(int i=0; i<hh; ++i){
			if(i%2==0) ans_B[i*ww +index]=row0[3*i/2];
			else ans_B[i*ww +index]=row1[3*(i-1)/2 +2];
			e_aft+=ans_B[i*ww +index];
		}
		B_rate=(float)e_aft/((float)B_ori*2.0/3.0);
		if(B_rate<1.0){
			for(int i=0; i<hh; ++i){
				temp=(int)ans_B[i*ww +index];
				temp=(int)((float)temp/B_rate);
				if(temp>255) temp=255;
				else if(temp<0) temp=0;
				ans_B[i*ww +index]=(unsigned char)temp;
			}
		}
		else{
			for(int i=0; i<hh; ++i){
				temp=(int)ans_B[i*ww +index];
				temp=(int)((float)temp/B_rate);
				ans_B[i*ww +index]=(unsigned char)temp;
			}
		}
	}
}

__global__ void run_row(int round, unsigned char *ans_R, unsigned char *ans_G, unsigned char *ans_B, int w, int h, int ww, int hh){
	//int bid = blockIdx.x;
	//int tid = threadIdx.x;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<h){
		//__shared__ int weight[4]; // R_ori, G_ori, B_ori, e_aft
		//__shared__ float rate[3]; // R_rate, G_rate, B_rate
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		int index=(round+tid)*w;
		
		for(int i=0; i<w; ++i){ // compute weight
			R_ori+=(int)tex1Dfetch(TR, index +i);
			G_ori+=(int)tex1Dfetch(TG, index +i);
			B_ori+=(int)tex1Dfetch(TB, index +i);
		}

		unsigned char row0[1280];
		unsigned char row1[1280];
		index=round+tid;
		//////////////////////////////// red ////////////////////////////////////
		//setup_row(row0, row1, w, round+tid, TR);
		int temp;
		temp=(int)(d_d0[1]*(int)tex1Dfetch(TR, index*w +0)+d_d0[2]*(int)tex1Dfetch(TR, index*w +1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[0]=(unsigned char)temp;

		temp=(int)(d_d1[1]*(int)tex1Dfetch(TR, index*w +0)+d_d1[2]*(int)tex1Dfetch(TR, index*w +1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[0]=(unsigned char)temp;

		temp=(int)(d_d0[0]*(int)tex1Dfetch(TR, index*w +w-2)+d_d0[1]*(int)tex1Dfetch(TR, index*w +w-1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[w-1]=(unsigned char)temp;

		temp=(int)(d_d1[0]*(int)tex1Dfetch(TR, index*w +w-2)+d_d1[1]*(int)tex1Dfetch(TR, index*w +w-1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[w-1]=(unsigned char)temp;

		#pragma unroll
		for(int i=1; i<w-1; ++i){
			temp=(int)(d_d0[0]*(int)tex1Dfetch(TR, index*w +i-1)+d_d0[1]*(int)tex1Dfetch(TR, index*w +i)+d_d0[2]*(int)tex1Dfetch(TR, index*w +i+1));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row0[i]=(unsigned char)temp;
	
			temp=(int)(d_d1[0]*(int)tex1Dfetch(TR, index*w +i-1)+d_d1[1]*(int)tex1Dfetch(TR, index*w +i)+d_d1[2]*(int)tex1Dfetch(TR, index*w +i+1));	
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row1[i]=(unsigned char)temp;
		}
		// setup_row() finish


		e_aft=0;
		index=(round+tid)*ww;
		for(int i=0; i<ww; ++i){
			if(i%2==0) ans_R[index +i]=row0[3*i/2];
			else ans_R[index +i]=row1[3*(i-1)/2 +2];
			e_aft+=(int)ans_R[index +i];
		}
		R_rate=(float)e_aft/((float)R_ori*2.0/3.0);
		if(R_rate<1.0){
			for(int i=0; i<ww; ++i){
				temp=(int)ans_R[index +i];
				temp=(int)((float)temp/R_rate);
				if(temp>255) temp=255;
				else if(temp<0) temp=0;
				ans_R[index +i]=(unsigned char)temp;
			}
		}
		else{
			for(int i=0; i<ww; ++i){
				temp=(int)ans_R[index +i];
				temp=(int)((float)temp/R_rate);
				ans_R[index +i]=(unsigned char)temp;
			}
		}
		////////////////////// green //////////////////////////////
		//setup_row(row0, row1, w, round+tid, TG);
		index=round+tid;
		temp=(int)(d_d0[1]*(int)tex1Dfetch(TG, index*w +0)+d_d0[2]*(int)tex1Dfetch(TG, index*w +1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[0]=(unsigned char)temp;

		temp=(int)(d_d1[1]*(int)tex1Dfetch(TG, index*w +0)+d_d1[2]*(int)tex1Dfetch(TR, index*w +1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[0]=(unsigned char)temp;

		temp=(int)(d_d0[0]*(int)tex1Dfetch(TG, index*w +w-2)+d_d0[1]*(int)tex1Dfetch(TG, index*w +w-1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[w-1]=(unsigned char)temp;

		temp=(int)(d_d1[0]*(int)tex1Dfetch(TG, index*w +w-2)+d_d1[1]*(int)tex1Dfetch(TG, index*w +w-1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[w-1]=(unsigned char)temp;

		#pragma unroll
		for(int i=1; i<w-1; ++i){
			temp=(int)(d_d0[0]*(int)tex1Dfetch(TG, index*w +i-1)+d_d0[1]*(int)tex1Dfetch(TG, index*w +i)+d_d0[2]*(int)tex1Dfetch(TG, index*w +i+1));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row0[i]=(unsigned char)temp;
				
			temp=(int)(d_d1[0]*(int)tex1Dfetch(TG, index*w +i-1)+d_d1[1]*(int)tex1Dfetch(TG, index*w +i)+d_d1[2]*(int)tex1Dfetch(TG, index*w +i+1));	
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row1[i]=(unsigned char)temp;
		}
		// setup_row() finish

		e_aft=0;
		index=(round+tid)*w*2/3;
		for(int i=0; i<w*2/3; ++i){
			if(i%2==0) ans_G[index +i]=row0[3*i/2];
			else ans_G[index +i]=row1[3*(i-1)/2 +2];
			e_aft+=(int)ans_G[index +i];
		}
		G_rate=(float)e_aft/((float)G_ori*2.0/3.0);
		if(G_rate<1.0){
			for(int i=0; i<ww; ++i){
				temp=(int)ans_G[index +i];
				temp=(int)((float)temp/G_rate);
				if(temp>255) temp=255;
				else if(temp<0) temp=0;
				ans_G[index +i]=(unsigned char)temp;
			}
		}
		else{
			for(int i=0; i<ww; ++i){
				temp=(int)ans_G[index +i];
				temp=(int)((float)temp/G_rate);
				ans_G[index +i]=(unsigned char)temp;
			}
		}
		
		////////////////////////// blue ////////////////////////////
		
		//setup_row(row0, row1, w, round+tid, TB);
		index=round+tid;
		temp=(int)(d_d0[1]*(int)tex1Dfetch(TB, index*w +0)+d_d0[2]*(int)tex1Dfetch(TB, index*w +1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[0]=(unsigned char)temp;

		temp=(int)(d_d1[1]*(int)tex1Dfetch(TB, index*w +0)+d_d1[2]*(int)tex1Dfetch(TB, index*w +1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[0]=(unsigned char)temp;

		temp=(int)(d_d0[0]*(int)tex1Dfetch(TB, index*w +w-2)+d_d0[1]*(int)tex1Dfetch(TB, index*w +w-1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row0[w-1]=(unsigned char)temp;
	
		temp=(int)(d_d1[0]*(int)tex1Dfetch(TB, index*w +w-2)+d_d1[1]*(int)tex1Dfetch(TB, index*w +w-1));
		if(temp>255) temp=255;
		else if(temp<0) temp=0;
		row1[w-1]=(unsigned char)temp;
	
		#pragma unroll
		for(int i=1; i<w-1; ++i){
			temp=(int)(d_d0[0]*(int)tex1Dfetch(TB, index*w +i-1)+d_d0[1]*(int)tex1Dfetch(TB, index*w +i)+d_d0[2]*(int)tex1Dfetch(TB, index*w +i+1));
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row0[i]=(unsigned char)temp;
	
			temp=(int)(d_d1[0]*(int)tex1Dfetch(TB, index*w +i-1)+d_d1[1]*(int)tex1Dfetch(TB, index*w +i)+d_d1[2]*(int)tex1Dfetch(TB, index*w +i+1));	
			if(temp>255) temp=255;
			else if(temp<0) temp=0;
			row1[i]=(unsigned char)temp;
		}
		// setup_row() finish

		e_aft=0;
		index=(round+tid)*w*2/3;
		for(int i=0; i<w*2/3; ++i){
			if(i%2==0) ans_B[index +i]=row0[3*i/2];
			else ans_B[index +i]=row1[3*(i-1)/2 +2];
			e_aft+=ans_B[index +i];
		}
		B_rate=(float)e_aft/((float)B_ori*2.0/3.0);
		if(B_rate<1.0){
			for(int i=0; i<ww; ++i){
				temp=(int)ans_B[index +i];
				temp=(int)((float)temp/B_rate);
				if(temp>255) temp=255;
				else if(temp<0) temp=0;
				ans_B[index +i]=(unsigned char)temp;
			}
		}
		else{
			for(int i=0; i<ww; ++i){
				temp=(int)ans_B[index +i];
				temp=(int)((float)temp/B_rate);
				ans_B[index +i]=(unsigned char)temp;
			}
		}
		
	}
}

void SR_kernel_down(
	unsigned char *ori_R, unsigned char *ori_G, unsigned char *ori_B,
	unsigned char *aft_R, unsigned char *aft_G, unsigned char *aft_B,
	int w, int h){
	float d0[3]={-0.022, 0.974, 0.227};
	float d1[3]={0.227, 0.974, -0.022};

	unsigned char *R, *G, *B;
	unsigned char *ans_R, *ans_G, *ans_B;
	int ww=w*2/3;
	int hh=h*2/3;

	hipMalloc((void**)&R, w*h*sizeof(unsigned char));
	hipMalloc((void**)&G, w*h*sizeof(unsigned char));
	hipMalloc((void**)&B, w*h*sizeof(unsigned char));
	hipMalloc((void**)&ans_R, w*h*sizeof(unsigned char)*2/3);
	hipMalloc((void**)&ans_G, w*h*sizeof(unsigned char)*2/3);
	hipMalloc((void**)&ans_B, w*h*sizeof(unsigned char)*2/3);
	
	hipMemcpy(R, ori_R, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(G, ori_G, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(B, ori_B, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);

	hipBindTexture(0, TR, R);
	hipBindTexture(0, TG, G);
	hipBindTexture(0, TB, B);
	set_filter(d0, d1);

	int threads=64;
	int blocks=64;
	/* i want each block do a row, and each thread in a block handle a pixel */
	for(int i=0; i<(h-1)/(threads*blocks) +1; ++i)
		run_row<<<blocks, threads>>>(i*threads*blocks, ans_R, ans_G, ans_B, w, h, ww, hh);
	
	for(int i=0; i<(ww-1)/(threads*blocks) +1; ++i)
		run_col<<<blocks, threads>>>(i*threads*blocks, ans_R, ans_G, ans_B, w, h, ww, hh);
	

	hipMemcpy(aft_R, ans_R, w*h*sizeof(unsigned char)*4/9, hipMemcpyDeviceToHost);
	hipMemcpy(aft_G, ans_G, w*h*sizeof(unsigned char)*4/9, hipMemcpyDeviceToHost);
	hipMemcpy(aft_B, ans_B, w*h*sizeof(unsigned char)*4/9, hipMemcpyDeviceToHost);

	hipUnbindTexture(TR);
	hipUnbindTexture(TG);
	hipUnbindTexture(TB);
	hipFree(R);
	hipFree(G);
	hipFree(B);
	hipFree(ans_R);
	hipFree(ans_G);
	hipFree(ans_B);
}
