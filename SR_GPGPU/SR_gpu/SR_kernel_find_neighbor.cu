#include "hip/hip_runtime.h"

texture<unsigned char, 2, hipReadModeElementType> TIR; // ww*hh
texture<unsigned char, 2, hipReadModeElementType> TIG;
texture<unsigned char, 2, hipReadModeElementType> TIB;

texture<unsigned char, 2, hipReadModeElementType> TLR; // w*h
texture<unsigned char, 2, hipReadModeElementType> TLG;
texture<unsigned char, 2, hipReadModeElementType> TLB;

texture<unsigned char, 2, hipReadModeElementType> THR; // w*h
texture<unsigned char, 2, hipReadModeElementType> THG;
texture<unsigned char, 2, hipReadModeElementType> THB;

#include <stdio.h>
#include <time.h>

__device__ int calc_dist(
		int x, int y, int low_x, int low_y, int w, int ww, int min, int *pos,
		texture<unsigned char, 2, hipReadModeElementType> TI, texture<unsigned char, 2, hipReadModeElementType> TL){
	int dist=0;
	int dtex;

	for(int t=0; t<3; ++t){
		for(int k=0; k<3; ++k){
			dtex=(int)tex2D(TI, x+k, y+t)-(int)tex2D(TL, low_x+k, low_y+t);
			dist+=dtex*dtex;
		}
	}
	
	if(dist<min){
		pos[0]=low_x;
		pos[1]=low_y;
		return dist;
	}
	else return min;
}

__global__ void find_neighbor(
	int round,
	int w, int h, int ww, int hh, uchar4 *final_ans_ptr){
	char* final_ans = (char*)final_ans_ptr;
	//int tid=blockDim.x * blockIdx.x + threadIdx.x;
	//int tidy=blockDim.y * blockIdx.y + threadIdx.y;
	int tid = threadIdx.x;
	int bid = blockIdx.x;

	if( round+bid<(hh/3) && tid<ww ){
		__shared__ unsigned char map_I[1024*3];
		__shared__ unsigned char map_L[1024*5];
		
		int y=(round+bid)*3;
		map_I[tid]=tex2D(TIR, tid, y);
		map_I[tid +ww]=tex2D(TIR, tid, y+1);
		map_I[tid +2*ww]=tex2D(TIR, tid, y+2);

		if(tid<w){
			if(y==0){
				map_L[tid +w]=tex2D(TLR, tid, 0);
				map_L[tid +2*w]=tex2D(TLR, tid, 1);
				map_L[tid +3*w]=tex2D(TLR, tid, 2);
				map_L[tid +4*w]=tex2D(TLR, tid, 3);
			}
			else{
				map_L[tid]=tex2D(TLR, tid, y*2/3-1);
				map_L[tid +w]=tex2D(TLR, tid, y*2/3);
				map_L[tid +2*w]=tex2D(TLR, tid, y*2/3+1);
				map_L[tid +3*w]=tex2D(TLR, tid, y*2/3+2);
				map_L[tid +4*w]=tex2D(TLR, tid, y*2/3+3);
			}
		}
		__syncthreads();

		if(tid<ww/3){
			int x=(tid)*3;	
			int low_x=x*2/3;
			int low_y=y*2/3;
			int min_R=585526; // 255*255*9=585525
			//int min_G=585526;
			//int min_B=585526;
			int min_pos_R[2];
			//int min_pos_G[2];
			//int min_pos_B[2];
			int dtex;
			int dist;
		
			for(int j=0; j<=2; ++j){ // find neighbor in 3*3 block
				for(int i=0; i<=2; ++i){
					if( low_x+i-1>=0 && low_x+i-1<=w-3 && low_y+j-1>=0 && low_y+j-1<=h-3 ){
						dist=0;
						for(int t=0; t<3; ++t){
							for(int k=0; k<3; ++k){
								dtex=(int)map_I[t*ww +k]-(int)map_L[(j+t)*w +i+k];
								dist+=dtex*dtex;
							}
						}

						if(dist<min_R){
							min_pos_R[0]=low_x+i-1;
							min_pos_R[1]=low_y+j-1;
							min_R=dist;
						}

						//min_R=calc_dist(x, y, low_x+i, low_y+j, w, ww, min_R, min_pos_R, TIR, TLR);
						//min_G=calc_dist(x, y, low_x+i, low_y+j, w, ww, min_G, min_pos_G, TIG, TLG);
						//min_B=calc_dist(x, y, low_x+i, low_y+j, w, ww, min_B, min_pos_B, TIB, TLB);
					}
				}
			}
		
			int mmm, nnn;
			for(int j=0; j<3; ++j){
				for(int i=0; i<3; ++i){
					mmm=(int)tex2D(THR, min_pos_R[0]+i, min_pos_R[1]+j);
					nnn=(int)tex2D(TIR, x+i, y+j);
					mmm+=nnn;
					if(mmm>255) mmm=255;
					else if(mmm<0) mmm=0;
					final_ans[((y+j)*ww +x+i)*4]=(unsigned char)mmm;
						
					mmm=(int)tex2D(THG, min_pos_R[0]+i, min_pos_R[1]+j);
					nnn=(int)tex2D(TIG, x+i, y+j);
					mmm+=nnn;
					if(mmm>255) mmm=255;
					else if(mmm<0) mmm=0;
					final_ans[((y+j)*ww +x+i)*4 +1]=(unsigned char)mmm;

					mmm=(int)tex2D(THB, min_pos_R[0]+i, min_pos_R[1]+j);
					nnn=(int)tex2D(TIB, x+i, y+j);
					mmm+=nnn;
					if(mmm>255) mmm=255;
					else if(mmm<0) mmm=0;
					final_ans[((y+j)*ww +x+i)*4 +2]=(unsigned char)mmm;

					final_ans[((y+j)*ww +x+i)*4 +3]=(unsigned char)255;
				}
			}
		}
	}
	
	//__syncthreads();
}

void SR_kernel_find_neighbor(
	unsigned char *I_R, unsigned char *I_G, unsigned char *I_B,
	unsigned char *L_R, unsigned char *L_G, unsigned char *L_B,
	unsigned char *H_R, unsigned char *H_G, unsigned char *H_B,
	int w, int h, int ww, int hh, uchar4* tex){
	
	//int *d_IR, *d_IG, *d_IB; // img(up)
	//int *d_LR, *d_LG, *d_LB; // img(up(down))
	//int *d_HR, *d_HG, *d_HB; // img(original) - img(up(down))
	//unsigned char *d_ansR, *d_ansG, *d_ansB;
	//hipMalloc((void**)&d_ansR, ww*hh*sizeof(unsigned char));
	//hipMalloc((void**)&d_ansG, ww*hh*sizeof(unsigned char));
	//hipMalloc((void**)&d_ansB, ww*hh*sizeof(unsigned char));
	//hipMalloc((void**)&final_ans, ww*hh*4*sizeof(char));
	hipChannelFormatDesc Desc=hipCreateChannelDesc<unsigned char>();
	hipArray *d_IR, *d_IG, *d_IB, *d_LR, *d_LG, *d_LB, *d_HR, *d_HG, *d_HB;
	hipMallocArray(&d_IR, &Desc, ww, hh);
	hipMallocArray(&d_IG, &Desc, ww, hh);
	hipMallocArray(&d_IB, &Desc, ww, hh);
	hipMallocArray(&d_LR, &Desc, w, h);
	hipMallocArray(&d_LG, &Desc, w, h);
	hipMallocArray(&d_LB, &Desc, w, h);
	hipMallocArray(&d_HR, &Desc, w, h);
	hipMallocArray(&d_HG, &Desc, w, h);
	hipMallocArray(&d_HB, &Desc, w, h);
	
	hipMemcpyToArray(d_IR, 0, 0, I_R, ww*hh*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_IG, 0, 0, I_G, ww*hh*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_IB, 0, 0, I_B, ww*hh*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_LR, 0, 0, L_R, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_LG, 0, 0, L_G, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_LB, 0, 0, L_B, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_HR, 0, 0, H_R, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_HG, 0, 0, H_G, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_HB, 0, 0, H_B, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);


	hipBindTextureToArray(TIR, d_IR);
	hipBindTextureToArray(TIG, d_IG);
	hipBindTextureToArray(TIB, d_IB);
	hipBindTextureToArray(TLR, d_LR);
	hipBindTextureToArray(TLG, d_LG);
	hipBindTextureToArray(TLB, d_LB);
	hipBindTextureToArray(THR, d_HR);
	hipBindTextureToArray(THG, d_HG);
	hipBindTextureToArray(THB, d_HB);

	int threads=1024;
	int blocks=256;
	//for(int i=0; i<((ww/3)*(hh/3)-1)/(threads*blocks) +1; ++i){
	for(int i=0; i<((hh/3)-1)/(blocks) +1; ++i){
		find_neighbor<<<blocks, threads>>>(i*blocks, w, h, ww, hh, tex);
		
		//printf("error1: %s\n", hipGetErrorString(hipPeekAtLastError()));
		//printf("error2: %s\n", hipGetErrorString(hipDeviceSynchronize()));
	}

	

	//hipMemcpy(ans_R, d_ansR, ww*hh*sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(ans_G, d_ansG, ww*hh*sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(ans_B, d_ansB, ww*hh*sizeof(int), hipMemcpyDeviceToHost);

	hipUnbindTexture(TIR);
	hipUnbindTexture(TIG);
	hipUnbindTexture(TIB);
	hipUnbindTexture(TLR);
	hipUnbindTexture(TLG);
	hipUnbindTexture(TLB);
	hipUnbindTexture(THR);
	hipUnbindTexture(THG);
	hipUnbindTexture(THB);

	hipFreeArray(d_IR);
	hipFreeArray(d_IG);
	hipFreeArray(d_IB);
	hipFreeArray(d_LR);
	hipFreeArray(d_LG);
	hipFreeArray(d_LB);
	hipFreeArray(d_HR);
	hipFreeArray(d_HG);
	hipFreeArray(d_HB);
}
