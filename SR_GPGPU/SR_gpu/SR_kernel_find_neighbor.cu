#include "hip/hip_runtime.h"

texture<int, 2, hipReadModeElementType> TIR; // ww*hh
texture<int, 2, hipReadModeElementType> TIG;
texture<int, 2, hipReadModeElementType> TIB;

texture<int, 2, hipReadModeElementType> TLR; // w*h
texture<int, 2, hipReadModeElementType> TLG;
texture<int, 2, hipReadModeElementType> TLB;

texture<int, 2, hipReadModeElementType> THR; // w*h
texture<int, 2, hipReadModeElementType> THG;
texture<int, 2, hipReadModeElementType> THB;

#include <stdio.h>
#include <time.h>

__device__ int calc_dist(
		int x, int y, int low_x, int low_y, int w, int ww, int min, int *pos,
		texture<int, 2, hipReadModeElementType> TI, texture<int, 2, hipReadModeElementType> TL){
	int dist=0;
	
	for(int j=0; j<3; ++j){
		for(int i=0; i<3; ++i){
			dist+=( tex2D(TI, x+i, y+j)-tex2D(TL, low_x+i, low_y+j) )*( tex2D(TI, x+i, y+j)-tex2D(TL, low_x+i, low_y+j) );
		}
	}
	
	if(dist<min){
		pos[0]=low_x;
		pos[1]=low_y;
		return dist;
	}
	else return min;
}

__global__ void find_neighbor(int round, int *dans_R, int *dans_G, int *dans_B, int w, int h, int ww, int hh){
	int tid=blockDim.x * blockIdx.x + threadIdx.x;
	//int tidy=blockDim.y * blockIdx.y + threadIdx.y;
	
	if( round+tid<(ww/3)*(hh/3) ){
		int x=((round+tid)%(ww/3))*3;
		int y=((round+tid)/(ww/3))*3;
		
		int low_x=x*2/3;
		int low_y=y*2/3;
		int min_R=585526; // 255*255*9=585525
		int min_G=585526;
		int min_B=585526;
		int min_pos_R[2];
		int min_pos_G[2];
		int min_pos_B[2];
		
		for(int j=-1; j<=1; ++j){ // find neighbor in 3*3 block
			for(int i=-1; i<=1; ++i){
				if( low_x+i>=0 && low_x+i<=w-3 && low_y+j>=0 && low_y+j<=h-3 ){
					min_R=calc_dist(x, y, low_x+i, low_y+j, w, ww, min_R, min_pos_R, TIR, TLR);
					min_G=calc_dist(x, y, low_x+i, low_y+j, w, ww, min_G, min_pos_G, TIG, TLG);
					min_B=calc_dist(x, y, low_x+i, low_y+j, w, ww, min_B, min_pos_B, TIB, TLB);
				}
			}
		}
		
		for(int j=0; j<3; ++j){
			for(int i=0; i<3; ++i){
				dans_R[(y+j)*ww +x+i]=tex2D(THR, min_pos_R[0]+i, min_pos_R[1]+j);
				dans_G[(y+j)*ww +x+i]=tex2D(THG, min_pos_G[0]+i, min_pos_G[1]+j);
				dans_B[(y+j)*ww +x+i]=tex2D(THB, min_pos_B[0]+i, min_pos_B[1]+j);
			}
		}
	}
	
	__syncthreads();
}

void SR_kernel_find_neighbor(
	int *I_R, int *I_G, int *I_B,
	int *L_R, int *L_G, int *L_B,
	int *H_R, int *H_G, int *H_B,
	int *ans_R, int *ans_G, int *ans_B,
	int w, int h, int ww, int hh){
	
	//int *d_IR, *d_IG, *d_IB; // img(up)
	//int *d_LR, *d_LG, *d_LB; // img(up(down))
	//int *d_HR, *d_HG, *d_HB; // img(original) - img(up(down))
	int *d_ansR, *d_ansG, *d_ansB;
	hipMalloc((void**)&d_ansR, ww*hh*sizeof(int));
	hipMalloc((void**)&d_ansG, ww*hh*sizeof(int));
	hipMalloc((void**)&d_ansB, ww*hh*sizeof(int));

	hipChannelFormatDesc Desc=hipCreateChannelDesc<int>();

	hipArray *d_IR, *d_IG, *d_IB, *d_LR, *d_LG, *d_LB, *d_HR, *d_HG, *d_HB;
	hipMallocArray(&d_IR, &Desc, ww, hh);
	hipMallocArray(&d_IG, &Desc, ww, hh);
	hipMallocArray(&d_IB, &Desc, ww, hh);
	hipMallocArray(&d_LR, &Desc, w, h);
	hipMallocArray(&d_LG, &Desc, w, h);
	hipMallocArray(&d_LB, &Desc, w, h);
	hipMallocArray(&d_HR, &Desc, w, h);
	hipMallocArray(&d_HG, &Desc, w, h);
	hipMallocArray(&d_HB, &Desc, w, h);
	
	hipMemcpyToArray(d_IR, 0, 0, I_R, ww*hh*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_IG, 0, 0, I_G, ww*hh*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_IB, 0, 0, I_B, ww*hh*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_LR, 0, 0, L_R, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_LG, 0, 0, L_G, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_LB, 0, 0, L_B, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_HR, 0, 0, H_R, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_HG, 0, 0, H_G, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_HB, 0, 0, H_B, w*h*sizeof(int), hipMemcpyHostToDevice);

	hipBindTextureToArray(TIR, d_IR);
	hipBindTextureToArray(TIG, d_IG);
	hipBindTextureToArray(TIB, d_IB);
	hipBindTextureToArray(TLR, d_LR);
	hipBindTextureToArray(TLG, d_LG);
	hipBindTextureToArray(TLB, d_LB);
	hipBindTextureToArray(THR, d_HR);
	hipBindTextureToArray(THG, d_HG);
	hipBindTextureToArray(THB, d_HB);

	int threads=1024;
	int blocks=64;
	for(int i=0; i<((ww/3)*(hh/3)-1)/(threads*blocks) +1; ++i){
		find_neighbor<<<blocks, threads>>>(i*threads*blocks, d_ansR, d_ansG, d_ansB, w, h, ww, hh);
		
		//printf("error1: %s\n", hipGetErrorString(hipPeekAtLastError()));
		//printf("error2: %s\n", hipGetErrorString(hipDeviceSynchronize()));
	}

	hipMemcpy(ans_R, d_ansR, ww*hh*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(ans_G, d_ansG, ww*hh*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(ans_B, d_ansB, ww*hh*sizeof(int), hipMemcpyDeviceToHost);

	hipUnbindTexture(TIR);
	hipUnbindTexture(TIG);
	hipUnbindTexture(TIB);
	hipUnbindTexture(TLR);
	hipUnbindTexture(TLG);
	hipUnbindTexture(TLB);
	hipUnbindTexture(THR);
	hipUnbindTexture(THG);
	hipUnbindTexture(THB);

	hipFreeArray(d_IR);
	hipFreeArray(d_IG);
	hipFreeArray(d_IB);
	hipFreeArray(d_LR);
	hipFreeArray(d_LG);
	hipFreeArray(d_LB);
	hipFreeArray(d_HR);
	hipFreeArray(d_HG);
	hipFreeArray(d_HB);
	hipFree(d_ansR);
	hipFree(d_ansG);
	hipFree(d_ansB);
}
