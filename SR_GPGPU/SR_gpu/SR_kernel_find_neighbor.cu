#include "hip/hip_runtime.h"

texture<unsigned char, 2, hipReadModeElementType> TIR; // ww*hh
texture<unsigned char, 2, hipReadModeElementType> TIG;
texture<unsigned char, 2, hipReadModeElementType> TIB;

texture<unsigned char, 2, hipReadModeElementType> TLR; // w*h
texture<unsigned char, 2, hipReadModeElementType> TLG;
texture<unsigned char, 2, hipReadModeElementType> TLB;

texture<unsigned char, 2, hipReadModeElementType> THR; // w*h
texture<unsigned char, 2, hipReadModeElementType> THG;
texture<unsigned char, 2, hipReadModeElementType> THB;

#include <stdio.h>
#include <time.h>

__device__ int calc_dist(
		int x, int y, int low_x, int low_y, int w, int ww, int min, int *pos,
		texture<unsigned char, 2, hipReadModeElementType> TI, texture<unsigned char, 2, hipReadModeElementType> TL){
	int dist=0;
	int dtex;
	for(int j=0; j<3; ++j){
		for(int i=0; i<3; ++i){
			dtex=(int)tex2D(TI, x+i, y+j)-(int)tex2D(TL, low_x+i, low_y+j);
			dist+=dtex*dtex;
		}
	}
	
	if(dist<min){
		pos[0]=low_x;
		pos[1]=low_y;
		return dist;
	}
	else return min;
}

__global__ void find_neighbor(
	int round,
	int w, int h, int ww, int hh, uchar4 *final_ans_ptr){
	char* final_ans = (char*)final_ans_ptr;
	int tid=blockDim.x * blockIdx.x + threadIdx.x;
	//int tidy=blockDim.y * blockIdx.y + threadIdx.y;
	if( round+tid<(hh/3)*(ww/3) ){
		int x=((round+tid)%(ww/3))*3;
		int y=((round+tid)/(ww/3))*3;
		
		int low_x=x*2/3;
		int low_y=y*2/3;
		int min_R=585526; // 255*255*9=585525
		//int min_G=585526;
		//int min_B=585526;
		int min_pos_R[2];
		//int min_pos_G[2];
		//int min_pos_B[2];
		
		for(int j=-1; j<=1; ++j){ // find neighbor in 3*3 block
			for(int i=-1; i<=1; ++i){
				if( low_x+i>=0 && low_x+i<=w-3 && low_y+j>=0 && low_y+j<=h-3 ){
					min_R=calc_dist(x, y, low_x+i, low_y+j, w, ww, min_R, min_pos_R, TIR, TLR);
					//min_G=calc_dist(x, y, low_x+i, low_y+j, w, ww, min_G, min_pos_G, TIG, TLG);
					//min_B=calc_dist(x, y, low_x+i, low_y+j, w, ww, min_B, min_pos_B, TIB, TLB);
				}
			}
		}
		
		int mmm, nnn;
		for(int j=0; j<3; ++j){
			for(int i=0; i<3; ++i){
				mmm=(int)tex2D(THR, min_pos_R[0]+i, min_pos_R[1]+j);
				nnn=(int)tex2D(TIR, x+i, y+j);
				mmm+=nnn;
				if(mmm>255) mmm=255;
				else if(mmm<0) mmm=0;
				final_ans[((y+j)*ww +x+i)*4]=(unsigned char)mmm;

				mmm=(int)tex2D(THG, min_pos_R[0]+i, min_pos_R[1]+j);
				nnn=(int)tex2D(TIG, x+i, y+j);
				mmm+=nnn;
				if(mmm>255) mmm=255;
				else if(mmm<0) mmm=0;
				final_ans[((y+j)*ww +x+i)*4 +1]=(unsigned char)mmm;

				mmm=(int)tex2D(THB, min_pos_R[0]+i, min_pos_R[1]+j);
				nnn=(int)tex2D(TIB, x+i, y+j);
				mmm+=nnn;
				if(mmm>255) mmm=255;
				else if(mmm<0) mmm=0;
				final_ans[((y+j)*ww +x+i)*4 +2]=(unsigned char)mmm;
				/*
				dans_R[(y+j)*ww +x+i]=tex2D(THR, min_pos_R[0]+i, min_pos_R[1]+j)+tex2D(TIR, x+i, y+j);
				dans_G[(y+j)*ww +x+i]=tex2D(THG, min_pos_R[0]+i, min_pos_R[1]+j)+tex2D(TIG, x+i, y+j);
				dans_B[(y+j)*ww +x+i]=tex2D(THB, min_pos_R[0]+i, min_pos_R[1]+j)+tex2D(TIB, x+i, y+j);

				if( dans_R[(y+j)*ww +x+i]>255 ) dans_R[(y+j)*ww +x+i]=255;
				else if( dans_R[(y+j)*ww +x+i]<0 ) dans_R[(y+j)*ww +x+i]=0;
				if( dans_G[(y+j)*ww +x+i]>255 ) dans_G[(y+j)*ww +x+i]=255;
				else if( dans_G[(y+j)*ww +x+i]<0 ) dans_G[(y+j)*ww +x+i]=0;
				if( dans_B[(y+j)*ww +x+i]>255 ) dans_B[(y+j)*ww +x+i]=255;
				else if( dans_B[(y+j)*ww +x+i]<0 ) dans_B[(y+j)*ww +x+i]=0;

				final_ans[((y+j)*ww +x+i)*4 +0]=(unsigned char)dans_R[(y+j)*ww +x+i];
				final_ans[((y+j)*ww +x+i)*4 +1]=(unsigned char)dans_G[(y+j)*ww +x+i];
				final_ans[((y+j)*ww +x+i)*4 +2]=(unsigned char)dans_B[(y+j)*ww +x+i];
				*/
				final_ans[((y+j)*ww +x+i)*4 +3]=(unsigned char)255;
				
			}
		}
	}
	
	__syncthreads();
}

void SR_kernel_find_neighbor(
	unsigned char *I_R, unsigned char *I_G, unsigned char *I_B,
	unsigned char *L_R, unsigned char *L_G, unsigned char *L_B,
	unsigned char *H_R, unsigned char *H_G, unsigned char *H_B,
	int w, int h, int ww, int hh, uchar4* tex){
	
	//int *d_IR, *d_IG, *d_IB; // img(up)
	//int *d_LR, *d_LG, *d_LB; // img(up(down))
	//int *d_HR, *d_HG, *d_HB; // img(original) - img(up(down))
	//unsigned char *d_ansR, *d_ansG, *d_ansB;
	//hipMalloc((void**)&d_ansR, ww*hh*sizeof(unsigned char));
	//hipMalloc((void**)&d_ansG, ww*hh*sizeof(unsigned char));
	//hipMalloc((void**)&d_ansB, ww*hh*sizeof(unsigned char));
	//hipMalloc((void**)&final_ans, ww*hh*4*sizeof(char));
	hipChannelFormatDesc Desc=hipCreateChannelDesc<unsigned char>();
	hipArray *d_IR, *d_IG, *d_IB, *d_LR, *d_LG, *d_LB, *d_HR, *d_HG, *d_HB;
	/*
	hipChannelFormatDesc Desc2=hipCreateChannelDesc<unsigned char>();
	hipChannelFormatDesc Desc3=hipCreateChannelDesc<unsigned char>();
	hipChannelFormatDesc Desc4=hipCreateChannelDesc<unsigned char>();
	hipChannelFormatDesc Desc5=hipCreateChannelDesc<unsigned char>();
	hipChannelFormatDesc Desc6=hipCreateChannelDesc<unsigned char>();
	hipChannelFormatDesc Desc7=hipCreateChannelDesc<unsigned char>();
	hipChannelFormatDesc Desc8=hipCreateChannelDesc<unsigned char>();
	hipChannelFormatDesc Desc9=hipCreateChannelDesc<unsigned char>();
	*/
	hipMallocArray(&d_IR, &Desc, ww, hh);
	hipMallocArray(&d_IG, &Desc, ww, hh);
	hipMallocArray(&d_IB, &Desc, ww, hh);
	hipMallocArray(&d_LR, &Desc, w, h);
	hipMallocArray(&d_LG, &Desc, w, h);
	hipMallocArray(&d_LB, &Desc, w, h);
	hipMallocArray(&d_HR, &Desc, w, h);
	hipMallocArray(&d_HG, &Desc, w, h);
	hipMallocArray(&d_HB, &Desc, w, h);
	
	hipMemcpyToArray(d_IR, 0, 0, I_R, ww*hh*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_IG, 0, 0, I_G, ww*hh*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_IB, 0, 0, I_B, ww*hh*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_LR, 0, 0, L_R, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_LG, 0, 0, L_G, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_LB, 0, 0, L_B, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_HR, 0, 0, H_R, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_HG, 0, 0, H_G, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyToArray(d_HB, 0, 0, H_B, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);

	hipBindTextureToArray(TIR, d_IR);
	hipBindTextureToArray(TIG, d_IG);
	hipBindTextureToArray(TIB, d_IB);
	hipBindTextureToArray(TLR, d_LR);
	hipBindTextureToArray(TLG, d_LG);
	hipBindTextureToArray(TLB, d_LB);
	hipBindTextureToArray(THR, d_HR);
	hipBindTextureToArray(THG, d_HG);
	hipBindTextureToArray(THB, d_HB);

	int threads=300;
	int blocks=64;
	//for(int i=0; i<((ww/3)*(hh/3)-1)/(threads*blocks) +1; ++i){
	for(int i=0; i<((ww/3)*(hh/3)-1)/(threads*blocks) +1; ++i){
		find_neighbor<<<blocks, threads>>>(i*threads*blocks, w, h, ww, hh, tex);
		
		//printf("error1: %s\n", hipGetErrorString(hipPeekAtLastError()));
		//printf("error2: %s\n", hipGetErrorString(hipDeviceSynchronize()));
	}

	

	//hipMemcpy(ans_R, d_ansR, ww*hh*sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(ans_G, d_ansG, ww*hh*sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(ans_B, d_ansB, ww*hh*sizeof(int), hipMemcpyDeviceToHost);

	hipUnbindTexture(TIR);
	hipUnbindTexture(TIG);
	hipUnbindTexture(TIB);
	hipUnbindTexture(TLR);
	hipUnbindTexture(TLG);
	hipUnbindTexture(TLB);
	hipUnbindTexture(THR);
	hipUnbindTexture(THG);
	hipUnbindTexture(THB);

	hipFreeArray(d_IR);
	hipFreeArray(d_IG);
	hipFreeArray(d_IB);
	hipFreeArray(d_LR);
	hipFreeArray(d_LG);
	hipFreeArray(d_LB);
	hipFreeArray(d_HR);
	hipFreeArray(d_HG);
	hipFreeArray(d_HB);
}
