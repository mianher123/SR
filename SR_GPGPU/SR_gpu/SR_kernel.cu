#include "hip/hip_runtime.h"
/*
#pragma comment(lib, "cv.lib")
#pragma comment(lib, "highgui.lib")
#include <cv.h>
#include <highgui.h>
*/

texture<int, 1, hipReadModeElementType> TR;
texture<int, 1, hipReadModeElementType> TG;
texture<int, 1, hipReadModeElementType> TB;
texture<float, 1, hipReadModeElementType> Tu0;
texture<float, 1, hipReadModeElementType> Tu1;

__device__ int convolusion_col(int index, int ww, int hh, int *ans, int *row0, int *row1){
	int e_aft=0;
	int temp[2];
	for(int i=0; i<hh; ++i){
		if(i==0){
			temp[0]=(int)(tex1Dfetch(Tu0, 2)*row0[0]+tex1Dfetch(Tu0, 3)*row0[1]+tex1Dfetch(Tu0, 4)*row0[2]);
			temp[1]=(int)(tex1Dfetch(Tu1, 2)*row1[0]+tex1Dfetch(Tu1, 3)*row1[1]+tex1Dfetch(Tu1, 4)*row1[2]);
		}
		else if(i==1){
			temp[0]=(int)(tex1Dfetch(Tu0, 1)*row0[0]+tex1Dfetch(Tu0, 2)*row0[1]+tex1Dfetch(Tu0, 3)*row0[2]+tex1Dfetch(Tu0, 4)*row0[3]);
			temp[1]=(int)(tex1Dfetch(Tu1, 1)*row1[0]+tex1Dfetch(Tu1, 2)*row1[1]+tex1Dfetch(Tu1, 3)*row1[2]+tex1Dfetch(Tu1, 4)*row1[3]);
		}
		else if(i==hh-2){
			temp[0]=(int)(tex1Dfetch(Tu0, 0)*row0[hh-4]+tex1Dfetch(Tu0, 1)*row0[hh-3]+tex1Dfetch(Tu0, 2)*row0[hh-2]+tex1Dfetch(Tu0, 3)*row0[hh-1]);
			temp[1]=(int)(tex1Dfetch(Tu1, 0)*row1[hh-4]+tex1Dfetch(Tu1, 1)*row1[hh-3]+tex1Dfetch(Tu1, 2)*row1[hh-2]+tex1Dfetch(Tu1, 3)*row1[hh-1]);
		}
		else if(i==hh-1){
			temp[0]=(int)(tex1Dfetch(Tu0, 0)*row0[hh-3]+tex1Dfetch(Tu0, 1)*row0[hh-2]+tex1Dfetch(Tu0, 2)*row0[hh-1]);
			temp[1]=(int)(tex1Dfetch(Tu1, 0)*row1[hh-3]+tex1Dfetch(Tu1, 1)*row1[hh-2]+tex1Dfetch(Tu1, 2)*row1[hh-1]);
		}
		else{
			temp[0]=(int)(tex1Dfetch(Tu0, 0)*row0[i-2]+tex1Dfetch(Tu0, 1)*row0[i-1]+tex1Dfetch(Tu0, 2)*row0[i]+tex1Dfetch(Tu0, 3)*row0[i+1]+tex1Dfetch(Tu0, 4)*row0[i+2]);
			temp[1]=(int)(tex1Dfetch(Tu1, 0)*row1[i-2]+tex1Dfetch(Tu1, 1)*row1[i-1]+tex1Dfetch(Tu1, 2)*row1[i]+tex1Dfetch(Tu1, 3)*row1[i+1]+tex1Dfetch(Tu1, 4)*row1[i+2]);
		}
		ans[i*ww +index]=temp[0]+temp[1];
		e_aft+=(temp[0]+temp[1]);
	}
	return e_aft;
}

__device__ int convolusion_row(int index, int w, int ww, int *ans, int *row0, int *row1){
	int e_aft=0;
	int temp[2];
	for(int i=0; i<ww; ++i){
		if(i==0){
			temp[0]=(int)(tex1Dfetch(Tu0, 2)*row0[0]+tex1Dfetch(Tu0, 3)*row0[1]+tex1Dfetch(Tu0, 4)*row0[2]);
			temp[1]=(int)(tex1Dfetch(Tu1, 2)*row1[0]+tex1Dfetch(Tu1, 3)*row1[1]+tex1Dfetch(Tu1, 4)*row1[2]);
		}
		else if(i==1){
			temp[0]=(int)(tex1Dfetch(Tu0, 1)*row0[0]+tex1Dfetch(Tu0, 2)*row0[1]+tex1Dfetch(Tu0, 3)*row0[2]+tex1Dfetch(Tu0, 4)*row0[3]);
			temp[1]=(int)(tex1Dfetch(Tu1, 1)*row1[0]+tex1Dfetch(Tu1, 2)*row1[1]+tex1Dfetch(Tu1, 3)*row1[2]+tex1Dfetch(Tu1, 4)*row1[3]);
		}
		else if(i==ww-2){
			temp[0]=(int)(tex1Dfetch(Tu0, 0)*row0[ww-4]+tex1Dfetch(Tu0, 1)*row0[ww-3]+tex1Dfetch(Tu0, 2)*row0[ww-2]+tex1Dfetch(Tu0, 3)*row0[ww-1]);
			temp[1]=(int)(tex1Dfetch(Tu1, 0)*row1[ww-4]+tex1Dfetch(Tu1, 1)*row1[ww-3]+tex1Dfetch(Tu1, 2)*row1[ww-2]+tex1Dfetch(Tu1, 3)*row1[ww-1]);
		}
		else if(i==ww-1){
			temp[0]=(int)(tex1Dfetch(Tu0, 0)*row0[ww-3]+tex1Dfetch(Tu0, 1)*row0[ww-2]+tex1Dfetch(Tu0, 2)*row0[ww-1]);
			temp[1]=(int)(tex1Dfetch(Tu1, 0)*row1[ww-3]+tex1Dfetch(Tu1, 1)*row1[ww-2]+tex1Dfetch(Tu1, 2)*row1[ww-1]);
		}
		else{
			temp[0]=(int)(tex1Dfetch(Tu0, 0)*row0[i-2]+tex1Dfetch(Tu0, 1)*row0[i-1]+tex1Dfetch(Tu0, 2)*row0[i]+tex1Dfetch(Tu0, 3)*row0[i+1]+tex1Dfetch(Tu0, 4)*row0[i+2]);
			temp[1]=(int)(tex1Dfetch(Tu1, 0)*row1[i-2]+tex1Dfetch(Tu1, 1)*row1[i-1]+tex1Dfetch(Tu1, 2)*row1[i]+tex1Dfetch(Tu1, 3)*row1[i+1]+tex1Dfetch(Tu1, 4)*row1[i+2]);
		}
		ans[index*ww +i]=temp[0]+temp[1];
		e_aft+=(temp[0]+temp[1]);
	}
	return e_aft;
}

__global__ void run_cuda_col(int round, int *ans_R, int *ans_G, int *ans_B, int w, int h, int ww, int hh){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<ww){
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		for(int i=0; i<h; ++i){ // compute weight
			R_ori+=ans_R[i*ww +round+tid];
			G_ori+=ans_G[i*ww +round+tid];
			B_ori+=ans_B[i*ww +round+tid];
		}
		int row0[405];
		int row1[405];
		// red
		for(int i=0; i<405; ++i){
			if(i%3==0) row0[i]=ans_R[(i*2/3)*ww +round+tid];
			else row0[i]=0;

			if(i%3==2) row1[i]=ans_R[((i-2)*2/3+1)*ww +round+tid];
			else row1[i]=0;
		}
		e_aft=convolusion_col(round+tid, ww, hh, ans_R, row0, row1);
		R_rate=(float)e_aft/(float)(R_ori*3/2);
		// green
		for(int i=0; i<405; ++i){
			if(i%3==0) row0[i]=ans_G[(i*2/3)*ww +round+tid];
			else row0[i]=0;

			if(i%3==2) row1[i]=ans_G[((i-2)*2/3+1)*ww +round+tid];
			else row1[i]=0;
		}
		e_aft=convolusion_col(round+tid, ww, hh, ans_G, row0, row1);
		G_rate=(float)e_aft/(float)(G_ori*3/2);
		// blue
		for(int i=0; i<405; ++i){
			if(i%3==0) row0[i]=ans_B[(i*2/3)*ww +round+tid];
			else row0[i]=0;

			if(i%3==2) row1[i]=ans_B[((i-2)*2/3+1)*ww +round+tid];
			else row1[i]=0;
		}
		e_aft=convolusion_col(round+tid, ww, hh, ans_B, row0, row1);
		B_rate=(float)e_aft/(float)(B_ori*3/2);
		
		for(int i=0; i<405; ++i){
			ans_R[i*ww +round+tid]=(int)((float)ans_R[i*ww +round+tid]/R_rate);
			ans_G[i*ww +round+tid]=(int)((float)ans_G[i*ww +round+tid]/G_rate);
			ans_B[i*ww +round+tid]=(int)((float)ans_B[i*ww +round+tid]/B_rate);

			if(ans_R[i*ww +round+tid]>255) ans_R[i*ww +round+tid]=255;
			else if(ans_R[i*ww +round+tid]<0) ans_R[i*ww +round+tid]=0;

			if(ans_G[i*ww +round+tid]>255) ans_G[i*ww +round+tid]=255;
			else if(ans_G[i*ww +round+tid]<0) ans_G[i*ww +round+tid]=0;

			if(ans_B[i*ww +round+tid]>255) ans_B[i*ww +round+tid]=255;
			else if(ans_B[i*ww +round+tid]<0) ans_B[i*ww +round+tid]=0;
		}
		
	}
}

__global__ void run_cuda_row(int round, int *ans_R, int *ans_G, int *ans_B, int w, int h, int ww, int hh){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<h){
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		for(int i=0; i<w; ++i){ // compute weight
			/*
			ans_R[(round+tid)*w +i]=tex1Dfetch(TR, (round+tid)*w +i);
			ans_G[(round+tid)*w +i]=tex1Dfetch(TG, (round+tid)*w +i);
			ans_B[(round+tid)*w +i]=tex1Dfetch(TB, (round+tid)*w +i);
			*/
			
			R_ori+=tex1Dfetch(TR, (round+tid)*w +i);
			G_ori+=tex1Dfetch(TG, (round+tid)*w +i);
			B_ori+=tex1Dfetch(TB, (round+tid)*w +i);
			
		}
		int row0[540];
		int row1[540];
		// red
		for(int i=0; i<ww; ++i){ // setup row
			if(i%3==0) row0[i]=tex1Dfetch(TR, (round+tid)*w +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TR, (round+tid)*w +(i-2)*2/3+1);
			else row1[i]=0;
		}
		e_aft=convolusion_row(round+tid, w, ww, ans_R, row0, row1);
		R_rate=(float)e_aft/(float)(R_ori*3/2);

		// green
		for(int i=0; i<ww; ++i){ // setup row
			if(i%3==0) row0[i]=tex1Dfetch(TG, (round+tid)*w +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TG, (round+tid)*w +(i-2)*2/3+1);
			else row1[i]=0;
		}
		e_aft=convolusion_row(round+tid, w, ww, ans_G, row0, row1);
		G_rate=(float)e_aft/(float)(G_ori*3/2);

		// blue
		for(int i=0; i<ww; ++i){ // setup row
			if(i%3==0) row0[i]=tex1Dfetch(TB, (round+tid)*w +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TB, (round+tid)*w +(i-2)*2/3+1);
			else row1[i]=0;
		}
		e_aft=convolusion_row(round+tid, w, ww, ans_B, row0, row1);
		B_rate=(float)e_aft/(float)(B_ori*3/2);
		
		for(int i=0; i<ww; ++i){
			ans_R[(round+tid)*ww +i]=(int)((float)ans_R[(round+tid)*ww +i]/R_rate);
			ans_G[(round+tid)*ww +i]=(int)((float)ans_G[(round+tid)*ww +i]/G_rate);
			ans_B[(round+tid)*ww +i]=(int)((float)ans_B[(round+tid)*ww +i]/B_rate);
			
			if(ans_R[(round+tid)*ww +i]>255) ans_R[(round+tid)*ww +i]=255;
			else if(ans_R[(round+tid)*ww +i]<0) ans_R[(round+tid)*ww +i]=0;

			if(ans_G[(round+tid)*ww +i]>255) ans_G[(round+tid)*ww +i]=255;
			else if(ans_G[(round+tid)*ww +i]<0) ans_G[(round+tid)*ww +i]=0;

			if(ans_B[(round+tid)*ww +i]>255) ans_B[(round+tid)*ww +i]=255;
			else if(ans_B[(round+tid)*ww +i]<0) ans_B[(round+tid)*ww +i]=0;
		}
	}
}

void SR_kernel(int *ori_R, int *ori_G, int *ori_B, int *aft_R, int *aft_G, int *aft_B, int w, int h){
	float u1[5]={-0.1, 0.119, 0.927, 0.6, -0.047};
	float u0[5]={-0.047, 0.6, 0.927, 0.119, -0.1};
	
	int *R;
	int *G;
	int *B;
	int *ans_R;
	int *ans_G;
	int *ans_B;
	int *d_u0;
	int *d_u1;

	int ww=w*3/2;
	int hh=h*3/2;

	hipMalloc((void**)&R, w*h*sizeof(int));
	hipMalloc((void**)&G, w*h*sizeof(int));
	hipMalloc((void**)&B, w*h*sizeof(int));
	hipMalloc((void**)&ans_R, w*h*sizeof(int)*9/4);
	hipMalloc((void**)&ans_G, w*h*sizeof(int)*9/4);
	hipMalloc((void**)&ans_B, w*h*sizeof(int)*9/4);
	hipMalloc((void**)&d_u0, 5*sizeof(float));
	hipMalloc((void**)&d_u1, 5*sizeof(float));
	
	hipMemcpy(R, ori_R, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(G, ori_G, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B, ori_B, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_u0, u0, 5*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_u1, u1, 5*sizeof(float), hipMemcpyHostToDevice);

	hipBindTexture(0, TR, R);
	hipBindTexture(0, TG, G);
	hipBindTexture(0, TB, B);
	hipBindTexture(0, Tu0, d_u0);
	hipBindTexture(0, Tu1, d_u1);

	int threads=64;
	int blocks=64;
	
	for(int i=0; i<(h-1)/(threads*blocks) +1; ++i) // a thread do a row
		run_cuda_row<<<threads, blocks>>>(i*threads*blocks, ans_R, ans_G, ans_B, w, h, ww, hh);
	
	for(int i=0; i<(ww-1)/(threads*blocks) +1; ++i)
		run_cuda_col<<<threads, blocks>>>(i*threads*blocks, ans_R, ans_G, ans_B, w, h, ww, hh);
		
	hipMemcpy(aft_R, ans_R, w*h*sizeof(int)*9/4, hipMemcpyDeviceToHost);
	hipMemcpy(aft_G, ans_G, w*h*sizeof(int)*9/4, hipMemcpyDeviceToHost);
	hipMemcpy(aft_B, ans_B, w*h*sizeof(int)*9/4, hipMemcpyDeviceToHost);

	hipUnbindTexture(TR);
	hipUnbindTexture(TG);
	hipUnbindTexture(TB);
	hipFree(R);
	hipFree(G);
	hipFree(B);
}