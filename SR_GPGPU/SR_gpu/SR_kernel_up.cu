#include "hip/hip_runtime.h"
//#include "SR_kernel_start.cu"
#include <stdio.h>
#include <time.h>

texture<unsigned char, 1, hipReadModeElementType> TR;
texture<unsigned char, 1, hipReadModeElementType> TG;
texture<unsigned char, 1, hipReadModeElementType> TB;
texture<unsigned char ,1, hipReadModeElementType> TansR;
texture<unsigned char ,1, hipReadModeElementType> TansG;
texture<unsigned char ,1, hipReadModeElementType> TansB;

//__shared__ unsigned char share_mem[1024];

__constant__ float d_u0[5];
__constant__ float d_u1[5];

extern "C" void set_filter_up(float *u0, float *u1){
	hipMemcpyToSymbol(HIP_SYMBOL(d_u0), u0, 5*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_u1), u1, 5*sizeof(float));
}

__device__ unsigned char up_clamp(int value){
	if(value > 255) return (unsigned char)255;
	else if(value < 0) return (unsigned char)0;
	else return value;
}

__global__ void run_cuda_col(int round, unsigned char *ans_R, unsigned char *ans_G, unsigned char *ans_B, int w, int h, int ww, int hh, uchar4* tex_trivial){
	//int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int bid = blockIdx.x;

	if(round+bid<ww && tid<hh){
		//int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		//int e_aft;
		//float R_rate, G_rate, B_rate;
		int index=(round+bid)*h;
		int a_index=round+bid;
		unsigned char window1[5]={0, 0, 0, 0 ,0};
		unsigned char window2[5]={0, 0, 0, 0, 0};
		unsigned char* tex_trivial_p = (unsigned char*)tex_trivial;
		//unsigned char window1[6]={0, 0, 0, 0 ,0, 0};
		//unsigned char window2[6]={0, 0, 0, 0 ,0, 0};
		__shared__ unsigned char share_mem[1024];


		int temp[2];
		unsigned char sum;
		int bi;
		//int mod;

		if(tid<h)
			share_mem[tid]=tex1Dfetch(TansR, index +tid);
		__syncthreads();

		// red
		if(tid==0){
			window1[2]=share_mem[0];
			window2[4]=share_mem[1];
		}
		else if(tid==1){
			window1[1]=share_mem[0];
			window1[4]=share_mem[2];
			window2[3]=share_mem[1];
		}
		else if(tid%3==0){
			bi=tid*2/3;
			window1[2]=share_mem[bi];
			window2[1]=share_mem[bi-1];
			window2[4]=share_mem[bi+1];
		}
		else if(tid%3==1){
			bi=(tid-1)*2/3;
			window1[1]=share_mem[bi];
			window1[4]=share_mem[bi+2];
			window2[0]=share_mem[bi-1];
			window2[3]=share_mem[bi+1];
		}
		else{
			bi=(tid-2)*2/3;
			window1[0]=share_mem[bi];
			window1[3]=share_mem[bi+2];
			window2[2]=share_mem[bi+1];
		}

		temp[0]=(int)(d_u0[0]*(int)window1[0]+d_u0[1]*(int)window1[1]+d_u0[2]*(int)window1[2]+d_u0[3]*(int)window1[3]+d_u0[4]*(int)window1[4]);
		temp[1]=(int)(d_u1[0]*(int)window2[0]+d_u1[1]*(int)window2[1]+d_u1[2]*(int)window2[2]+d_u1[3]*(int)window2[3]+d_u1[4]*(int)window2[4]);
		sum=up_clamp(temp[0]+temp[1]);
		ans_R[tid*ww +a_index]=(unsigned char)sum;
		tex_trivial_p[(tid*ww + a_index)*4] = sum;

		//e_aft+=(int)sum;
		//R_rate=(float)e_aft/(float)(R_ori*3/2);

		// green
		window1[0]=window1[1]=window1[2]=window1[3]=window1[4]=0;
		window2[0]=window2[1]=window2[2]=window2[3]=window2[4]=0;
		
		if(tid<h)
			share_mem[tid]=tex1Dfetch(TansG, index +tid);
		__syncthreads();

		// green
		if(tid==0){
			window1[2]=share_mem[0];
			window2[4]=share_mem[1];
		}
		else if(tid==1){
			window1[1]=share_mem[0];
			window1[4]=share_mem[2];
			window2[3]=share_mem[1];
		}
		else if(tid%3==0){
			bi=tid*2/3;
			window1[2]=share_mem[bi];
			window2[1]=share_mem[bi-1];
			window2[4]=share_mem[bi+1];
		}
		else if(tid%3==1){
			bi=(tid-1)*2/3;
			window1[1]=share_mem[bi];
			window1[4]=share_mem[bi+2];
			window2[0]=share_mem[bi-1];
			window2[3]=share_mem[bi+1];
		}
		else{
			bi=(tid-2)*2/3;
			window1[0]=share_mem[bi];
			window1[3]=share_mem[bi+2];
			window2[2]=share_mem[bi+1];
		}
		temp[0]=(int)(d_u0[0]*(int)window1[0]+d_u0[1]*(int)window1[1]+d_u0[2]*(int)window1[2]+d_u0[3]*(int)window1[3]+d_u0[4]*(int)window1[4]);
		temp[1]=(int)(d_u1[0]*(int)window2[0]+d_u1[1]*(int)window2[1]+d_u1[2]*(int)window2[2]+d_u1[3]*(int)window2[3]+d_u1[4]*(int)window2[4]);
		sum=up_clamp(temp[0]+temp[1]);
		ans_G[tid*ww +a_index]=(unsigned char)sum;
		tex_trivial_p[(tid*ww + a_index)*4+1] = sum;
		//G_rate=(float)e_aft/(float)(G_ori*3/2);

		// blue
		window1[0]=window1[1]=window1[2]=window1[3]=window1[4]=0;
		window2[0]=window2[1]=window2[2]=window2[3]=window2[4]=0;
		
		if(tid<h)
			share_mem[tid]=tex1Dfetch(TansB, index +tid);
		__syncthreads();

		// blue
		if(tid==0){
			window1[2]=share_mem[0];
			window2[4]=share_mem[1];
		}
		else if(tid==1){
			window1[1]=share_mem[0];
			window1[4]=share_mem[2];
			window2[3]=share_mem[1];
		}
		else if(tid%3==0){
			bi=tid*2/3;
			window1[2]=share_mem[bi];
			window2[1]=share_mem[bi-1];
			window2[4]=share_mem[bi+1];
		}
		else if(tid%3==1){
			bi=(tid-1)*2/3;
			window1[1]=share_mem[bi];
			window1[4]=share_mem[bi+2];
			window2[0]=share_mem[bi-1];
			window2[3]=share_mem[bi+1];
		}
		else{
			bi=(tid-2)*2/3;
			window1[0]=share_mem[bi];
			window1[3]=share_mem[bi+2];
			window2[2]=share_mem[bi+1];
		}
		temp[0]=(int)(d_u0[0]*(int)window1[0]+d_u0[1]*(int)window1[1]+d_u0[2]*(int)window1[2]+d_u0[3]*(int)window1[3]+d_u0[4]*(int)window1[4]);
		temp[1]=(int)(d_u1[0]*(int)window2[0]+d_u1[1]*(int)window2[1]+d_u1[2]*(int)window2[2]+d_u1[3]*(int)window2[3]+d_u1[4]*(int)window2[4]);
		sum=up_clamp(temp[0]+temp[1]);
		ans_B[tid*ww +a_index]=(unsigned char)sum;
		tex_trivial_p[(tid*ww + a_index)*4+2] = sum;
		//B_rate=(float)e_aft/(float)(B_ori*3/2);
		

		tex_trivial_p[(tid*ww + a_index)*4+3] = 255;
		/*
		#pragma unroll
		for(int i=0; i<hh; ++i){
			temp[0]=(int)ans_R[i*ww +a_index];
			temp[0]=(int)((float)temp[0]/R_rate);
			if(temp[0]>255) temp[0]=255;
			else if(temp[0]<0) temp[0]=0;
			ans_R[i*ww +a_index]=(unsigned char)temp[0];

			temp[0]=(int)ans_G[i*ww +a_index];
			temp[0]=(int)((float)temp[0]/G_rate);
			if(temp[0]>255) temp[0]=255;
			else if(temp[0]<0) temp[0]=0;
			ans_G[i*ww +a_index]=(unsigned char)temp[0];

			temp[0]=(int)ans_B[i*ww +a_index];
			temp[0]=(int)((float)temp[0]/B_rate);
			if(temp[0]>255) temp[0]=255;
			else if(temp[0]<0) temp[0]=0;
			ans_B[i*ww +a_index]=(unsigned char)temp[0];
		}*/
	}
}

__global__ void run_cuda_row(
	int round,
	unsigned char *ans_R, unsigned char *ans_G, unsigned char *ans_B,
	int w, int h, int ww, int hh,
	unsigned char *temp_R, unsigned char *temp_G, unsigned char *temp_B){

	//int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int bid = blockIdx.x;

	__shared__ unsigned char share_mem[1024];

	if(round+bid<h && tid<ww){
		//__shared__ int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		//__shared__ int e_aft;
		//__shared__ float R_rate, G_rate, B_rate;
		int index=(round+bid)*w;
		int a_index=(round+bid)*ww;
		int bi;
		int temp[2];

		unsigned char sum;
		unsigned char window1[5]={0, 0, 0, 0 ,0};
		unsigned char window2[5]={0, 0, 0, 0, 0};
		//unsigned char window1[6]={0, 0, 0, 0 ,0, 0};
		//unsigned char window2[6]={0, 0, 0, 0 ,0, 0};

		if(tid<w)
			share_mem[tid]=tex1Dfetch(TR, index +tid);
		__syncthreads();

		// red

		//int ori_index1=2;
		//int ori_index2=1;
		//e_aft=0;
		//R_ori+=window1[3]=tex1Dfetch(TR, index);
		if(tid==0){
			window1[2]=share_mem[0];
			window2[4]=share_mem[1];
		}
		else if(tid==1){
			window1[1]=share_mem[0];
			window1[4]=share_mem[2];
			window2[3]=share_mem[1];
		}
		else if(tid%3==0){
			bi=tid*2/3;
			window1[2]=share_mem[bi];
			window2[1]=share_mem[bi-1];
			window2[4]=share_mem[bi+1];
		}
		else if(tid%3==1){
			bi=(tid-1)*2/3;
			window1[1]=share_mem[bi];
			window1[4]=share_mem[bi+2];
			window2[0]=share_mem[bi-1];
			window2[3]=share_mem[bi+1];
		}
		else{
			bi=(tid-2)*2/3;
			window1[0]=share_mem[bi];
			window1[3]=share_mem[bi+2];
			window2[2]=share_mem[bi+1];
		}
		/*
		if(tid==0){
			window1[2]=share_mem[0];
			window2[4]=share_mem[1];
		}
		else if(tid==1){
			window1[1]=share_mem[0];
			window1[4]=share_mem[2];
			window2[3]=share_mem[1];
		}
		else if(tid==2){
			window1[0]=share_mem[0];
			window1[3]=share_mem[2];
			window2[2]=share_mem[1];
		}
		else{
			mod=tid%3;
			bi=(tid-mod)*2/3;
			window1[2-mod]=share_mem[bi];
			window1[5-mod]=share_mem[bi+2];
			window2[(7-mod)%6]=share_mem[bi-1];
			window2[4-mod]=share_mem[bi+1];
		}*/
		temp[0]=(int)(d_u0[0]*(int)window1[0]+d_u0[1]*(int)window1[1]+d_u0[2]*(int)window1[2]+d_u0[3]*(int)window1[3]+d_u0[4]*(int)window1[4]);
		temp[1]=(int)(d_u1[0]*(int)window2[0]+d_u1[1]*(int)window2[1]+d_u1[2]*(int)window2[2]+d_u1[3]*(int)window2[3]+d_u1[4]*(int)window2[4]);
		sum=up_clamp(temp[0]+temp[1]);
		temp_R[tid*h +round+bid]=ans_R[a_index +tid]=(unsigned char)sum;
		//R_rate=(float)e_aft/(float)(R_ori*3/2);
		
		// green
		window1[0]=window1[1]=window1[2]=window1[3]=window1[4]=0;
		window2[0]=window2[1]=window2[2]=window2[3]=window2[4]=0;
		if(tid<w)
			share_mem[tid]=tex1Dfetch(TG, index +tid);
		__syncthreads();
		//ori_index1=2;
		//ori_index2=1;
		//e_aft=0;
		//G_ori+=window1[3]=tex1Dfetch(TG, index);
		
		if(tid==0){
			window1[2]=share_mem[0];
			window2[4]=share_mem[1];
		}
		else if(tid==1){
			window1[1]=share_mem[0];
			window1[4]=share_mem[2];
			window2[3]=share_mem[1];
		}
		else if(tid%3==0){
			bi=tid*2/3;
			window1[2]=share_mem[bi];
			window2[1]=share_mem[bi-1];
			window2[4]=share_mem[bi+1];
		}
		else if(tid%3==1){
			bi=(tid-1)*2/3;
			window1[1]=share_mem[bi];
			window1[4]=share_mem[bi+2];
			window2[0]=share_mem[bi-1];
			window2[3]=share_mem[bi+1];
		}
		else{
			bi=(tid-2)*2/3;
			window1[0]=share_mem[bi];
			window1[3]=share_mem[bi+2];
			window2[2]=share_mem[bi+1];
		}
		temp[0]=(int)(d_u0[0]*(int)window1[0]+d_u0[1]*(int)window1[1]+d_u0[2]*(int)window1[2]+d_u0[3]*(int)window1[3]+d_u0[4]*(int)window1[4]);
		temp[1]=(int)(d_u1[0]*(int)window2[0]+d_u1[1]*(int)window2[1]+d_u1[2]*(int)window2[2]+d_u1[3]*(int)window2[3]+d_u1[4]*(int)window2[4]);
		sum=up_clamp(temp[0]+temp[1]);
		temp_G[tid*h +round+bid]=ans_G[a_index +tid]=(unsigned char)sum;
		// convolution finish
		//G_rate=(float)e_aft/(float)(G_ori*3/2);

		// blue
		window1[0]=window1[1]=window1[2]=window1[3]=window1[4]=0;
		window2[0]=window2[1]=window2[2]=window2[3]=window2[4]=0;
		if(tid<w)
			share_mem[tid]=tex1Dfetch(TB, index +tid);
		__syncthreads();
		//ori_index1=2;
		//ori_index2=1;
		//e_aft=0;
		//B_ori+=window1[3]=tex1Dfetch(TB, index);
		
		if(tid==0){
			window1[2]=share_mem[0];
			window2[4]=share_mem[1];
		}
		else if(tid==1){
			window1[1]=share_mem[0];
			window1[4]=share_mem[2];
			window2[3]=share_mem[1];
		}
		else if(tid%3==0){
			bi=tid*2/3;
			window1[2]=share_mem[bi];
			window2[1]=share_mem[bi-1];
			window2[4]=share_mem[bi+1];
		}
		else if(tid%3==1){
			bi=(tid-1)*2/3;
			window1[1]=share_mem[bi];
			window1[4]=share_mem[bi+2];
			window2[0]=share_mem[bi-1];
			window2[3]=share_mem[bi+1];
		}
		else{
			bi=(tid-2)*2/3;
			window1[0]=share_mem[bi];
			window1[3]=share_mem[bi+2];
			window2[2]=share_mem[bi+1];
		}
		temp[0]=(int)(d_u0[0]*(int)window1[0]+d_u0[1]*(int)window1[1]+d_u0[2]*(int)window1[2]+d_u0[3]*(int)window1[3]+d_u0[4]*(int)window1[4]);
		temp[1]=(int)(d_u1[0]*(int)window2[0]+d_u1[1]*(int)window2[1]+d_u1[2]*(int)window2[2]+d_u1[3]*(int)window2[3]+d_u1[4]*(int)window2[4]);
		sum=up_clamp(temp[0]+temp[1]);
		temp_B[tid*h +round+bid]=ans_B[a_index +tid]=(unsigned char)sum;
		// convolution finish
		//B_rate=(float)e_aft/(float)(B_ori*3/2);
		
		/*
		#pragma unroll
		for(int i=0; i<ww; ++i){
			temp[0]=(int)ans_R[a_index +i];
			temp[0]=(int)((float)temp[0]/R_rate);
			if(temp[0]>255) temp[0]=255;
			else if(temp[0]<0) temp[0]=0;
			temp_R[i*h +round+tid]=ans_R[a_index +i]=(unsigned char)temp[0];

			temp[0]=(int)ans_G[a_index +i];
			temp[0]=(int)((float)temp[0]/G_rate);
			if(temp[0]>255) temp[0]=255;
			else if(temp[0]<0) temp[0]=0;
			temp_G[i*h +round+tid]=ans_G[a_index +i]=(unsigned char)temp[0];

			temp[0]=(int)ans_B[a_index +i];
			temp[0]=(int)((float)temp[0]/B_rate);
			if(temp[0]>255) temp[0]=255;
			else if(temp[0]<0) temp[0]=0;
			temp_B[i*h +round+tid]=ans_B[a_index +i]=(unsigned char)temp[0];
		}*/
	}

	//__syncthreads();
}

void SR_kernel_up(
	unsigned char *ori_R, unsigned char *ori_G, unsigned char *ori_B,
	unsigned char *aft_R, unsigned char *aft_G, unsigned char *aft_B,
	int w, int h, int ww, int hh, uchar4* tex_trivial){
	float u0[5]={-0.047, 0.6, 0.927, 0.119, -0.1};
	float u1[5]={-0.1, 0.119, 0.927, 0.6, -0.047};
	
	unsigned char *d_ori_R, *d_ori_G, *d_ori_B;
	unsigned char *d_ans_R, *d_ans_G, *d_ans_B;
	unsigned char *temp_R, *temp_G, *temp_B;

	//printf("in up, w=%d, h=%d, ww=%d, hh=%d\n", w, h, ww, hh);
	hipMalloc((void**)&d_ori_R, w*h*sizeof(unsigned char));
	hipMalloc((void**)&d_ori_G, w*h*sizeof(unsigned char));
	hipMalloc((void**)&d_ori_B, w*h*sizeof(unsigned char));
	hipMalloc((void**)&temp_R, ww*h*sizeof(unsigned char));
	hipMalloc((void**)&temp_G, ww*h*sizeof(unsigned char));
	hipMalloc((void**)&temp_B, ww*h*sizeof(unsigned char));
	hipMalloc((void**)&d_ans_R, ww*hh*sizeof(unsigned char));
	hipMalloc((void**)&d_ans_G, ww*hh*sizeof(unsigned char));
	hipMalloc((void**)&d_ans_B, ww*hh*sizeof(unsigned char));
	
	hipMemcpy(d_ori_R, ori_R, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(d_ori_G, ori_G, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(d_ori_B, ori_B, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);

	hipBindTexture(0, TR, d_ori_R);
	hipBindTexture(0, TG, d_ori_G);
	hipBindTexture(0, TB, d_ori_B);
	set_filter_up(u0, u1);

	int threads=750;
	int blocks=256;
	for(int i=0; i<(h-1)/(blocks) +1; ++i) // a thread do a row
		run_cuda_row<<<blocks, threads>>>(i*blocks, d_ans_R, d_ans_G, d_ans_B, w, h, ww, hh, temp_R, temp_G, temp_B);
		//run_cuda_row<<<blocks, threads, threads*sizeof(int)*ww*2>>>(i*threads*blocks, d_ans_R, d_ans_G, d_ans_B, w, h, ww, hh, temp_R, temp_G, temp_B);
	
	hipBindTexture(0, TansR, temp_R);
	hipBindTexture(0, TansG, temp_G);
	hipBindTexture(0, TansB, temp_B);

	for(int i=0; i<(ww-1)/(blocks) +1; ++i) // a thread do a column
		run_cuda_col<<<blocks, threads>>>(i*blocks, d_ans_R, d_ans_G, d_ans_B, w, h, ww, hh, tex_trivial);
	
	hipMemcpy(aft_R, d_ans_R, ww*hh*sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipMemcpy(aft_G, d_ans_G, ww*hh*sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipMemcpy(aft_B, d_ans_B, ww*hh*sizeof(unsigned char), hipMemcpyDeviceToHost);

	
	hipUnbindTexture(TR);
	hipUnbindTexture(TG);
	hipUnbindTexture(TB);
	hipUnbindTexture(TansR);
	hipUnbindTexture(TansG);
	hipUnbindTexture(TansB);
	
	hipFree(d_ori_R);
	hipFree(d_ori_G);
	hipFree(d_ori_B);
	hipFree(d_ans_R);
	hipFree(d_ans_G);
	hipFree(d_ans_B);
	hipFree(temp_R);
	hipFree(temp_G);
	hipFree(temp_B);
}
