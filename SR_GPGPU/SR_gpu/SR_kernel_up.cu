#include "hip/hip_runtime.h"
//#include "SR_kernel_start.cu"
extern int *test;
texture<int, 1, hipReadModeElementType> TR;
texture<int, 1, hipReadModeElementType> TG;
texture<int, 1, hipReadModeElementType> TB;
texture<float, 1, hipReadModeElementType> Tu0;
texture<float, 1, hipReadModeElementType> Tu1;

__device__ int convolusion_col(int index, int ww, int hh, int *ans, int *row0, int *row1){
	int e_aft=0;
	int temp[2];
	for(int i=0; i<hh; ++i){
		if(i==0){
			temp[0]=(int)(tex1Dfetch(Tu0, 2)*row0[0]+tex1Dfetch(Tu0, 3)*row0[1]+tex1Dfetch(Tu0, 4)*row0[2]);
			temp[1]=(int)(tex1Dfetch(Tu1, 2)*row1[0]+tex1Dfetch(Tu1, 3)*row1[1]+tex1Dfetch(Tu1, 4)*row1[2]);
		}
		else if(i==1){
			temp[0]=(int)(tex1Dfetch(Tu0, 1)*row0[0]+tex1Dfetch(Tu0, 2)*row0[1]+tex1Dfetch(Tu0, 3)*row0[2]+tex1Dfetch(Tu0, 4)*row0[3]);
			temp[1]=(int)(tex1Dfetch(Tu1, 1)*row1[0]+tex1Dfetch(Tu1, 2)*row1[1]+tex1Dfetch(Tu1, 3)*row1[2]+tex1Dfetch(Tu1, 4)*row1[3]);
		}
		else if(i==hh-2){
			temp[0]=(int)(tex1Dfetch(Tu0, 0)*row0[hh-4]+tex1Dfetch(Tu0, 1)*row0[hh-3]+tex1Dfetch(Tu0, 2)*row0[hh-2]+tex1Dfetch(Tu0, 3)*row0[hh-1]);
			temp[1]=(int)(tex1Dfetch(Tu1, 0)*row1[hh-4]+tex1Dfetch(Tu1, 1)*row1[hh-3]+tex1Dfetch(Tu1, 2)*row1[hh-2]+tex1Dfetch(Tu1, 3)*row1[hh-1]);
		}
		else if(i==hh-1){
			temp[0]=(int)(tex1Dfetch(Tu0, 0)*row0[hh-3]+tex1Dfetch(Tu0, 1)*row0[hh-2]+tex1Dfetch(Tu0, 2)*row0[hh-1]);
			temp[1]=(int)(tex1Dfetch(Tu1, 0)*row1[hh-3]+tex1Dfetch(Tu1, 1)*row1[hh-2]+tex1Dfetch(Tu1, 2)*row1[hh-1]);
		}
		else{
			temp[0]=(int)(tex1Dfetch(Tu0, 0)*row0[i-2]+tex1Dfetch(Tu0, 1)*row0[i-1]+tex1Dfetch(Tu0, 2)*row0[i]+tex1Dfetch(Tu0, 3)*row0[i+1]+tex1Dfetch(Tu0, 4)*row0[i+2]);
			temp[1]=(int)(tex1Dfetch(Tu1, 0)*row1[i-2]+tex1Dfetch(Tu1, 1)*row1[i-1]+tex1Dfetch(Tu1, 2)*row1[i]+tex1Dfetch(Tu1, 3)*row1[i+1]+tex1Dfetch(Tu1, 4)*row1[i+2]);
		}
		ans[i*ww +index]=temp[0]+temp[1];
		e_aft+=(temp[0]+temp[1]);
	}
	return e_aft;
}

__device__ int convolusion_row(int index, int w, int ww, int *ans, int *row0, int *row1){
	int e_aft=0;
	int temp[2];
	for(int i=0; i<ww; ++i){
		if(i==0){
			temp[0]=(int)(tex1Dfetch(Tu0, 2)*row0[0]+tex1Dfetch(Tu0, 3)*row0[1]+tex1Dfetch(Tu0, 4)*row0[2]);
			temp[1]=(int)(tex1Dfetch(Tu1, 2)*row1[0]+tex1Dfetch(Tu1, 3)*row1[1]+tex1Dfetch(Tu1, 4)*row1[2]);
		}
		else if(i==1){
			temp[0]=(int)(tex1Dfetch(Tu0, 1)*row0[0]+tex1Dfetch(Tu0, 2)*row0[1]+tex1Dfetch(Tu0, 3)*row0[2]+tex1Dfetch(Tu0, 4)*row0[3]);
			temp[1]=(int)(tex1Dfetch(Tu1, 1)*row1[0]+tex1Dfetch(Tu1, 2)*row1[1]+tex1Dfetch(Tu1, 3)*row1[2]+tex1Dfetch(Tu1, 4)*row1[3]);
		}
		else if(i==ww-2){
			temp[0]=(int)(tex1Dfetch(Tu0, 0)*row0[ww-4]+tex1Dfetch(Tu0, 1)*row0[ww-3]+tex1Dfetch(Tu0, 2)*row0[ww-2]+tex1Dfetch(Tu0, 3)*row0[ww-1]);
			temp[1]=(int)(tex1Dfetch(Tu1, 0)*row1[ww-4]+tex1Dfetch(Tu1, 1)*row1[ww-3]+tex1Dfetch(Tu1, 2)*row1[ww-2]+tex1Dfetch(Tu1, 3)*row1[ww-1]);
		}
		else if(i==ww-1){
			temp[0]=(int)(tex1Dfetch(Tu0, 0)*row0[ww-3]+tex1Dfetch(Tu0, 1)*row0[ww-2]+tex1Dfetch(Tu0, 2)*row0[ww-1]);
			temp[1]=(int)(tex1Dfetch(Tu1, 0)*row1[ww-3]+tex1Dfetch(Tu1, 1)*row1[ww-2]+tex1Dfetch(Tu1, 2)*row1[ww-1]);
		}
		else{
			temp[0]=(int)(tex1Dfetch(Tu0, 0)*row0[i-2]+tex1Dfetch(Tu0, 1)*row0[i-1]+tex1Dfetch(Tu0, 2)*row0[i]+tex1Dfetch(Tu0, 3)*row0[i+1]+tex1Dfetch(Tu0, 4)*row0[i+2]);
			temp[1]=(int)(tex1Dfetch(Tu1, 0)*row1[i-2]+tex1Dfetch(Tu1, 1)*row1[i-1]+tex1Dfetch(Tu1, 2)*row1[i]+tex1Dfetch(Tu1, 3)*row1[i+1]+tex1Dfetch(Tu1, 4)*row1[i+2]);
		}
		ans[index*ww +i]=temp[0]+temp[1];
		e_aft+=(temp[0]+temp[1]);
	}
	return e_aft;
}

__global__ void run_cuda_col(int round, int *ans_R, int *ans_G, int *ans_B, int w, int h, int ww, int hh){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<ww){
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		for(int i=0; i<h; ++i){ // compute weight
			R_ori+=ans_R[i*ww +round+tid];
			G_ori+=ans_G[i*ww +round+tid];
			B_ori+=ans_B[i*ww +round+tid];
		}
		int row0[405];
		int row1[405];
		// red
		for(int i=0; i<405; ++i){
			if(i%3==0) row0[i]=ans_R[(i*2/3)*ww +round+tid];
			else row0[i]=0;

			if(i%3==2) row1[i]=ans_R[((i-2)*2/3+1)*ww +round+tid];
			else row1[i]=0;
		}
		e_aft=convolusion_col(round+tid, ww, hh, ans_R, row0, row1);
		R_rate=(float)e_aft/(float)(R_ori*3/2);
		// green
		for(int i=0; i<405; ++i){
			if(i%3==0) row0[i]=ans_G[(i*2/3)*ww +round+tid];
			else row0[i]=0;

			if(i%3==2) row1[i]=ans_G[((i-2)*2/3+1)*ww +round+tid];
			else row1[i]=0;
		}
		e_aft=convolusion_col(round+tid, ww, hh, ans_G, row0, row1);
		G_rate=(float)e_aft/(float)(G_ori*3/2);
		// blue
		for(int i=0; i<405; ++i){
			if(i%3==0) row0[i]=ans_B[(i*2/3)*ww +round+tid];
			else row0[i]=0;

			if(i%3==2) row1[i]=ans_B[((i-2)*2/3+1)*ww +round+tid];
			else row1[i]=0;
		}
		e_aft=convolusion_col(round+tid, ww, hh, ans_B, row0, row1);
		B_rate=(float)e_aft/(float)(B_ori*3/2);
		
		for(int i=0; i<405; ++i){
			ans_R[i*ww +round+tid]=(int)((float)ans_R[i*ww +round+tid]/R_rate);
			ans_G[i*ww +round+tid]=(int)((float)ans_G[i*ww +round+tid]/G_rate);
			ans_B[i*ww +round+tid]=(int)((float)ans_B[i*ww +round+tid]/B_rate);

			if(ans_R[i*ww +round+tid]>255) ans_R[i*ww +round+tid]=255;
			else if(ans_R[i*ww +round+tid]<0) ans_R[i*ww +round+tid]=0;

			if(ans_G[i*ww +round+tid]>255) ans_G[i*ww +round+tid]=255;
			else if(ans_G[i*ww +round+tid]<0) ans_G[i*ww +round+tid]=0;

			if(ans_B[i*ww +round+tid]>255) ans_B[i*ww +round+tid]=255;
			else if(ans_B[i*ww +round+tid]<0) ans_B[i*ww +round+tid]=0;
		}
		
	}
}

__global__ void run_cuda_row(int round, int *ans_R, int *ans_G, int *ans_B, int w, int h, int ww, int hh){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<h){
		//test[0]=1139;
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		for(int i=0; i<w; ++i){ // compute weight
			R_ori+=tex1Dfetch(TR, (round+tid)*w +i);
			G_ori+=tex1Dfetch(TG, (round+tid)*w +i);
			B_ori+=tex1Dfetch(TB, (round+tid)*w +i);
		}
		int row0[540];
		int row1[540];
		// red
		for(int i=0; i<ww; ++i){ // setup row
			if(i%3==0) row0[i]=tex1Dfetch(TR, (round+tid)*w +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TR, (round+tid)*w +(i-2)*2/3+1);
			else row1[i]=0;
		}
		e_aft=convolusion_row(round+tid, w, ww, ans_R, row0, row1);
		R_rate=(float)e_aft/(float)(R_ori*3/2);

		// green
		for(int i=0; i<ww; ++i){ // setup row
			if(i%3==0) row0[i]=tex1Dfetch(TG, (round+tid)*w +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TG, (round+tid)*w +(i-2)*2/3+1);
			else row1[i]=0;
		}
		e_aft=convolusion_row(round+tid, w, ww, ans_G, row0, row1);
		G_rate=(float)e_aft/(float)(G_ori*3/2);

		// blue
		for(int i=0; i<ww; ++i){ // setup row
			if(i%3==0) row0[i]=tex1Dfetch(TB, (round+tid)*w +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TB, (round+tid)*w +(i-2)*2/3+1);
			else row1[i]=0;
		}
		e_aft=convolusion_row(round+tid, w, ww, ans_B, row0, row1);
		B_rate=(float)e_aft/(float)(B_ori*3/2);
		
		for(int i=0; i<ww; ++i){
			ans_R[(round+tid)*ww +i]=(int)((float)ans_R[(round+tid)*ww +i]/R_rate);
			ans_G[(round+tid)*ww +i]=(int)((float)ans_G[(round+tid)*ww +i]/G_rate);
			ans_B[(round+tid)*ww +i]=(int)((float)ans_B[(round+tid)*ww +i]/B_rate);
			
			if(ans_R[(round+tid)*ww +i]>255) ans_R[(round+tid)*ww +i]=255;
			else if(ans_R[(round+tid)*ww +i]<0) ans_R[(round+tid)*ww +i]=0;

			if(ans_G[(round+tid)*ww +i]>255) ans_G[(round+tid)*ww +i]=255;
			else if(ans_G[(round+tid)*ww +i]<0) ans_G[(round+tid)*ww +i]=0;

			if(ans_B[(round+tid)*ww +i]>255) ans_B[(round+tid)*ww +i]=255;
			else if(ans_B[(round+tid)*ww +i]<0) ans_B[(round+tid)*ww +i]=0;
		}
	}
}

void SR_kernel_up(int *ori_R, int *ori_G, int *ori_B, int *aft_R, int *aft_G, int *aft_B, int w, int h){
	float u0[5]={-0.047, 0.6, 0.927, 0.119, -0.1};
	float u1[5]={-0.1, 0.119, 0.927, 0.6, -0.047};
	
	int *d_ori_R, *d_ori_G, *d_ori_B;
	int *d_ans_R, *d_ans_G, *d_ans_B;
	int *d_u0, *d_u1;

	int ww=w*3/2;
	int hh=h*3/2;
	
	hipMalloc((void**)&d_ori_R, w*h*sizeof(int));
	hipMalloc((void**)&d_ori_G, w*h*sizeof(int));
	hipMalloc((void**)&d_ori_B, w*h*sizeof(int));
	hipMalloc((void**)&d_ans_R, ww*hh*sizeof(int));
	hipMalloc((void**)&d_ans_G, ww*hh*sizeof(int));
	hipMalloc((void**)&d_ans_B, ww*hh*sizeof(int));
	
	hipMalloc((void**)&d_u0, 5*sizeof(float));
	hipMalloc((void**)&d_u1, 5*sizeof(float));
	
	hipMemcpy(d_ori_R, ori_R, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ori_G, ori_G, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ori_B, ori_B, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_u0, u0, 5*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_u1, u1, 5*sizeof(float), hipMemcpyHostToDevice);

	hipBindTexture(0, TR, d_ori_R);
	hipBindTexture(0, TG, d_ori_G);
	hipBindTexture(0, TB, d_ori_B);
	hipBindTexture(0, Tu0, d_u0);
	hipBindTexture(0, Tu1, d_u1);

	int threads=64;
	int blocks=64;
	for(int i=0; i<(h-1)/(threads*blocks) +1; ++i) // a thread do a row
		run_cuda_row<<<threads, blocks>>>(i*threads*blocks, d_ans_R, d_ans_G, d_ans_B, w, h, ww, hh);
	
	for(int i=0; i<(ww-1)/(threads*blocks) +1; ++i) // a thread do a column
		run_cuda_col<<<threads, blocks>>>(i*threads*blocks, d_ans_R, d_ans_G, d_ans_B, w, h, ww, hh);
	
		
	hipMemcpy(aft_R, d_ans_R, ww*hh*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(aft_G, d_ans_G, ww*hh*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(aft_B, d_ans_B, ww*hh*sizeof(int), hipMemcpyDeviceToHost);

	hipUnbindTexture(TR);
	hipUnbindTexture(TG);
	hipUnbindTexture(TB);
	
	hipFree(d_ori_R);
	hipFree(d_ori_G);
	hipFree(d_ori_B);
	hipFree(d_ans_R);
	hipFree(d_ans_G);
	hipFree(d_ans_B);
	
	hipFree(d_u0);
	hipFree(d_u1);
}