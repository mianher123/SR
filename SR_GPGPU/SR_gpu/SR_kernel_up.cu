#include "hip/hip_runtime.h"
//#include "SR_kernel_start.cu"
#include <stdio.h>
#include <time.h>

texture<int, 1, hipReadModeElementType> TR;
texture<int, 1, hipReadModeElementType> TG;
texture<int, 1, hipReadModeElementType> TB;
texture<int ,1, hipReadModeElementType> TansR;
texture<int ,1, hipReadModeElementType> TansG;
texture<int ,1, hipReadModeElementType> TansB;

//extern __shared__ int row[];
__constant__ float d_u0[5];
__constant__ float d_u1[5];

extern "C" void set_filter_up(float *u0, float *u1){
	hipMemcpyToSymbol(HIP_SYMBOL(d_u0), u0, 5 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_u1), u1, 5 * sizeof(float));
}

__device__ int convolusion_col(int index, int ww, int hh, int *ans, int *row0, int *row1){
	int e_aft=0;
	int temp[2];
	// i==0
	temp[0]=(int)(d_u0[2]*row0[0]+d_u0[3]*row0[1]+d_u0[4]*row0[2]);
	temp[1]=(int)(d_u1[2]*row1[0]+d_u1[3]*row1[1]+d_u1[4]*row1[2]);
	ans[index]=temp[0]+temp[1];
	e_aft+=(temp[0]+temp[1]);
	// i==1
	temp[0]=(int)(d_u0[1]*row0[0]+d_u0[2]*row0[1]+d_u0[3]*row0[2]+d_u0[4]*row0[3]);
	temp[1]=(int)(d_u1[1]*row1[0]+d_u1[2]*row1[1]+d_u1[3]*row1[2]+d_u1[4]*row1[3]);
	ans[ww +index]=temp[0]+temp[1];
	e_aft+=(temp[0]+temp[1]);
	// i==hh-2
	temp[0]=(int)(d_u0[0]*row0[hh-4]+d_u0[1]*row0[hh-3]+d_u0[2]*row0[hh-2]+d_u0[3]*row0[hh-1]);
	temp[1]=(int)(d_u1[0]*row1[hh-4]+d_u1[1]*row1[hh-3]+d_u1[2]*row1[hh-2]+d_u1[3]*row1[hh-1]);
	ans[(hh-2)*ww +index]=temp[0]+temp[1];
	e_aft+=(temp[0]+temp[1]);
	// i==hh-1
	temp[0]=(int)(d_u0[0]*row0[hh-3]+d_u0[1]*row0[hh-2]+d_u0[2]*row0[hh-1]);
	temp[1]=(int)(d_u1[0]*row1[hh-3]+d_u1[1]*row1[hh-2]+d_u1[2]*row1[hh-1]);
	ans[(hh-1)*ww +index]=temp[0]+temp[1];
	e_aft+=(temp[0]+temp[1]);
	//#pragma unroll
	for(int i=2; i<hh-2; ++i){
		temp[0]=(int)(d_u0[0]*row0[i-2]+d_u0[1]*row0[i-1]+d_u0[2]*row0[i]+d_u0[3]*row0[i+1]+d_u0[4]*row0[i+2]);
		temp[1]=(int)(d_u1[0]*row1[i-2]+d_u1[1]*row1[i-1]+d_u1[2]*row1[i]+d_u1[3]*row1[i+1]+d_u1[4]*row1[i+2]);
		
		ans[i*ww +index]=temp[0]+temp[1];
		e_aft+=(temp[0]+temp[1]);
	}
	return e_aft;
}

__device__ int convolusion_row(int a_index, int w, int ww, int *ans, int index, int *row0, int *row1){
	int e_aft=0;
	int temp[2];
	
	// i==0
	temp[0]=(int)(d_u0[2]*row0[0]+d_u0[3]*row0[1]+d_u0[4]*row0[2]);
	temp[1]=(int)(d_u1[2]*row1[0]+d_u1[3]*row1[1]+d_u1[4]*row1[2]);
	ans[a_index*ww]=temp[0]+temp[1];
	e_aft+=(temp[0]+temp[1]);
	// i==1
	temp[0]=(int)(d_u0[1]*row0[0]+d_u0[2]*row0[1]+d_u0[3]*row0[2]+d_u0[4]*row0[3]);
	temp[1]=(int)(d_u1[1]*row1[0]+d_u1[2]*row1[1]+d_u1[3]*row1[2]+d_u1[4]*row1[3]);
	ans[a_index*ww +1]=temp[0]+temp[1];
	e_aft+=(temp[0]+temp[1]);
	// i==ww-2
	temp[0]=(int)(d_u0[0]*row0[ww-4]+d_u0[1]*row0[ww-3]+d_u0[2]*row0[ww-2]+d_u0[3]*row0[ww-1]);
	temp[1]=(int)(d_u1[0]*row1[ww-4]+d_u1[1]*row1[ww-3]+d_u1[2]*row1[ww-2]+d_u1[3]*row1[ww-1]);
	ans[a_index*ww +ww-2]=temp[0]+temp[1];
	e_aft+=(temp[0]+temp[1]);
	// i==ww-1
	temp[0]=(int)(d_u0[0]*row0[ww-3]+d_u0[1]*row0[ww-2]+d_u0[2]*row0[ww-1]);
	temp[1]=(int)(d_u1[0]*row1[ww-3]+d_u1[1]*row1[ww-2]+d_u1[2]*row1[ww-1]);
	ans[a_index*ww +ww-1]=temp[0]+temp[1];
	e_aft+=(temp[0]+temp[1]);
	
	//#pragma unroll
	for(int i=2; i<ww-2; ++i){
		temp[0]=(int)(d_u0[0]*row0[i-2]+d_u0[1]*row0[i-1]+d_u0[2]*row0[i]+d_u0[3]*row0[i+1]+d_u0[4]*row0[i+2]);
		temp[1]=(int)(d_u1[0]*row1[i-2]+d_u1[1]*row1[i-1]+d_u1[2]*row1[i]+d_u1[3]*row1[i+1]+d_u1[4]*row1[i+2]);
		ans[a_index*ww +i]=temp[0]+temp[1];
		e_aft+=(temp[0]+temp[1]);
	}
	return e_aft;
}

__global__ void run_cuda_col(int round, int *ans_R, int *ans_G, int *ans_B, int w, int h, int ww, int hh){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<ww){
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		int index=(round+tid)*h;
		//#pragma unroll
		for(int i=0; i<h; ++i){ // compute weight
			R_ori+=tex1Dfetch(TansR, index +i);
			G_ori+=tex1Dfetch(TansG, index +i);
			B_ori+=tex1Dfetch(TansB, index +i);
			/*
			R_ori+=ans_R[i*ww +index];
			G_ori+=ans_G[i*ww +index];
			B_ori+=ans_B[i*ww +index];
			*/
		}
		int row0[1080];
		int row1[1080];
		// red
		//#pragma unroll
		for(int i=0; i<hh; ++i){
			
			if(i%3==0) row0[i]=tex1Dfetch(TansR, index +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TansR, index +(i-2)*2/3+1);
			else row1[i]=0;
			
			/*
			if(i%3==0) row0[i]=ans_R[(i*2/3)*ww +index];
			else row0[i]=0;

			if(i%3==2) row1[i]=ans_R[((i-2)*2/3+1)*ww +index];
			else row1[i]=0;
			*/
		}
		e_aft=convolusion_col(round+tid, ww, hh, ans_R, row0, row1);
		R_rate=(float)e_aft/(float)(R_ori*3/2);
		// green
		//#pragma unroll
		for(int i=0; i<hh; ++i){
			
			if(i%3==0) row0[i]=tex1Dfetch(TansG, index +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TansG, index +(i-2)*2/3+1);
			else row1[i]=0;
			
			/*
			if(i%3==0) row0[i]=ans_G[(i*2/3)*ww +index];
			else row0[i]=0;

			if(i%3==2) row1[i]=ans_G[((i-2)*2/3+1)*ww +index];
			else row1[i]=0;
			*/
		}
		e_aft=convolusion_col(round+tid, ww, hh, ans_G, row0, row1);
		G_rate=(float)e_aft/(float)(G_ori*3/2);
		// blue
		//#pragma unroll
		for(int i=0; i<hh; ++i){
			
			if(i%3==0) row0[i]=tex1Dfetch(TansB, index +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TansB, index +(i-2)*2/3+1);
			else row1[i]=0;
			
			/*
			if(i%3==0) row0[i]=ans_B[(i*2/3)*ww +index];
			else row0[i]=0;

			if(i%3==2) row1[i]=ans_B[((i-2)*2/3+1)*ww +index];
			else row1[i]=0;
			*/
		}
		e_aft=convolusion_col(round+tid, ww, hh, ans_B, row0, row1);
		B_rate=(float)e_aft/(float)(B_ori*3/2);
		
		index=round+tid;
		//#pragma unroll
		for(int i=0; i<hh; ++i){
			ans_R[i*ww +index]=(int)((float)ans_R[i*ww +index]/R_rate);
			ans_G[i*ww +index]=(int)((float)ans_G[i*ww +index]/G_rate);
			ans_B[i*ww +index]=(int)((float)ans_B[i*ww +index]/B_rate);
			/*
			if(ans_R[i*ww +round+tid]>255) ans_R[i*ww +round+tid]=255;
			else if(ans_R[i*ww +round+tid]<0) ans_R[i*ww +round+tid]=0;

			if(ans_G[i*ww +round+tid]>255) ans_G[i*ww +round+tid]=255;
			else if(ans_G[i*ww +round+tid]<0) ans_G[i*ww +round+tid]=0;

			if(ans_B[i*ww +round+tid]>255) ans_B[i*ww +round+tid]=255;
			else if(ans_B[i*ww +round+tid]<0) ans_B[i*ww +round+tid]=0;
			*/
		}
		
	}
}

__global__ void run_cuda_row(int round, int *ans_R, int *ans_G, int *ans_B, int w, int h, int ww, int hh, int *temp_R, int *temp_G, int *temp_B){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	//__shared__ int row[540*2*8];
	if(round+tid<h){
		//test[0]=1139;
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		int index=(round+tid)*w;
		//#pragma unroll
		for(int i=0; i<w; ++i){ // compute weight
			R_ori+=tex1Dfetch(TR, index +i);
			G_ori+=tex1Dfetch(TG, index +i);
			B_ori+=tex1Dfetch(TB, index +i);
		}
		
		int row0[1920];
		int row1[1920];
		
		// red
		//#pragma unroll
		for(int i=0; i<ww; ++i){ // setup row
			/*
			if(i%3==0) row[threadIdx.x*ww*2 +i]=tex1Dfetch(TR, index +i*2/3);
			else row[threadIdx.x*ww*2 +i]=0;

			if(i%3==2) row[threadIdx.x*ww*2 +ww+i]=tex1Dfetch(TR, index +(i-2)*2/3+1);
			else row[threadIdx.x*ww*2 +ww+i]=0;
			*/
			if(i%3==0) row0[i]=tex1Dfetch(TR, index +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TR, index +(i-2)*2/3+1);
			else row1[i]=0;
			
		}
		e_aft=convolusion_row(round+tid, w, ww, ans_R, threadIdx.x*ww*2, row0, row1);
		R_rate=(float)e_aft/(float)(R_ori*3/2);

		// green
		//#pragma unroll
		for(int i=0; i<ww; ++i){ // setup row
			/*
			if(i%3==0) row[threadIdx.x*ww*2 +i]=tex1Dfetch(TG, index +i*2/3);
			else row[threadIdx.x*ww*2 +i]=0;

			if(i%3==2) row[threadIdx.x*ww*2 +ww+i]=tex1Dfetch(TG, index +(i-2)*2/3+1);
			else row[threadIdx.x*ww*2 +ww+i]=0;
			*/
			
			if(i%3==0) row0[i]=tex1Dfetch(TG, index +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TG, index +(i-2)*2/3+1);
			else row1[i]=0;
			
		}
		e_aft=convolusion_row(round+tid, w, ww, ans_G, threadIdx.x*ww*2, row0, row1);
		G_rate=(float)e_aft/(float)(G_ori*3/2);

		// blue
		//#pragma unroll
		for(int i=0; i<ww; ++i){ // setup row
			/*
			if(i%3==0) row[threadIdx.x*ww*2 +i]=tex1Dfetch(TB, index +i*2/3);
			else row[threadIdx.x*ww*2 +i]=0;

			if(i%3==2) row[threadIdx.x*ww*2 +ww+i]=tex1Dfetch(TB, index +(i-2)*2/3+1);
			else row[threadIdx.x*ww*2 +ww+i]=0;
			*/
			
			if(i%3==0) row0[i]=tex1Dfetch(TB, index +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TB, index +(i-2)*2/3+1);
			else row1[i]=0;
			
		}
		e_aft=convolusion_row(round+tid, w, ww, ans_B, threadIdx.x*ww*2, row0, row1);
		B_rate=(float)e_aft/(float)(B_ori*3/2);
		
		index=(round+tid)*ww;
		//#pragma unroll
		for(int i=0; i<ww; ++i){
			temp_R[i*h +round+tid]=ans_R[index +i]=(int)((float)ans_R[index +i]/R_rate);
			temp_G[i*h +round+tid]=ans_G[index +i]=(int)((float)ans_G[index +i]/G_rate);
			temp_B[i*h +round+tid]=ans_B[index +i]=(int)((float)ans_B[index +i]/B_rate);

			/*
			if(ans_R[(round+tid)*ww +i]>255) ans_R[(round+tid)*ww +i]=255;
			else if(ans_R[(round+tid)*ww +i]<0) ans_R[(round+tid)*ww +i]=0;

			if(ans_G[(round+tid)*ww +i]>255) ans_G[(round+tid)*ww +i]=255;
			else if(ans_G[(round+tid)*ww +i]<0) ans_G[(round+tid)*ww +i]=0;

			if(ans_B[(round+tid)*ww +i]>255) ans_B[(round+tid)*ww +i]=255;
			else if(ans_B[(round+tid)*ww +i]<0) ans_B[(round+tid)*ww +i]=0;
			*/
		}
	}

	__syncthreads();
}

void SR_kernel_up(int *ori_R, int *ori_G, int *ori_B, int *aft_R, int *aft_G, int *aft_B, int w, int h, int ww, int hh){
	float u0[5]={-0.047, 0.6, 0.927, 0.119, -0.1};
	float u1[5]={-0.1, 0.119, 0.927, 0.6, -0.047};
	
	int *d_ori_R, *d_ori_G, *d_ori_B;
	int *d_ans_R, *d_ans_G, *d_ans_B;
	int *temp_R, *temp_G, *temp_B;

	//printf("in up, w=%d, h=%d, ww=%d, hh=%d\n", w, h, ww, hh);
	hipMalloc((void**)&d_ori_R, w*h*sizeof(int));
	hipMalloc((void**)&d_ori_G, w*h*sizeof(int));
	hipMalloc((void**)&d_ori_B, w*h*sizeof(int));
	hipMalloc((void**)&temp_R, ww*h*sizeof(int));
	hipMalloc((void**)&temp_G, ww*h*sizeof(int));
	hipMalloc((void**)&temp_B, ww*h*sizeof(int));
	hipMalloc((void**)&d_ans_R, ww*hh*sizeof(int));
	hipMalloc((void**)&d_ans_G, ww*hh*sizeof(int));
	hipMalloc((void**)&d_ans_B, ww*hh*sizeof(int));
	
	hipMemcpy(d_ori_R, ori_R, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ori_G, ori_G, w*h*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ori_B, ori_B, w*h*sizeof(int), hipMemcpyHostToDevice);

	hipBindTexture(0, TR, d_ori_R);
	hipBindTexture(0, TG, d_ori_G);
	hipBindTexture(0, TB, d_ori_B);
	set_filter_up(u0, u1);

	int threads=64;
	int blocks=64;
	for(int i=0; i<(h-1)/(threads*blocks) +1; ++i) // a thread do a row
		run_cuda_row<<<blocks, threads>>>(i*threads*blocks, d_ans_R, d_ans_G, d_ans_B, w, h, ww, hh, temp_R, temp_G, temp_B);
		//run_cuda_row<<<blocks, threads, threads*sizeof(int)*ww*2>>>(i*threads*blocks, d_ans_R, d_ans_G, d_ans_B, w, h, ww, hh, temp_R, temp_G, temp_B);
	
	hipBindTexture(0, TansR, temp_R);
	hipBindTexture(0, TansG, temp_G);
	hipBindTexture(0, TansB, temp_B);

	for(int i=0; i<(ww-1)/(threads*blocks) +1; ++i) // a thread do a column
		run_cuda_col<<<blocks, threads>>>(i*threads*blocks, d_ans_R, d_ans_G, d_ans_B, w, h, ww, hh);
	
	hipMemcpy(aft_R, d_ans_R, ww*hh*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(aft_G, d_ans_G, ww*hh*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(aft_B, d_ans_B, ww*hh*sizeof(int), hipMemcpyDeviceToHost);

	
	hipUnbindTexture(TR);
	hipUnbindTexture(TG);
	hipUnbindTexture(TB);
	hipUnbindTexture(TansR);
	hipUnbindTexture(TansG);
	hipUnbindTexture(TansB);
	
	hipFree(d_ori_R);
	hipFree(d_ori_G);
	hipFree(d_ori_B);
	hipFree(d_ans_R);
	hipFree(d_ans_G);
	hipFree(d_ans_B);
	hipFree(temp_R);
	hipFree(temp_G);
	hipFree(temp_B);
}
