#include "hip/hip_runtime.h"
//#include "SR_kernel_start.cu"
#include <stdio.h>
#include <time.h>

texture<unsigned char, 1, hipReadModeElementType> TR;
texture<unsigned char, 1, hipReadModeElementType> TG;
texture<unsigned char, 1, hipReadModeElementType> TB;
texture<unsigned char ,1, hipReadModeElementType> TansR;
texture<unsigned char ,1, hipReadModeElementType> TansG;
texture<unsigned char ,1, hipReadModeElementType> TansB;

//extern __shared__ int row[];
__constant__ float d_u0[5];
__constant__ float d_u1[5];

extern "C" void set_filter_up(float *u0, float *u1){
	hipMemcpyToSymbol(HIP_SYMBOL(d_u0), u0, 5*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_u1), u1, 5*sizeof(float));
}

__device__ int convolusion_col(int index, int ww, int hh, unsigned char *ans, unsigned char *row0, unsigned char *row1){
	int e_aft=0;
	int temp[2];
	int sum;
	// i==0
	temp[0]=(int)(d_u0[2]*(int)row0[0]+d_u0[3]*(int)row0[1]+d_u0[4]*(int)row0[2]);
	temp[1]=(int)(d_u1[2]*(int)row1[0]+d_u1[3]*(int)row1[1]+d_u1[4]*(int)row1[2]);
	sum=temp[0]+temp[1];
	if(sum>255) sum=255;
	else if(sum<0) sum=0;
	ans[index]=(unsigned char)sum;
	e_aft+=sum;
	// i==1
	temp[0]=(int)(d_u0[1]*(int)row0[0]+d_u0[2]*(int)row0[1]+d_u0[3]*(int)row0[2]+d_u0[4]*(int)row0[3]);
	temp[1]=(int)(d_u1[1]*(int)row1[0]+d_u1[2]*(int)row1[1]+d_u1[3]*(int)row1[2]+d_u1[4]*(int)row1[3]);
	sum=temp[0]+temp[1];
	if(sum>255) sum=255;
	else if(sum<0) sum=0;
	ans[ww +index]=(unsigned char)sum;
	e_aft+=sum;
	// i==hh-2
	temp[0]=(int)(d_u0[0]*(int)row0[hh-4]+d_u0[1]*(int)row0[hh-3]+d_u0[2]*(int)row0[hh-2]+d_u0[3]*(int)row0[hh-1]);
	temp[1]=(int)(d_u1[0]*(int)row1[hh-4]+d_u1[1]*(int)row1[hh-3]+d_u1[2]*(int)row1[hh-2]+d_u1[3]*(int)row1[hh-1]);
	sum=temp[0]+temp[1];
	if(sum>255) sum=255;
	else if(sum<0) sum=0;
	ans[(hh-2)*ww +index]=(unsigned char)sum;
	e_aft+=sum;
	// i==hh-1
	temp[0]=(int)(d_u0[0]*(int)row0[hh-3]+d_u0[1]*(int)row0[hh-2]+d_u0[2]*(int)row0[hh-1]);
	temp[1]=(int)(d_u1[0]*(int)row1[hh-3]+d_u1[1]*(int)row1[hh-2]+d_u1[2]*(int)row1[hh-1]);
	sum=temp[0]+temp[1];
	if(sum>255) sum=255;
	else if(sum<0) sum=0;
	ans[(hh-1)*ww +index]=(unsigned char)sum;
	e_aft+=sum;
	//#pragma unroll
	for(int i=2; i<hh-2; ++i){
		temp[0]=(int)(d_u0[0]*(int)row0[i-2]+d_u0[1]*(int)row0[i-1]+d_u0[2]*(int)row0[i]+d_u0[3]*(int)row0[i+1]+d_u0[4]*(int)row0[i+2]);
		temp[1]=(int)(d_u1[0]*(int)row1[i-2]+d_u1[1]*(int)row1[i-1]+d_u1[2]*(int)row1[i]+d_u1[3]*(int)row1[i+1]+d_u1[4]*(int)row1[i+2]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans[i*ww +index]=(unsigned char)sum;
		e_aft+=sum;
	}
	return e_aft;
}

__device__ int convolusion_row(int a_index, int w, int ww, unsigned char *ans, int index, unsigned char *row0, unsigned char *row1){
	int e_aft=0;
	int temp[2];
	int sum;
	
	// i==0
	temp[0]=(int)(d_u0[2]*(int)row0[0]+d_u0[3]*(int)row0[1]+d_u0[4]*(int)row0[2]);
	temp[1]=(int)(d_u1[2]*(int)row1[0]+d_u1[3]*(int)row1[1]+d_u1[4]*(int)row1[2]);
	sum=temp[0]+temp[1];
	if(sum>255) sum=255;
	else if(sum<0) sum=0;
	ans[a_index*ww]=(unsigned char)sum;
	e_aft+=sum;
	// i==1
	temp[0]=(int)(d_u0[1]*(int)row0[0]+d_u0[2]*(int)row0[1]+d_u0[3]*(int)row0[2]+d_u0[4]*(int)row0[3]);
	temp[1]=(int)(d_u1[1]*(int)row1[0]+d_u1[2]*(int)row1[1]+d_u1[3]*(int)row1[2]+d_u1[4]*(int)row1[3]);
	sum=temp[0]+temp[1];
	if(sum>255) sum=255;
	else if(sum<0) sum=0;
	ans[a_index*ww +1]=(unsigned char)sum;
	e_aft+=sum;
	// i==ww-2
	temp[0]=(int)(d_u0[0]*(int)row0[ww-4]+d_u0[1]*(int)row0[ww-3]+d_u0[2]*(int)row0[ww-2]+d_u0[3]*(int)row0[ww-1]);
	temp[1]=(int)(d_u1[0]*(int)row1[ww-4]+d_u1[1]*(int)row1[ww-3]+d_u1[2]*(int)row1[ww-2]+d_u1[3]*(int)row1[ww-1]);
	sum=temp[0]+temp[1];
	if(sum>255) sum=255;
	else if(sum<0) sum=0;
	ans[a_index*ww +ww-2]=(unsigned char)sum;
	e_aft+=sum;
	// i==ww-1
	temp[0]=(int)(d_u0[0]*(int)row0[ww-3]+d_u0[1]*(int)row0[ww-2]+d_u0[2]*(int)row0[ww-1]);
	temp[1]=(int)(d_u1[0]*(int)row1[ww-3]+d_u1[1]*(int)row1[ww-2]+d_u1[2]*(int)row1[ww-1]);
	sum=temp[0]+temp[1];
	if(sum>255) sum=255;
	else if(sum<0) sum=0;
	ans[a_index*ww +ww-1]=(unsigned char)sum;
	e_aft+=sum;
	
	//#pragma unroll
	for(int i=2; i<ww-2; ++i){
		temp[0]=(int)(d_u0[0]*(int)row0[i-2]+d_u0[1]*(int)row0[i-1]+d_u0[2]*(int)row0[i]+d_u0[3]*(int)row0[i+1]+d_u0[4]*(int)row0[i+2]);
		temp[1]=(int)(d_u1[0]*(int)row1[i-2]+d_u1[1]*(int)row1[i-1]+d_u1[2]*(int)row1[i]+d_u1[3]*(int)row1[i+1]+d_u1[4]*(int)row1[i+2]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans[a_index*ww +i]=(unsigned char)sum;
		e_aft+=sum;
	}
	return e_aft;
}

__global__ void run_cuda_col(int round, unsigned char *ans_R, unsigned char *ans_G, unsigned char *ans_B, int w, int h, int ww, int hh){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(round+tid<ww){
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		int index=(round+tid)*h;
		//#pragma unroll
		for(int i=0; i<h; ++i){ // compute weight
			R_ori+=(int)tex1Dfetch(TansR, index +i);
			G_ori+=(int)tex1Dfetch(TansG, index +i);
			B_ori+=(int)tex1Dfetch(TansB, index +i);
			/*
			R_ori+=ans_R[i*ww +index];
			G_ori+=ans_G[i*ww +index];
			B_ori+=ans_B[i*ww +index];
			*/
		}
		unsigned char row0[1080];
		unsigned char row1[1080];
		// red
		//#pragma unroll
		for(int i=0; i<hh; ++i){
			
			if(i%3==0) row0[i]=tex1Dfetch(TansR, index +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TansR, index +(i-2)*2/3+1);
			else row1[i]=0;
			
			/*
			if(i%3==0) row0[i]=ans_R[(i*2/3)*ww +index];
			else row0[i]=0;

			if(i%3==2) row1[i]=ans_R[((i-2)*2/3+1)*ww +index];
			else row1[i]=0;
			*/
		}
		//e_aft=convolusion_col(round+tid, ww, hh, ans_R, row0, row1);
		int temp[2];
		int sum;
		sum=0;
		e_aft=0;	
		index=round+tid;
		// i==0
		temp[0]=(int)(d_u0[2]*(int)row0[0]+d_u0[3]*(int)row0[1]+d_u0[4]*(int)row0[2]);
		temp[1]=(int)(d_u1[2]*(int)row1[0]+d_u1[3]*(int)row1[1]+d_u1[4]*(int)row1[2]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_R[index]=(unsigned char)sum;
		e_aft+=sum;
		// i==1
		temp[0]=(int)(d_u0[1]*(int)row0[0]+d_u0[2]*(int)row0[1]+d_u0[3]*(int)row0[2]+d_u0[4]*(int)row0[3]);
		temp[1]=(int)(d_u1[1]*(int)row1[0]+d_u1[2]*(int)row1[1]+d_u1[3]*(int)row1[2]+d_u1[4]*(int)row1[3]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_R[ww +index]=(unsigned char)sum;
		e_aft+=sum;
		// i==hh-2
		temp[0]=(int)(d_u0[0]*(int)row0[hh-4]+d_u0[1]*(int)row0[hh-3]+d_u0[2]*(int)row0[hh-2]+d_u0[3]*(int)row0[hh-1]);
		temp[1]=(int)(d_u1[0]*(int)row1[hh-4]+d_u1[1]*(int)row1[hh-3]+d_u1[2]*(int)row1[hh-2]+d_u1[3]*(int)row1[hh-1]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_R[(hh-2)*ww +index]=(unsigned char)sum;
		e_aft+=sum;
		// i==hh-1
		temp[0]=(int)(d_u0[0]*(int)row0[hh-3]+d_u0[1]*(int)row0[hh-2]+d_u0[2]*(int)row0[hh-1]);
		temp[1]=(int)(d_u1[0]*(int)row1[hh-3]+d_u1[1]*(int)row1[hh-2]+d_u1[2]*(int)row1[hh-1]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_R[(hh-1)*ww +index]=(unsigned char)sum;
		e_aft+=sum;
		//#pragma unroll
		for(int i=2; i<hh-2; ++i){
			temp[0]=(int)(d_u0[0]*(int)row0[i-2]+d_u0[1]*(int)row0[i-1]+d_u0[2]*(int)row0[i]+d_u0[3]*(int)row0[i+1]+d_u0[4]*(int)row0[i+2]);
			temp[1]=(int)(d_u1[0]*(int)row1[i-2]+d_u1[1]*(int)row1[i-1]+d_u1[2]*(int)row1[i]+d_u1[3]*(int)row1[i+1]+d_u1[4]*(int)row1[i+2]);
			sum=temp[0]+temp[1];
			if(sum>255) sum=255;
			else if(sum<0) sum=0;
			ans_R[i*ww +index]=(unsigned char)sum;
			e_aft+=sum;
		}
		// convolution finish

		
		R_rate=(float)e_aft/(float)(R_ori*3/2);
		// green
		//#pragma unroll
		index=(round+tid)*h;
		for(int i=0; i<hh; ++i){
			
			if(i%3==0) row0[i]=tex1Dfetch(TansG, index +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TansG, index +(i-2)*2/3+1);
			else row1[i]=0;
			
			/*
			if(i%3==0) row0[i]=ans_G[(i*2/3)*ww +index];
			else row0[i]=0;

			if(i%3==2) row1[i]=ans_G[((i-2)*2/3+1)*ww +index];
			else row1[i]=0;
			*/
		}
		//e_aft=convolusion_col(round+tid, ww, hh, ans_G, row0, row1);
		sum=0;
		e_aft=0;
		index=round+tid;
		// i==0
		temp[0]=(int)(d_u0[2]*(int)row0[0]+d_u0[3]*(int)row0[1]+d_u0[4]*(int)row0[2]);
		temp[1]=(int)(d_u1[2]*(int)row1[0]+d_u1[3]*(int)row1[1]+d_u1[4]*(int)row1[2]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_G[index]=(unsigned char)sum;
		e_aft+=sum;
		// i==1
		temp[0]=(int)(d_u0[1]*(int)row0[0]+d_u0[2]*(int)row0[1]+d_u0[3]*(int)row0[2]+d_u0[4]*(int)row0[3]);
		temp[1]=(int)(d_u1[1]*(int)row1[0]+d_u1[2]*(int)row1[1]+d_u1[3]*(int)row1[2]+d_u1[4]*(int)row1[3]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_G[ww +index]=(unsigned char)sum;
		e_aft+=sum;
		// i==hh-2
		temp[0]=(int)(d_u0[0]*(int)row0[hh-4]+d_u0[1]*(int)row0[hh-3]+d_u0[2]*(int)row0[hh-2]+d_u0[3]*(int)row0[hh-1]);
		temp[1]=(int)(d_u1[0]*(int)row1[hh-4]+d_u1[1]*(int)row1[hh-3]+d_u1[2]*(int)row1[hh-2]+d_u1[3]*(int)row1[hh-1]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_G[(hh-2)*ww +index]=(unsigned char)sum;
		e_aft+=sum;
		// i==hh-1
		temp[0]=(int)(d_u0[0]*(int)row0[hh-3]+d_u0[1]*(int)row0[hh-2]+d_u0[2]*(int)row0[hh-1]);
		temp[1]=(int)(d_u1[0]*(int)row1[hh-3]+d_u1[1]*(int)row1[hh-2]+d_u1[2]*(int)row1[hh-1]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_G[(hh-1)*ww +index]=(unsigned char)sum;
		e_aft+=sum;
		//#pragma unroll
		for(int i=2; i<hh-2; ++i){
			temp[0]=(int)(d_u0[0]*(int)row0[i-2]+d_u0[1]*(int)row0[i-1]+d_u0[2]*(int)row0[i]+d_u0[3]*(int)row0[i+1]+d_u0[4]*(int)row0[i+2]);
			temp[1]=(int)(d_u1[0]*(int)row1[i-2]+d_u1[1]*(int)row1[i-1]+d_u1[2]*(int)row1[i]+d_u1[3]*(int)row1[i+1]+d_u1[4]*(int)row1[i+2]);
			sum=temp[0]+temp[1];
			if(sum>255) sum=255;
			else if(sum<0) sum=0;
			ans_G[i*ww +index]=(unsigned char)sum;
			e_aft+=sum;
		}
		// convolution finish

		G_rate=(float)e_aft/(float)(G_ori*3/2);
		// blue
		//#pragma unroll
		index=(round+tid)*h;
		for(int i=0; i<hh; ++i){
			
			if(i%3==0) row0[i]=tex1Dfetch(TansB, index +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TansB, index +(i-2)*2/3+1);
			else row1[i]=0;
			
			/*
			if(i%3==0) row0[i]=ans_B[(i*2/3)*ww +index];
			else row0[i]=0;

			if(i%3==2) row1[i]=ans_B[((i-2)*2/3+1)*ww +index];
			else row1[i]=0;
			*/
		}
		//e_aft=convolusion_col(round+tid, ww, hh, ans_B, row0, row1);
		sum=0;
		e_aft=0;
		index=round+tid;
		// i==0
		temp[0]=(int)(d_u0[2]*(int)row0[0]+d_u0[3]*(int)row0[1]+d_u0[4]*(int)row0[2]);
		temp[1]=(int)(d_u1[2]*(int)row1[0]+d_u1[3]*(int)row1[1]+d_u1[4]*(int)row1[2]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_B[index]=(unsigned char)sum;
		e_aft+=sum;
		// i==1
		temp[0]=(int)(d_u0[1]*(int)row0[0]+d_u0[2]*(int)row0[1]+d_u0[3]*(int)row0[2]+d_u0[4]*(int)row0[3]);
		temp[1]=(int)(d_u1[1]*(int)row1[0]+d_u1[2]*(int)row1[1]+d_u1[3]*(int)row1[2]+d_u1[4]*(int)row1[3]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_B[ww +index]=(unsigned char)sum;
		e_aft+=sum;
		// i==hh-2
		temp[0]=(int)(d_u0[0]*(int)row0[hh-4]+d_u0[1]*(int)row0[hh-3]+d_u0[2]*(int)row0[hh-2]+d_u0[3]*(int)row0[hh-1]);
		temp[1]=(int)(d_u1[0]*(int)row1[hh-4]+d_u1[1]*(int)row1[hh-3]+d_u1[2]*(int)row1[hh-2]+d_u1[3]*(int)row1[hh-1]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_B[(hh-2)*ww +index]=(unsigned char)sum;
		e_aft+=sum;
		// i==hh-1
		temp[0]=(int)(d_u0[0]*(int)row0[hh-3]+d_u0[1]*(int)row0[hh-2]+d_u0[2]*(int)row0[hh-1]);
		temp[1]=(int)(d_u1[0]*(int)row1[hh-3]+d_u1[1]*(int)row1[hh-2]+d_u1[2]*(int)row1[hh-1]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_B[(hh-1)*ww +index]=(unsigned char)sum;
		e_aft+=sum;
		//#pragma unroll
		for(int i=2; i<hh-2; ++i){
			temp[0]=(int)(d_u0[0]*(int)row0[i-2]+d_u0[1]*(int)row0[i-1]+d_u0[2]*(int)row0[i]+d_u0[3]*(int)row0[i+1]+d_u0[4]*(int)row0[i+2]);
			temp[1]=(int)(d_u1[0]*(int)row1[i-2]+d_u1[1]*(int)row1[i-1]+d_u1[2]*(int)row1[i]+d_u1[3]*(int)row1[i+1]+d_u1[4]*(int)row1[i+2]);
			sum=temp[0]+temp[1];
			if(sum>255) sum=255;
			else if(sum<0) sum=0;
			ans_B[i*ww +index]=(unsigned char)sum;
			e_aft+=sum;
		}
		// convolution finish
		B_rate=(float)e_aft/(float)(B_ori*3/2);
		
		index=round+tid;
		//#pragma unroll
		for(int i=0; i<hh; ++i){
			temp[0]=(int)ans_R[i*ww +index];
			temp[0]=(int)((float)temp[0]/R_rate);
			if(temp[0]>255) temp[0]=255;
			else if(temp[0]<0) temp[0]=0;
			ans_R[i*ww +index]=(unsigned char)temp[0];

			temp[0]=(int)ans_G[i*ww +index];
			temp[0]=(int)((float)temp[0]/G_rate);
			if(temp[0]>255) temp[0]=255;
			else if(temp[0]<0) temp[0]=0;
			ans_G[i*ww +index]=(unsigned char)temp[0];

			temp[0]=(int)ans_B[i*ww +index];
			temp[0]=(int)((float)temp[0]/B_rate);
			if(temp[0]>255) temp[0]=255;
			else if(temp[0]<0) temp[0]=0;
			ans_B[i*ww +index]=(unsigned char)temp[0];
			/*
			ans_R[i*ww +index]=(unsigned char)(int)((float)ans_R[i*ww +index]/R_rate);
			ans_G[i*ww +index]=(unsigned char)(int)((float)ans_G[i*ww +index]/G_rate);
			ans_B[i*ww +index]=(unsigned char)(int)((float)ans_B[i*ww +index]/B_rate);
			*/
			/*
			if(ans_R[i*ww +round+tid]>255) ans_R[i*ww +round+tid]=255;
			else if(ans_R[i*ww +round+tid]<0) ans_R[i*ww +round+tid]=0;

			if(ans_G[i*ww +round+tid]>255) ans_G[i*ww +round+tid]=255;
			else if(ans_G[i*ww +round+tid]<0) ans_G[i*ww +round+tid]=0;

			if(ans_B[i*ww +round+tid]>255) ans_B[i*ww +round+tid]=255;
			else if(ans_B[i*ww +round+tid]<0) ans_B[i*ww +round+tid]=0;
			*/
		}
		
	}
}

__global__ void run_cuda_row(
	int round,
	unsigned char *ans_R, unsigned char *ans_G, unsigned char *ans_B,
	int w, int h, int ww, int hh,
	unsigned char *temp_R, unsigned char *temp_G, unsigned char *temp_B){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	//__shared__ int row[540*2*8];
	if(round+tid<h){
		//test[0]=1139;
		int R_ori=0, G_ori=0, B_ori=0; // store weight of original img
		int e_aft;
		float R_rate, G_rate, B_rate;
		int index=(round+tid)*w;
		//#pragma unroll
		for(int i=0; i<w; ++i){ // compute weight
			R_ori+=(int)tex1Dfetch(TR, index +i);
			G_ori+=(int)tex1Dfetch(TG, index +i);
			B_ori+=(int)tex1Dfetch(TB, index +i);
		}
		
		unsigned char row0[1920];
		unsigned char row1[1920];
		
		// red
		//#pragma unroll
		for(int i=0; i<ww; ++i){ // setup row
			/*
			if(i%3==0) row[threadIdx.x*ww*2 +i]=tex1Dfetch(TR, index +i*2/3);
			else row[threadIdx.x*ww*2 +i]=0;

			if(i%3==2) row[threadIdx.x*ww*2 +ww+i]=tex1Dfetch(TR, index +(i-2)*2/3+1);
			else row[threadIdx.x*ww*2 +ww+i]=0;
			*/
			if(i%3==0) row0[i]=tex1Dfetch(TR, index +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TR, index +(i-2)*2/3+1);
			else row1[i]=0;
			
		}
		//e_aft=convolusion_row(round+tid, w, ww, ans_R, threadIdx.x*ww*2, row0, row1);

		int temp[2];
		int sum;
		e_aft=0;
		index=round+tid;
		// i==0
		temp[0]=(int)(d_u0[2]*(int)row0[0]+d_u0[3]*(int)row0[1]+d_u0[4]*(int)row0[2]);
		temp[1]=(int)(d_u1[2]*(int)row1[0]+d_u1[3]*(int)row1[1]+d_u1[4]*(int)row1[2]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_R[index*ww]=(unsigned char)sum;
		e_aft+=sum;
		// i==1
		temp[0]=(int)(d_u0[1]*(int)row0[0]+d_u0[2]*(int)row0[1]+d_u0[3]*(int)row0[2]+d_u0[4]*(int)row0[3]);
		temp[1]=(int)(d_u1[1]*(int)row1[0]+d_u1[2]*(int)row1[1]+d_u1[3]*(int)row1[2]+d_u1[4]*(int)row1[3]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_R[index*ww +1]=(unsigned char)sum;
		e_aft+=sum;
		// i==ww-2
		temp[0]=(int)(d_u0[0]*(int)row0[ww-4]+d_u0[1]*(int)row0[ww-3]+d_u0[2]*(int)row0[ww-2]+d_u0[3]*(int)row0[ww-1]);
		temp[1]=(int)(d_u1[0]*(int)row1[ww-4]+d_u1[1]*(int)row1[ww-3]+d_u1[2]*(int)row1[ww-2]+d_u1[3]*(int)row1[ww-1]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_R[index*ww +ww-2]=(unsigned char)sum;
		e_aft+=sum;
		// i==ww-1
		temp[0]=(int)(d_u0[0]*(int)row0[ww-3]+d_u0[1]*(int)row0[ww-2]+d_u0[2]*(int)row0[ww-1]);
		temp[1]=(int)(d_u1[0]*(int)row1[ww-3]+d_u1[1]*(int)row1[ww-2]+d_u1[2]*(int)row1[ww-1]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_R[index*ww +ww-1]=(unsigned char)sum;
		e_aft+=sum;
		
		//#pragma unroll
		for(int i=2; i<ww-2; ++i){
			temp[0]=(int)(d_u0[0]*(int)row0[i-2]+d_u0[1]*(int)row0[i-1]+d_u0[2]*(int)row0[i]+d_u0[3]*(int)row0[i+1]+d_u0[4]*(int)row0[i+2]);
			temp[1]=(int)(d_u1[0]*(int)row1[i-2]+d_u1[1]*(int)row1[i-1]+d_u1[2]*(int)row1[i]+d_u1[3]*(int)row1[i+1]+d_u1[4]*(int)row1[i+2]);
			sum=temp[0]+temp[1];
			if(sum>255) sum=255;
			else if(sum<0) sum=0;
			ans_R[index*ww +i]=(unsigned char)sum;
			e_aft+=sum;
		}
		// convolution finish

		R_rate=(float)e_aft/(float)(R_ori*3/2);

		// green
		//#pragma unroll
		index=(round+tid)*w;
		for(int i=0; i<ww; ++i){ // setup row
			/*
			if(i%3==0) row[threadIdx.x*ww*2 +i]=tex1Dfetch(TG, index +i*2/3);
			else row[threadIdx.x*ww*2 +i]=0;

			if(i%3==2) row[threadIdx.x*ww*2 +ww+i]=tex1Dfetch(TG, index +(i-2)*2/3+1);
			else row[threadIdx.x*ww*2 +ww+i]=0;
			*/
			
			if(i%3==0) row0[i]=tex1Dfetch(TG, index +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TG, index +(i-2)*2/3+1);
			else row1[i]=0;
			
		}
		//e_aft=convolusion_row(round+tid, w, ww, ans_G, threadIdx.x*ww*2, row0, row1);

		e_aft=0;
		index=round+tid;
		// i==0
		temp[0]=(int)(d_u0[2]*(int)row0[0]+d_u0[3]*(int)row0[1]+d_u0[4]*(int)row0[2]);
		temp[1]=(int)(d_u1[2]*(int)row1[0]+d_u1[3]*(int)row1[1]+d_u1[4]*(int)row1[2]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_G[index*ww]=(unsigned char)sum;
		e_aft+=sum;
		// i==1
		temp[0]=(int)(d_u0[1]*(int)row0[0]+d_u0[2]*(int)row0[1]+d_u0[3]*(int)row0[2]+d_u0[4]*(int)row0[3]);
		temp[1]=(int)(d_u1[1]*(int)row1[0]+d_u1[2]*(int)row1[1]+d_u1[3]*(int)row1[2]+d_u1[4]*(int)row1[3]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_G[index*ww +1]=(unsigned char)sum;
		e_aft+=sum;
		// i==ww-2
		temp[0]=(int)(d_u0[0]*(int)row0[ww-4]+d_u0[1]*(int)row0[ww-3]+d_u0[2]*(int)row0[ww-2]+d_u0[3]*(int)row0[ww-1]);
		temp[1]=(int)(d_u1[0]*(int)row1[ww-4]+d_u1[1]*(int)row1[ww-3]+d_u1[2]*(int)row1[ww-2]+d_u1[3]*(int)row1[ww-1]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_G[index*ww +ww-2]=(unsigned char)sum;
		e_aft+=sum;
		// i==ww-1
		temp[0]=(int)(d_u0[0]*(int)row0[ww-3]+d_u0[1]*(int)row0[ww-2]+d_u0[2]*(int)row0[ww-1]);
		temp[1]=(int)(d_u1[0]*(int)row1[ww-3]+d_u1[1]*(int)row1[ww-2]+d_u1[2]*(int)row1[ww-1]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_G[index*ww +ww-1]=(unsigned char)sum;
		e_aft+=sum;
	
		//#pragma unroll
		for(int i=2; i<ww-2; ++i){
			temp[0]=(int)(d_u0[0]*(int)row0[i-2]+d_u0[1]*(int)row0[i-1]+d_u0[2]*(int)row0[i]+d_u0[3]*(int)row0[i+1]+d_u0[4]*(int)row0[i+2]);
			temp[1]=(int)(d_u1[0]*(int)row1[i-2]+d_u1[1]*(int)row1[i-1]+d_u1[2]*(int)row1[i]+d_u1[3]*(int)row1[i+1]+d_u1[4]*(int)row1[i+2]);
			sum=temp[0]+temp[1];
			if(sum>255) sum=255;
			else if(sum<0) sum=0;
			ans_G[index*ww +i]=(unsigned char)sum;
			e_aft+=sum;
		}
		// convolution finish
		G_rate=(float)e_aft/(float)(G_ori*3/2);

		// blue
		//#pragma unroll
		index=(round+tid)*w;
		for(int i=0; i<ww; ++i){ // setup row
			/*
			if(i%3==0) row[threadIdx.x*ww*2 +i]=tex1Dfetch(TB, index +i*2/3);
			else row[threadIdx.x*ww*2 +i]=0;

			if(i%3==2) row[threadIdx.x*ww*2 +ww+i]=tex1Dfetch(TB, index +(i-2)*2/3+1);
			else row[threadIdx.x*ww*2 +ww+i]=0;
			*/
			
			if(i%3==0) row0[i]=tex1Dfetch(TB, index +i*2/3);
			else row0[i]=0;

			if(i%3==2) row1[i]=tex1Dfetch(TB, index +(i-2)*2/3+1);
			else row1[i]=0;
			
		}
		//e_aft=convolusion_row(round+tid, w, ww, ans_B, threadIdx.x*ww*2, row0, row1);

		e_aft=0;
		index=round+tid;
		// i==0
		temp[0]=(int)(d_u0[2]*(int)row0[0]+d_u0[3]*(int)row0[1]+d_u0[4]*(int)row0[2]);
		temp[1]=(int)(d_u1[2]*(int)row1[0]+d_u1[3]*(int)row1[1]+d_u1[4]*(int)row1[2]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_B[index*ww]=(unsigned char)sum;
		e_aft+=sum;
		// i==1
		temp[0]=(int)(d_u0[1]*(int)row0[0]+d_u0[2]*(int)row0[1]+d_u0[3]*(int)row0[2]+d_u0[4]*(int)row0[3]);	
		temp[1]=(int)(d_u1[1]*(int)row1[0]+d_u1[2]*(int)row1[1]+d_u1[3]*(int)row1[2]+d_u1[4]*(int)row1[3]);	
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_B[index*ww +1]=(unsigned char)sum;
		e_aft+=sum;
		// i==ww-2
		temp[0]=(int)(d_u0[0]*(int)row0[ww-4]+d_u0[1]*(int)row0[ww-3]+d_u0[2]*(int)row0[ww-2]+d_u0[3]*(int)row0[ww-1]);
		temp[1]=(int)(d_u1[0]*(int)row1[ww-4]+d_u1[1]*(int)row1[ww-3]+d_u1[2]*(int)row1[ww-2]+d_u1[3]*(int)row1[ww-1]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_B[index*ww +ww-2]=(unsigned char)sum;
		e_aft+=sum;
		// i==ww-1
		temp[0]=(int)(d_u0[0]*(int)row0[ww-3]+d_u0[1]*(int)row0[ww-2]+d_u0[2]*(int)row0[ww-1]);
		temp[1]=(int)(d_u1[0]*(int)row1[ww-3]+d_u1[1]*(int)row1[ww-2]+d_u1[2]*(int)row1[ww-1]);
		sum=temp[0]+temp[1];
		if(sum>255) sum=255;
		else if(sum<0) sum=0;
		ans_B[index*ww +ww-1]=(unsigned char)sum;
		e_aft+=sum;
		
		//#pragma unroll
		for(int i=2; i<ww-2; ++i){
			temp[0]=(int)(d_u0[0]*(int)row0[i-2]+d_u0[1]*(int)row0[i-1]+d_u0[2]*(int)row0[i]+d_u0[3]*(int)row0[i+1]+d_u0[4]*(int)row0[i+2]);
			temp[1]=(int)(d_u1[0]*(int)row1[i-2]+d_u1[1]*(int)row1[i-1]+d_u1[2]*(int)row1[i]+d_u1[3]*(int)row1[i+1]+d_u1[4]*(int)row1[i+2]);
			sum=temp[0]+temp[1];
			if(sum>255) sum=255;
			else if(sum<0) sum=0;
			ans_B[index*ww +i]=(unsigned char)sum;
			e_aft+=sum;
		}
		// convolution finish

		B_rate=(float)e_aft/(float)(B_ori*3/2);
		
		index=(round+tid)*ww;
		//#pragma unroll
		for(int i=0; i<ww; ++i){
			temp[0]=(int)ans_R[index +i];
			temp[0]=(int)((float)temp[0]/R_rate);
			if(temp[0]>255) temp[0]=255;
			else if(temp[0]<0) temp[0]=0;
			temp_R[i*h +round+tid]=ans_R[index +i]=(unsigned char)temp[0];

			temp[0]=(int)ans_G[index +i];
			temp[0]=(int)((float)temp[0]/G_rate);
			if(temp[0]>255) temp[0]=255;
			else if(temp[0]<0) temp[0]=0;
			temp_G[i*h +round+tid]=ans_G[index +i]=(unsigned char)temp[0];

			temp[0]=(int)ans_B[index +i];
			temp[0]=(int)((float)temp[0]/B_rate);
			if(temp[0]>255) temp[0]=255;
			else if(temp[0]<0) temp[0]=0;
			temp_B[i*h +round+tid]=ans_B[index +i]=(unsigned char)temp[0];

			/*
			if(ans_R[(round+tid)*ww +i]>255) ans_R[(round+tid)*ww +i]=255;
			else if(ans_R[(round+tid)*ww +i]<0) ans_R[(round+tid)*ww +i]=0;

			if(ans_G[(round+tid)*ww +i]>255) ans_G[(round+tid)*ww +i]=255;
			else if(ans_G[(round+tid)*ww +i]<0) ans_G[(round+tid)*ww +i]=0;

			if(ans_B[(round+tid)*ww +i]>255) ans_B[(round+tid)*ww +i]=255;
			else if(ans_B[(round+tid)*ww +i]<0) ans_B[(round+tid)*ww +i]=0;
			*/
		}
	}

	__syncthreads();
}

void SR_kernel_up(
	unsigned char *ori_R, unsigned char *ori_G, unsigned char *ori_B,
	unsigned char *aft_R, unsigned char *aft_G, unsigned char *aft_B,
	int w, int h, int ww, int hh){
	float u0[5]={-0.047, 0.6, 0.927, 0.119, -0.1};
	float u1[5]={-0.1, 0.119, 0.927, 0.6, -0.047};
	
	unsigned char *d_ori_R, *d_ori_G, *d_ori_B;
	unsigned char *d_ans_R, *d_ans_G, *d_ans_B;
	unsigned char *temp_R, *temp_G, *temp_B;

	//printf("in up, w=%d, h=%d, ww=%d, hh=%d\n", w, h, ww, hh);
	hipMalloc((void**)&d_ori_R, w*h*sizeof(unsigned char));
	hipMalloc((void**)&d_ori_G, w*h*sizeof(unsigned char));
	hipMalloc((void**)&d_ori_B, w*h*sizeof(unsigned char));
	hipMalloc((void**)&temp_R, ww*h*sizeof(unsigned char));
	hipMalloc((void**)&temp_G, ww*h*sizeof(unsigned char));
	hipMalloc((void**)&temp_B, ww*h*sizeof(unsigned char));
	hipMalloc((void**)&d_ans_R, ww*hh*sizeof(unsigned char));
	hipMalloc((void**)&d_ans_G, ww*hh*sizeof(unsigned char));
	hipMalloc((void**)&d_ans_B, ww*hh*sizeof(unsigned char));
	
	hipMemcpy(d_ori_R, ori_R, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(d_ori_G, ori_G, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(d_ori_B, ori_B, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);

	hipBindTexture(0, TR, d_ori_R);
	hipBindTexture(0, TG, d_ori_G);
	hipBindTexture(0, TB, d_ori_B);
	set_filter_up(u0, u1);

	int threads=64;
	int blocks=64;
	for(int i=0; i<(h-1)/(threads*blocks) +1; ++i) // a thread do a row
		run_cuda_row<<<blocks, threads>>>(i*threads*blocks, d_ans_R, d_ans_G, d_ans_B, w, h, ww, hh, temp_R, temp_G, temp_B);
		//run_cuda_row<<<blocks, threads, threads*sizeof(int)*ww*2>>>(i*threads*blocks, d_ans_R, d_ans_G, d_ans_B, w, h, ww, hh, temp_R, temp_G, temp_B);
	
	hipBindTexture(0, TansR, temp_R);
	hipBindTexture(0, TansG, temp_G);
	hipBindTexture(0, TansB, temp_B);

	for(int i=0; i<(ww-1)/(threads*blocks) +1; ++i) // a thread do a column
		run_cuda_col<<<blocks, threads>>>(i*threads*blocks, d_ans_R, d_ans_G, d_ans_B, w, h, ww, hh);
	
	hipMemcpy(aft_R, d_ans_R, ww*hh*sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipMemcpy(aft_G, d_ans_G, ww*hh*sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipMemcpy(aft_B, d_ans_B, ww*hh*sizeof(unsigned char), hipMemcpyDeviceToHost);

	
	hipUnbindTexture(TR);
	hipUnbindTexture(TG);
	hipUnbindTexture(TB);
	hipUnbindTexture(TansR);
	hipUnbindTexture(TansG);
	hipUnbindTexture(TansB);
	
	hipFree(d_ori_R);
	hipFree(d_ori_G);
	hipFree(d_ori_B);
	hipFree(d_ans_R);
	hipFree(d_ans_G);
	hipFree(d_ans_B);
	hipFree(temp_R);
	hipFree(temp_G);
	hipFree(temp_B);
}
